#include "hip/hip_runtime.h"
#include <Windows.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
typedef unsigned __int64 bit64;
typedef unsigned long bit32;
typedef unsigned char bit8;
#define ROL(x,r) ((x) = ((x) << (r)) | ((x) >> (32-(r))))
double PCFreq = 0.0;
__int64 CounterStart = 0;
__int64 blocksize = 512, threadsize = 1024, loop = 1024 * 256;
__int64 totalthreads = blocksize * threadsize;
//#define loop_d 128*1024
#define loop_d 1024*128
bit32 plaintext[4], ciphertext[4], key[8], w[132], k[132], phi = 0x9e3779b9;
bit32 x[4]; //plaintext words
int difference, devicenumber,userloop,pairs=1,numberofrounds, startingpoint=0;
float experiment=100; //number of keys
/* Serpent's S-boxes */
bit32 S[8][16] = {
	{ 0x3, 0x8, 0xf, 0x1, 0xa, 0x6, 0x5, 0xb, 0xe, 0xd, 0x4, 0x2, 0x7, 0x0, 0x9, 0xc },
	{ 0xf, 0xc, 0x2, 0x7, 0x9, 0x0, 0x5, 0xa, 0x1, 0xb, 0xe, 0x8, 0x6, 0xd, 0x3, 0x4 },
	{ 0x8, 0x6, 0x7, 0x9, 0x3, 0xc, 0xa, 0xf, 0xd, 0x1, 0xe, 0x4, 0x0, 0xb, 0x5, 0x2 },
	{ 0x0, 0xf, 0xb, 0x8, 0xc, 0x9, 0x6, 0x3, 0xd, 0x1, 0x2, 0x4, 0xa, 0x7, 0x5, 0xe },
	{ 0x1, 0xf, 0x8, 0x3, 0xc, 0x0, 0xb, 0x6, 0x2, 0x5, 0x4, 0xa, 0x9, 0xe, 0x7, 0xd },
	{ 0xf, 0x5, 0x2, 0xb, 0x4, 0xa, 0x9, 0xc, 0x0, 0x3, 0xe, 0x8, 0xd, 0x6, 0x7, 0x1 },
	{ 0x7, 0x2, 0xc, 0x5, 0x8, 0x4, 0x6, 0xb, 0xe, 0x9, 0x1, 0xf, 0xd, 0x3, 0xa, 0x0 },
	{ 0x1, 0xd, 0xf, 0x0, 0xe, 0x8, 0x2, 0xb, 0x7, 0x4, 0xc, 0xa, 0x9, 0x3, 0x5, 0x6 }
};
void StartCounter(){
	LARGE_INTEGER li;
	if (!QueryPerformanceFrequency(&li))
		printf("QueryPerformanceFrequency failed!\n");

	PCFreq = double(li.QuadPart) / 1000.0;

	QueryPerformanceCounter(&li);
	CounterStart = li.QuadPart;
}
double GetCounter(){
	LARGE_INTEGER li;
	QueryPerformanceCounter(&li);
	return double(li.QuadPart - CounterStart) / PCFreq;
}
void get_user_inputs() {
	key[4] = 0x80000000; key[5] = 0x00000000; key[6] = 0x00000000; key[7] = 0x00000000;
	printf("Select Cuda Device (0,1,2): ");
	scanf("%d", &devicenumber);
	printf(
		"(3) 3-round DL Experiment for [PZWD24]\n"
		"(32) SERPENT Benchmark\n"
		"(4) 4-round DL Experiment for our distinguisher (14 bit left rotation of [DIK08])\n"
		"(5) 5-round DL Experiment for [HDE24]\n"
		"(66) 6-round DL Experiment for [PZWD24] (final correction)\n"
		"(74) 4-round DL Experiment: Middle 4 rounds of 7-round DL of [PZWD24] \n"
		"(75) 5-round DL Experiment: Middle 5 rounds of 7-round DL of [PZWD24] \n"
		"(75) 6-round DL Experiment: Last 6 rounds of 7-round DL of [PZWD24] \n"
		"Choice: "
	);
//	printf("Select Number of rounds (4,5,6,7 or 32 for benchmark): ");
	scanf("%d", &numberofrounds);
	printf("Number of Pairs: 2^36+");
	scanf("%d", &userloop);
	printf("Starting point (default 0): ");
	scanf("%d", &startingpoint);
	for (int i = 0; i<userloop; i++) pairs *= 2;
}
void key_s(int counter, int sbox) {
	int i, j;
	bit32 temp, x_temp[4];
	for (i = 0; i<4; i++) x_temp[i] = 0;
	for (i = 0; i<32; i++) {
		temp = 0;
		for (j = 0; j<4; j++) temp = temp | (((w[4 * counter + j] >> i) & 0x1) << j);
		temp = S[sbox][temp];
		for (j = 0; j<4; j++) x_temp[j] = x_temp[j] | (((temp >> j) & 0x1) << i);
	}
	for (i = 0; i<4; i++) k[4 * counter + i] = x_temp[i];
}
void key_schedule() {
	int i;
	for (i = 0; i<8; i++) w[i] = key[i];
	for (i = 8; i<16; i++) {
		w[i] = w[i - 8] ^ w[i - 5] ^ w[i - 3] ^ w[i - 1] ^ phi ^ (i - 8);
		w[i] = (w[i] << 11) | (w[i] >> 21);
	}
	for (i = 0; i<8; i++) w[i] = w[i + 8];
	// Generate w[i]'s
	for (i = 8; i<132; i++) {
		w[i] = (w[i - 8] ^ w[i - 5] ^ w[i - 3] ^ w[i - 1] ^ phi^i);
		// w[i] = w[i] <<< 11
		w[i] = (w[i] << 11) | (w[i] >> 21);
	}
	// Generate k[i]'s
	key_s(0, 3);
	key_s(1, 2);	key_s(2, 1);	key_s(3, 0);	key_s(4, 7);	key_s(5, 6);	key_s(6, 5);	key_s(7, 4);
	key_s(8, 3);	key_s(9, 2);	key_s(10, 1); key_s(11, 0); key_s(12, 7); key_s(13, 6); key_s(14, 5); key_s(15, 4);
	key_s(16, 3); key_s(17, 2); key_s(18, 1); key_s(19, 0); key_s(20, 7); key_s(21, 6); key_s(22, 5); key_s(23, 4);
	key_s(24, 3); key_s(25, 2); key_s(26, 1); key_s(27, 0); key_s(28, 7); key_s(29, 6); key_s(30, 5); key_s(31, 4); key_s(32, 3);
}
void s(int sbox) {
	int i, j;
	bit32 temp, x_temp[4];
	for (i = 0; i<4; i++) x_temp[i] = 0;
	for (i = 0; i<32; i++) {
		temp = 0;
		for (j = 0; j<4; j++) temp = temp | (((x[j] >> i) & 0x1) << j);
		temp = S[sbox][temp];
		for (j = 0; j<4; j++) x_temp[j] = x_temp[j] | (((temp >> j) & 0x1) << i);
	}
	for (i = 0; i<4; i++) x[i] = x_temp[i];
}
void Sb0(bit32 a, bit32 b, bit32 c, bit32 d) {
	bit32 t1, t3, t4, t7, t12;
	t1 = a ^ d;
	t3 = c ^ t1;
	t4 = b ^ t3;
	x[3] = (a & d) ^ t4;
	t7 = a ^ (b & t1);
	x[2] = t4 ^ (c | t7);
	t12 = x[3] & (t3 ^ t7);
	x[1] = (~t3) ^ t12;
	x[0] = t12 ^ (~t7);
}
void Sb1(bit32 a, bit32 b, bit32 c, bit32 d) {
	bit32 t2, t5, t7, t8, t11;
	t2 = b ^ (~a);
	t5 = c ^ (a | t2);
	x[2] = d ^ t5;
	t7 = b ^ (d | t2);
	t8 = t2 ^ x[2];
	x[3] = t8 ^ (t5 & t7);
	t11 = t5 ^ t7;
	x[1] = x[3] ^ t11;
	x[0] = t5 ^ (t8 & t11);
}
void Sb2(bit32 a, bit32 b, bit32 c, bit32 d)   {
	bit32 t1, t2, t3, t5, t6, t7;
	t1 = ~a;
	t2 = b ^ d;
	t3 = c & t1;
	x[0] = t2 ^ t3;
	t5 = c ^ t1;
	t6 = c ^ x[0];
	t7 = b & t6;
	x[3] = t5 ^ t7;
	x[2] = a ^ ((d | t7) & (x[0] | t5));
	x[1] = (t2 ^ x[3]) ^ (x[2] ^ (d | t1));
}
void Sb3(bit32 a, bit32 b, bit32 c, bit32 d) {
	bit32 t1, t2, t3, t4, t5, t6, t8, t9, t10, t12;
	t1 = a ^ b;
	t2 = a & c;
	t3 = a | d;
	t4 = c ^ d;
	t5 = t1 & t3;
	t6 = t2 | t5;
	x[2] = t4 ^ t6;
	t8 = b ^ t3;
	t9 = t6 ^ t8;
	t10 = t4 & t9;
	x[0] = t1 ^ t10;
	t12 = x[2] & x[0];
	x[1] = t9 ^ t12;
	x[3] = (b | d) ^ (t4 ^ t12);
}
void Sb4(bit32 a, bit32 b, bit32 c, bit32 d){
	bit32 t1, t2, t3, t4, t6, t7, t9, t10, t11;
	t1 = a ^ d;
	t2 = d & t1;
	t3 = c ^ t2;
	t4 = b | t3;
	x[3] = t1 ^ t4;
	t6 = ~b;
	t7 = t1 | t6;
	x[0] = t3 ^ t7;
	t9 = a & x[0];
	t10 = t1 ^ t6;
	t11 = t4 & t10;
	x[2] = t9 ^ t11;
	x[1] = (a ^ t3) ^ (t10 & x[2]);
}
void Sb5(bit32 a, bit32 b, bit32 c, bit32 d) {
	bit32 t1, t2, t3, t4, t5, t7, t8, t10, t11, t12;
	t1 = ~a;
	t2 = a ^ b;
	t3 = a ^ d;
	t4 = c ^ t1;
	t5 = t2 | t3;
	x[0] = t4 ^ t5;
	t7 = d & x[0];
	t8 = t2 ^ x[0];
	x[1] = t7 ^ t8;
	t10 = t1 | x[0];
	t11 = t2 | t7;
	t12 = t3 ^ t10;
	x[2] = t11 ^ t12;
	x[3] = (b ^ t7) ^ (x[1] & t12);
}
void Sb6(bit32 a, bit32 b, bit32 c, bit32 d) {
	bit32 t1, t2, t3, t4, t5, t7, t8, t9, t11;
	t1 = ~a;
	t2 = a ^ d;
	t3 = b ^ t2;
	t4 = t1 | t2;
	t5 = c ^ t4;
	x[1] = b ^ t5;
	t7 = t2 | x[1];
	t8 = d ^ t7;
	t9 = t5 & t8;
	x[2] = t3 ^ t9;
	t11 = t5 ^ t8;
	x[0] = x[2] ^ t11;
	x[3] = (~t5) ^ (t3 & t11);
}
void Sb7(bit32 a, bit32 b, bit32 c, bit32 d) {
	bit32 t1, t2, t3, t4, t5, t6, t8, t9, t11, t12;
	t1 = b ^ c;
	t2 = c & t1;
	t3 = d ^ t2;
	t4 = a ^ t3;
	t5 = d | t1;
	t6 = t4 & t5;
	x[1] = b ^ t6;
	t8 = t3 | x[1];
	t9 = a & t4;
	x[3] = t1 ^ t9;
	t11 = t4 ^ t8;
	t12 = x[3] & t11;
	x[2] = t3 ^ t12;
	x[0] = (~t11) ^ (x[3] & x[2]);
}
void key_addition(int round) {	for (int i = 0; i<4; i++) x[i] = x[i] ^ k[4 * round + i];}
void linear_transformation() {
	// x0 = x0 <<< 13
	x[0] = (x[0] << 13) | x[0] >> 19;
	// x2 <<< 3
	x[2] = (x[2] << 3) | (x[2] >> 29);
	// x1 = x1^x0^x2
	x[1] = x[1] ^ x[0] ^ x[2];
	// x3 = x3^x2^(x0 << 3)
	x[3] = x[3] ^ x[2] ^ (x[0] << 3);
	// x1 = x1 <<< 1
	x[1] = (x[1] << 1) | (x[1] >> 31);
	// x3 = x3 <<< 7
	x[3] = (x[3] << 7) | (x[3] >> 25);
	// x0 = x0 ^ x1 ^ x3
	x[0] = x[0] ^ x[1] ^ x[3];
	// x2 = x2 ^ x3 ^ (x1 << 7)
	x[2] = x[2] ^ x[3] ^ (x[1] << 7);
	// x0 = x0 <<< 5
	x[0] = (x[0] << 5) | (x[0] >> 27);
	// x2 = x2 <<< 22
	x[2] = (x[2] << 22) | (x[2] >> 10);
}
__global__ void Serpent7round(bit32 p0, bit32 p1, bit32 p2, bit32 p3, bit64 *hit, bit32 *k_d) {
	__shared__ bit32  k[132];
	bit32 x0, x1, x2, x3, plaintext0 = p0, plaintext1 = p1, plaintext2 = p2, plaintext3 = p3 + blockIdx.x *blockDim.x + threadIdx.x;
	bit32 ciphertext0, ciphertext1, ciphertext2, ciphertext3;
	bit32 a, b, c, d;
	bit32 t1, t2, t3, t4, t5, t6, t7, t8, t9, t10, t11, t12;
	bit32 round;
	bit64 counter = 0;
	if (threadIdx.x<132) k[threadIdx.x] = k_d[threadIdx.x];
	__syncthreads();
	for (int i = 0; i < loop_d; i++) {
		x0 = plaintext0; x1 = plaintext1; x2 = plaintext2; x3 = plaintext3;
		// Round 2
		round = 2;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = b ^ d; t3 = c & t1; x0 = t2 ^ t3; t5 = c ^ t1; t6 = c ^ x0; t7 = b & t6; x3 = t5 ^ t7; x2 = a ^ ((d | t7) & (x0 | t5)); x1 = (t2 ^ x3) ^ (x2 ^ (d | t1));
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 3
		round = 3;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ b; t2 = a & c; t3 = a | d; t4 = c ^ d; t5 = t1 & t3; t6 = t2 | t5; x2 = t4 ^ t6; t8 = b ^ t3; t9 = t6 ^ t8; t10 = t4 & t9; x0 = t1 ^ t10; t12 = x2 & x0; x1 = t9 ^ t12; x3 = (b | d) ^ (t4 ^ t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 4
		round = 4;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 5
		round = 5;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 6
		round = 6;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 7
		round = 7;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = b ^ c; t2 = c & t1; t3 = d ^ t2; t4 = a ^ t3; t5 = d | t1; t6 = t4 & t5; x1 = b ^ t6; t8 = t3 | x1; t9 = a & t4; x3 = t1 ^ t9; t11 = t4 ^ t8; t12 = x3 & t11; x2 = t3 ^ t12; x0 = (~t11) ^ (x3 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 8
		round = 8;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t3 = c ^ t1; t4 = b ^ t3; x3 = (a & d) ^ t4; t7 = a ^ (b & t1); x2 = t4 ^ (c | t7); t12 = x3 & (t3 ^ t7); x1 = (~t3) ^ t12;	x0 = t12 ^ (~t7);
		
		ciphertext0 = x0; ciphertext1 = x1; ciphertext2 = x2; ciphertext3 = x3;
		// Give difference
		x0 = plaintext0 ^ 0x00000010;
		x1 = plaintext1;
		x2 = plaintext2 ^ 0x00000090;
		x3 = plaintext3;
		// Our Difference
		/*		x0 = plaintext0 ^ 0x00000002;
		x1 = plaintext1;
		x2 = plaintext2 ^ 0x00000012;
		x3 = plaintext3;*/
		// Round 2
		round = 2;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = b ^ d; t3 = c & t1; x0 = t2 ^ t3; t5 = c ^ t1; t6 = c ^ x0; t7 = b & t6; x3 = t5 ^ t7; x2 = a ^ ((d | t7) & (x0 | t5)); x1 = (t2 ^ x3) ^ (x2 ^ (d | t1));
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 3
		round = 3;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ b; t2 = a & c; t3 = a | d; t4 = c ^ d; t5 = t1 & t3; t6 = t2 | t5; x2 = t4 ^ t6; t8 = b ^ t3; t9 = t6 ^ t8; t10 = t4 & t9; x0 = t1 ^ t10; t12 = x2 & x0; x1 = t9 ^ t12; x3 = (b | d) ^ (t4 ^ t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 4
		round = 4;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 5
		round = 5;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 6
		round = 6;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 7
		round = 7;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = b ^ c; t2 = c & t1; t3 = d ^ t2; t4 = a ^ t3; t5 = d | t1; t6 = t4 & t5; x1 = b ^ t6; t8 = t3 | x1; t9 = a & t4; x3 = t1 ^ t9; t11 = t4 ^ t8; t12 = x3 & t11; x2 = t3 ^ t12; x0 = (~t11) ^ (x3 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 8
		round = 8;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t3 = c ^ t1; t4 = b ^ t3; x3 = (a & d) ^ t4; t7 = a ^ (b & t1); x2 = t4 ^ (c | t7); t12 = x3 & (t3 ^ t7); x1 = (~t3) ^ t12;	x0 = t12 ^ (~t7);

		// Check Difference
		x0 ^= ciphertext0;
		t1 = (x0 >> 1) & 0x1;
		t2 = (x0 >> 6) & 0x1;
		t3 = (x0 >> 11) & 0x1;
		t1 ^= t2; t1 ^= t3;
		if (t1 == 0) counter++;

		// Change plaintext
		plaintext0 = x0; plaintext1 = x1; plaintext2 = x2; plaintext3 = x3;
	}
	hit[blockIdx.x *blockDim.x + threadIdx.x] = counter;
}
__global__ void Serpent7round_optimized(bit32 p0, bit32 p1, bit32 p2, bit32 p3, bit64 *hit, bit32 *k_d) {
	__shared__ bit32  k[132];
	bit32 x0, x1, x2, x3, plaintext0 = p0, plaintext1 = p1, plaintext2 = p2, plaintext3 = p3 + blockIdx.x *blockDim.x + threadIdx.x;
	bit32 ciphertext0;
	bit32 a, b, c, d;
	bit32 t1, t2, t3, t4, t5, t6, t7, t8, t9, t10, t11, t12;
	bit64 counter = 0;
	if (threadIdx.x<132) k[threadIdx.x] = k_d[threadIdx.x];
	__syncthreads();
	for (int i = 0; i < loop_d; i++) {
		x0 = plaintext0; x1 = plaintext1; x2 = plaintext2; x3 = plaintext3;
		// Round 2
		x0 ^= k[8];	x1 ^= k[9];	x2 ^= k[10]; x3 ^= k[11];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = b ^ d; t3 = c & t1; x0 = t2 ^ t3; t5 = c ^ t1; t6 = c ^ x0; t7 = b & t6; x3 = t5 ^ t7; x2 = a ^ ((d | t7) & (x0 | t5)); x1 = (t2 ^ x3) ^ (x2 ^ (d | t1));
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 3
		x0 ^= k[12];	x1 ^= k[13];	x2 ^= k[14];	x3 ^= k[15];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ b; t2 = a & c; t3 = a | d; t4 = c ^ d; t5 = t1 & t3; t6 = t2 | t5; x2 = t4 ^ t6; t8 = b ^ t3; t9 = t6 ^ t8; t10 = t4 & t9; x0 = t1 ^ t10; t12 = x2 & x0; x1 = t9 ^ t12; x3 = (b | d) ^ (t4 ^ t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 4
		x0 ^= k[16];	x1 ^= k[17];	x2 ^= k[18];	x3 ^= k[19];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 5
		x0 ^= k[20];	x1 ^= k[21];	x2 ^= k[22];	x3 ^= k[23];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 6
		x0 ^= k[24];	x1 ^= k[25];	x2 ^= k[26];	x3 ^= k[27];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 7
		x0 ^= k[28];	x1 ^= k[29];	x2 ^= k[30];	x3 ^= k[31];
		a = x0; b = x1; c = x2; d = x3;
		t1 = b ^ c; t2 = c & t1; t3 = d ^ t2; t4 = a ^ t3; t5 = d | t1; t6 = t4 & t5; x1 = b ^ t6; t8 = t3 | x1; t9 = a & t4; x3 = t1 ^ t9; t11 = t4 ^ t8; t12 = x3 & t11; x2 = t3 ^ t12; x0 = (~t11) ^ (x3 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 8
		x0 ^= k[32];	x1 ^= k[33];	x2 ^= k[34];	x3 ^= k[35];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t3 = c ^ t1; t4 = b ^ t3; x3 = (a & d) ^ t4; t7 = a ^ (b & t1); x2 = t4 ^ (c | t7); t12 = x3 & (t3 ^ t7); x1 = (~t3) ^ t12; x0 = t12 ^ (~t7);
		// no linear transformation for the last round

		ciphertext0 = x0;
		// Give difference
		x0 = plaintext0 ^ 0x00000010;
		x1 = plaintext1;
		x2 = plaintext2 ^ 0x00000090;
		x3 = plaintext3;
		// Our Difference
		/*		x0 = plaintext0 ^ 0x00000002;
		x1 = plaintext1;
		x2 = plaintext2 ^ 0x00000012;
		x3 = plaintext3;*/
		// Round 2
		x0 ^= k[8];	x1 ^= k[9];	x2 ^= k[10]; x3 ^= k[11];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = b ^ d; t3 = c & t1; x0 = t2 ^ t3; t5 = c ^ t1; t6 = c ^ x0; t7 = b & t6; x3 = t5 ^ t7; x2 = a ^ ((d | t7) & (x0 | t5)); x1 = (t2 ^ x3) ^ (x2 ^ (d | t1));
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 3
		x0 ^= k[12];	x1 ^= k[13];	x2 ^= k[14];	x3 ^= k[15];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ b; t2 = a & c; t3 = a | d; t4 = c ^ d; t5 = t1 & t3; t6 = t2 | t5; x2 = t4 ^ t6; t8 = b ^ t3; t9 = t6 ^ t8; t10 = t4 & t9; x0 = t1 ^ t10; t12 = x2 & x0; x1 = t9 ^ t12; x3 = (b | d) ^ (t4 ^ t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 4
		x0 ^= k[16];	x1 ^= k[17];	x2 ^= k[18];	x3 ^= k[19];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 5
		x0 ^= k[20];	x1 ^= k[21];	x2 ^= k[22];	x3 ^= k[23];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 6
		x0 ^= k[24];	x1 ^= k[25];	x2 ^= k[26];	x3 ^= k[27];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 7
		x0 ^= k[28];	x1 ^= k[29];	x2 ^= k[30];	x3 ^= k[31];
		a = x0; b = x1; c = x2; d = x3;
		t1 = b ^ c; t2 = c & t1; t3 = d ^ t2; t4 = a ^ t3; t5 = d | t1; t6 = t4 & t5; x1 = b ^ t6; t8 = t3 | x1; t9 = a & t4; x3 = t1 ^ t9; t11 = t4 ^ t8; t12 = x3 & t11; x2 = t3 ^ t12; x0 = (~t11) ^ (x3 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 8
		x0 ^= k[32];	x1 ^= k[33];	x2 ^= k[34];	x3 ^= k[35];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t3 = c ^ t1; t4 = b ^ t3; x3 = (a & d) ^ t4; t7 = a ^ (b & t1); x2 = t4 ^ (c | t7); t12 = x3 & (t3 ^ t7); x1 = (~t3) ^ t12; x0 = t12 ^ (~t7);
		// no linear transformation for the last round

		x0 ^= ciphertext0;
		t1 = (x0 >> 1) & 0x1;
		t2 = (x0 >> 6) & 0x1;
		t3 = (x0 >> 11) & 0x1;
		t1 ^= t2; t1 ^= t3;
		if (t1 == 0) counter++;

		// Change plaintext
		plaintext0 = x0; plaintext1 = x1; plaintext2 = x2; plaintext3 = x3;
	}
	hit[blockIdx.x *blockDim.x + threadIdx.x] = counter;
}
__global__ void Serpent6round(bit32 p0, bit32 p1, bit32 p2, bit32 p3, bit64 *hit, bit32 *k_d) {
	__shared__ bit32  k[132];
	bit32 x0, x1, x2, x3, plaintext0 = p0, plaintext1 = p1, plaintext2 = p2, plaintext3 = p3 + blockIdx.x *blockDim.x + threadIdx.x;
	bit32 ciphertext0, ciphertext1, ciphertext2, ciphertext3;
	bit32 a, b, c, d;
	bit32 t1, t2, t3, t4, t5, t6, t7, t8, t9, t10, t11, t12;
	bit32 round;
	bit64 counter = 0;
	if (threadIdx.x<132) k[threadIdx.x] = k_d[threadIdx.x];
	__syncthreads();
	for (int i = 0; i < loop_d; i++) {
		x0 = plaintext0; x1 = plaintext1; x2 = plaintext2; x3 = plaintext3;
		// Round 2
		round = 2;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = b ^ d; t3 = c & t1; x0 = t2 ^ t3; t5 = c ^ t1; t6 = c ^ x0; t7 = b & t6; x3 = t5 ^ t7; x2 = a ^ ((d | t7) & (x0 | t5)); x1 = (t2 ^ x3) ^ (x2 ^ (d | t1));
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 3
		round = 3;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ b; t2 = a & c; t3 = a | d; t4 = c ^ d; t5 = t1 & t3; t6 = t2 | t5; x2 = t4 ^ t6; t8 = b ^ t3; t9 = t6 ^ t8; t10 = t4 & t9; x0 = t1 ^ t10; t12 = x2 & x0; x1 = t9 ^ t12; x3 = (b | d) ^ (t4 ^ t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 4
		round = 4;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 5
		round = 5;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 6
		round = 6;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 7
		round = 7;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = b ^ c; t2 = c & t1; t3 = d ^ t2; t4 = a ^ t3; t5 = d | t1; t6 = t4 & t5; x1 = b ^ t6; t8 = t3 | x1; t9 = a & t4; x3 = t1 ^ t9; t11 = t4 ^ t8; t12 = x3 & t11; x2 = t3 ^ t12; x0 = (~t11) ^ (x3 & x2);
		// no linear transformation for the last round

		ciphertext0 = x0; ciphertext1 = x1; ciphertext2 = x2; ciphertext3 = x3;
		// Give difference
		x0 = plaintext0 ^ 0x00000010;
		x1 = plaintext1;
		x2 = plaintext2 ^ 0x00000090;
		x3 = plaintext3;
		// Our Difference
		/*		x0 = plaintext0 ^ 0x00000002;
		x1 = plaintext1;
		x2 = plaintext2 ^ 0x00000012;
		x3 = plaintext3;*/
		// Round 2
		round = 2;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = b ^ d; t3 = c & t1; x0 = t2 ^ t3; t5 = c ^ t1; t6 = c ^ x0; t7 = b & t6; x3 = t5 ^ t7; x2 = a ^ ((d | t7) & (x0 | t5)); x1 = (t2 ^ x3) ^ (x2 ^ (d | t1));
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 3
		round = 3;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ b; t2 = a & c; t3 = a | d; t4 = c ^ d; t5 = t1 & t3; t6 = t2 | t5; x2 = t4 ^ t6; t8 = b ^ t3; t9 = t6 ^ t8; t10 = t4 & t9; x0 = t1 ^ t10; t12 = x2 & x0; x1 = t9 ^ t12; x3 = (b | d) ^ (t4 ^ t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 4
		round = 4;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 5
		round = 5;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 6
		round = 6;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 7
		round = 7;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = b ^ c; t2 = c & t1; t3 = d ^ t2; t4 = a ^ t3; t5 = d | t1; t6 = t4 & t5; x1 = b ^ t6; t8 = t3 | x1; t9 = a & t4; x3 = t1 ^ t9; t11 = t4 ^ t8; t12 = x3 & t11; x2 = t3 ^ t12; x0 = (~t11) ^ (x3 & x2);
		// no linear transformation for the last round

		// Check Difference
		x0 ^= ciphertext0;
		t1 = (x0 >> 20) & 0x1;
		t2 = (x0 >> 25) & 0x1;
		t1 ^= t2;
		if (t1 == 0) counter++;

		// Change plaintext
		plaintext0 = x0; plaintext1 = x1; plaintext2 = x2; plaintext3 = x3;
	}
	hit[blockIdx.x *blockDim.x + threadIdx.x] = counter;
}
__global__ void Serpent6round_optimized(bit32 p0, bit32 p1, bit32 p2, bit32 p3, bit64 *hit, bit32 *k_d) {
	__shared__ bit32  k[132];
	bit32 x0, x1, x2, x3, plaintext0 = p0, plaintext1 = p1, plaintext2 = p2, plaintext3 = p3 + blockIdx.x *blockDim.x + threadIdx.x;
	bit32 ciphertext0;
	bit32 a, b, c, d;
	bit32 t1, t2, t3, t4, t5, t6, t7, t8, t9, t10, t11, t12;
	bit64 counter = 0;
	if (threadIdx.x<132) k[threadIdx.x] = k_d[threadIdx.x];
	__syncthreads();
	for (int i = 0; i < loop_d; i++) {
		x0 = plaintext0; x1 = plaintext1; x2 = plaintext2; x3 = plaintext3;
		// Round 2
		x0 ^= k[8];	x1 ^= k[9];	x2 ^= k[10]; x3 ^= k[11];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = b ^ d; t3 = c & t1; x0 = t2 ^ t3; t5 = c ^ t1; t6 = c ^ x0; t7 = b & t6; x3 = t5 ^ t7; x2 = a ^ ((d | t7) & (x0 | t5)); x1 = (t2 ^ x3) ^ (x2 ^ (d | t1));
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 3
		x0 ^= k[12];	x1 ^= k[13];	x2 ^= k[14];	x3 ^= k[15];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ b; t2 = a & c; t3 = a | d; t4 = c ^ d; t5 = t1 & t3; t6 = t2 | t5; x2 = t4 ^ t6; t8 = b ^ t3; t9 = t6 ^ t8; t10 = t4 & t9; x0 = t1 ^ t10; t12 = x2 & x0; x1 = t9 ^ t12; x3 = (b | d) ^ (t4 ^ t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 4
		x0 ^= k[16];	x1 ^= k[17];	x2 ^= k[18];	x3 ^= k[19];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 5
		x0 ^= k[20];	x1 ^= k[21];	x2 ^= k[22];	x3 ^= k[23];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 6
		x0 ^= k[24];	x1 ^= k[25];	x2 ^= k[26];	x3 ^= k[27];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 7
		x0 ^= k[28];	x1 ^= k[29];	x2 ^= k[30];	x3 ^= k[31];
		a = x0; b = x1; c = x2; d = x3;
		t1 = b ^ c; t2 = c & t1; t3 = d ^ t2; t4 = a ^ t3; t5 = d | t1; t6 = t4 & t5; x1 = b ^ t6; t8 = t3 | x1; t9 = a & t4; x3 = t1 ^ t9; t11 = t4 ^ t8; t12 = x3 & t11; x2 = t3 ^ t12; x0 = (~t11) ^ (x3 & x2);
		// no linear transformation for the last round

		ciphertext0 = x0; 
		// Give difference
		x0 = plaintext0 ^ (0x00000010 << 14);
		x1 = plaintext1;
		x2 = plaintext2 ^ (0x00000090 << 14);
		x3 = plaintext3;
		// Our Difference
		/*		x0 = plaintext0 ^ 0x00000002;
		x1 = plaintext1;
		x2 = plaintext2 ^ 0x00000012;
		x3 = plaintext3;*/
		// Round 2
		x0 ^= k[8];	x1 ^= k[9];	x2 ^= k[10]; x3 ^= k[11];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = b ^ d; t3 = c & t1; x0 = t2 ^ t3; t5 = c ^ t1; t6 = c ^ x0; t7 = b & t6; x3 = t5 ^ t7; x2 = a ^ ((d | t7) & (x0 | t5)); x1 = (t2 ^ x3) ^ (x2 ^ (d | t1));
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 3
		x0 ^= k[12];	x1 ^= k[13];	x2 ^= k[14];	x3 ^= k[15];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ b; t2 = a & c; t3 = a | d; t4 = c ^ d; t5 = t1 & t3; t6 = t2 | t5; x2 = t4 ^ t6; t8 = b ^ t3; t9 = t6 ^ t8; t10 = t4 & t9; x0 = t1 ^ t10; t12 = x2 & x0; x1 = t9 ^ t12; x3 = (b | d) ^ (t4 ^ t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 4
		x0 ^= k[16];	x1 ^= k[17];	x2 ^= k[18];	x3 ^= k[19];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 5
		x0 ^= k[20];	x1 ^= k[21];	x2 ^= k[22];	x3 ^= k[23];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 6
		x0 ^= k[24];	x1 ^= k[25];	x2 ^= k[26];	x3 ^= k[27];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 7
		x0 ^= k[28];	x1 ^= k[29];	x2 ^= k[30];	x3 ^= k[31];
		a = x0; b = x1; c = x2; d = x3;
		t1 = b ^ c; t2 = c & t1; t3 = d ^ t2; t4 = a ^ t3; t5 = d | t1; t6 = t4 & t5; x1 = b ^ t6; t8 = t3 | x1; t9 = a & t4; x3 = t1 ^ t9; t11 = t4 ^ t8; t12 = x3 & t11; x2 = t3 ^ t12; x0 = (~t11) ^ (x3 & x2);
		// no linear transformation for the last round

		// Check Difference
		x0 ^= ciphertext0;
		t1 = (x0 >> 20) & 0x1;
		t2 = (x0 >> 25) & 0x1;
		t1 ^= t2;
		if (t1 == 0) counter++;

		// Change plaintext
		plaintext0 = x0; plaintext1 = x1; plaintext2 = x2; plaintext3 = x3;
	}
	hit[blockIdx.x *blockDim.x + threadIdx.x] = counter;
}
__global__ void Serpent6round_optimized_eprint(bit32 p0, bit32 p1, bit32 p2, bit32 p3, bit64* hit, bit32* k_d) {
	__shared__ bit32  k[132];
	bit32 x0, x1, x2, x3, plaintext0 = p0, plaintext1 = p1, plaintext2 = p2, plaintext3 = p3 + blockIdx.x * blockDim.x + threadIdx.x;
	bit32 ciphertext0, ciphertext1, ciphertext2, ciphertext3;
	bit32 a, b, c, d;
	bit32 t0, t1, t2, t3, t4, t5, t6, t7, t8, t9, t10, t11, t12;
	bit64 counter = 0;
	if (threadIdx.x < 132) k[threadIdx.x] = k_d[threadIdx.x];
	__syncthreads();
	for (int i = 0; i < loop_d; i++) {
		x0 = plaintext0; x1 = plaintext1; x2 = plaintext2; x3 = plaintext3;
		// Round 2
		x0 ^= k[8];	x1 ^= k[9];	x2 ^= k[10]; x3 ^= k[11];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = b ^ d; t3 = c & t1; x0 = t2 ^ t3; t5 = c ^ t1; t6 = c ^ x0; t7 = b & t6; x3 = t5 ^ t7; x2 = a ^ ((d | t7) & (x0 | t5)); x1 = (t2 ^ x3) ^ (x2 ^ (d | t1));
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 3
		x0 ^= k[12];	x1 ^= k[13];	x2 ^= k[14];	x3 ^= k[15];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ b; t2 = a & c; t3 = a | d; t4 = c ^ d; t5 = t1 & t3; t6 = t2 | t5; x2 = t4 ^ t6; t8 = b ^ t3; t9 = t6 ^ t8; t10 = t4 & t9; x0 = t1 ^ t10; t12 = x2 & x0; x1 = t9 ^ t12; x3 = (b | d) ^ (t4 ^ t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 4
		x0 ^= k[16];	x1 ^= k[17];	x2 ^= k[18];	x3 ^= k[19];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 5
		x0 ^= k[20];	x1 ^= k[21];	x2 ^= k[22];	x3 ^= k[23];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 6
		x0 ^= k[24];	x1 ^= k[25];	x2 ^= k[26];	x3 ^= k[27];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 7
		x0 ^= k[28];	x1 ^= k[29];	x2 ^= k[30];	x3 ^= k[31];
		a = x0; b = x1; c = x2; d = x3;
		t1 = b ^ c; t2 = c & t1; t3 = d ^ t2; t4 = a ^ t3; t5 = d | t1; t6 = t4 & t5; x1 = b ^ t6; t8 = t3 | x1; t9 = a & t4; x3 = t1 ^ t9; t11 = t4 ^ t8; t12 = x3 & t11; x2 = t3 ^ t12; x0 = (~t11) ^ (x3 & x2);
		// no linear transformation for the last round

					// Last round linear transformation
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);

	

		ciphertext0 = x0;
		ciphertext1 = x1;
		ciphertext2 = x2;
		ciphertext3 = x3;
		// Give difference	(eprint)	
		x0 = plaintext0 ^ (0x00800000);
		x1 = plaintext1 ^ (0x04800000);
		x2 = plaintext2 ^ (0x00800000);
		x3 = plaintext3 ^ (0x04000000);

		// Our best difference
/*		x0 = plaintext0 ^ (0x00000010 << 14);
		x1 = plaintext1;
		x2 = plaintext2 ^ (0x00000090 << 14);
		x3 = plaintext3;*/


		// Round 2
		x0 ^= k[8];	x1 ^= k[9];	x2 ^= k[10]; x3 ^= k[11];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = b ^ d; t3 = c & t1; x0 = t2 ^ t3; t5 = c ^ t1; t6 = c ^ x0; t7 = b & t6; x3 = t5 ^ t7; x2 = a ^ ((d | t7) & (x0 | t5)); x1 = (t2 ^ x3) ^ (x2 ^ (d | t1));
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 3
		x0 ^= k[12];	x1 ^= k[13];	x2 ^= k[14];	x3 ^= k[15];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ b; t2 = a & c; t3 = a | d; t4 = c ^ d; t5 = t1 & t3; t6 = t2 | t5; x2 = t4 ^ t6; t8 = b ^ t3; t9 = t6 ^ t8; t10 = t4 & t9; x0 = t1 ^ t10; t12 = x2 & x0; x1 = t9 ^ t12; x3 = (b | d) ^ (t4 ^ t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 4
		x0 ^= k[16];	x1 ^= k[17];	x2 ^= k[18];	x3 ^= k[19];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 5
		x0 ^= k[20];	x1 ^= k[21];	x2 ^= k[22];	x3 ^= k[23];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 6
		x0 ^= k[24];	x1 ^= k[25];	x2 ^= k[26];	x3 ^= k[27];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 7
		x0 ^= k[28];	x1 ^= k[29];	x2 ^= k[30];	x3 ^= k[31];
		a = x0; b = x1; c = x2; d = x3;
		t1 = b ^ c; t2 = c & t1; t3 = d ^ t2; t4 = a ^ t3; t5 = d | t1; t6 = t4 & t5; x1 = b ^ t6; t8 = t3 | x1; t9 = a & t4; x3 = t1 ^ t9; t11 = t4 ^ t8; t12 = x3 & t11; x2 = t3 ^ t12; x0 = (~t11) ^ (x3 & x2);
		// no linear transformation for the last round

					// Last round linear transformation
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);

		// Check Difference
		x0 ^= ciphertext0;
		x1 ^= ciphertext1;
		x2 ^= ciphertext2;
		x3 ^= ciphertext3;
		
		// Masked bits (eprint)
/*		t0 = x0 & 0x84001080;
		t1 = x1 & 0x0c284084;
		t2 = x2 & 0x00090800;
		t3 = x3 & 0x20000084;*/

		//mask without the final transformation (WRONG!!!!)
/*		t0 = x0 & 0x42000000;
		t1 = x1 & 0x42000000;
		t2 = x2 & 0x40000000;
		t3 = x3 & 0x00000000;*/
		

		t0 = x0 & 0x84001080;
		t1 = x1 & 0x0c284084;
		t2 = x2 & 0x00090800;
		t3 = x3 & 0x20000084;


		// Our best mask
/*		t0 = x0 & 0x02100000;
		t1 = x1 & 0x00000000;
		t2 = x2 & 0x00000000;
		t3 = x3 & 0x00000000;*/

		t0 ^= t0 >> 1;
		t0 ^= t0 >> 2;
		t0 = (t0 & 0x11111111U) * 0x11111111U;
		t0 = (t0 >> 28) & 1;

		t1 ^= t1 >> 1;
		t1 ^= t1 >> 2;
		t1 = (t1 & 0x11111111U) * 0x11111111U;
		t1 = (t1 >> 28) & 1;

		t2 ^= t2 >> 1;
		t2 ^= t2 >> 2;
		t2 = (t2 & 0x11111111U) * 0x11111111U;
		t2 = (t2 >> 28) & 1;

		t3 ^= t3 >> 1;
		t3 ^= t3 >> 2;
		t3 = (t3 & 0x11111111U) * 0x11111111U;
		t3 = (t3 >> 28) & 1;

		t0 = t0 ^ t1 ^ t2 ^ t3;

		if (t0 == 0) counter++;

		// Change plaintext
		plaintext0 = x0; plaintext1 = x1; plaintext2 = x2; plaintext3 = x3;
	}
	hit[blockIdx.x * blockDim.x + threadIdx.x] = counter;
}
__global__ void Serpent5round(bit32 p0, bit32 p1, bit32 p2, bit32 p3, bit64 *hit, bit32 *k_d) {
	__shared__ bit32  k[132];
	bit32 x0, x1, x2, x3, plaintext0 = p0, plaintext1 = p1, plaintext2 = p2, plaintext3 = p3 + blockIdx.x *blockDim.x + threadIdx.x;
	bit32 ciphertext0;// , ciphertext1, ciphertext2, ciphertext3;
	bit32 a, b, c, d;
	bit32 t1, t2, t3, t4, t5, t6, t7, t8, t9, t10, t11, t12;
	bit32 round;
	bit64 counter=0;
	if (threadIdx.x<132) k[threadIdx.x] = k_d[threadIdx.x];
	__syncthreads();
	for (int i = 0; i < loop_d; i++) {
		x0 = plaintext0; x1 = plaintext1; x2 = plaintext2; x3 = plaintext3;
		// Round 2
		round = 2;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = b ^ d; t3 = c & t1; x0 = t2 ^ t3; t5 = c ^ t1; t6 = c ^ x0; t7 = b & t6; x3 = t5 ^ t7; x2 = a ^ ((d | t7) & (x0 | t5)); x1 = (t2 ^ x3) ^ (x2 ^ (d | t1));
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 3
		round = 3;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ b; t2 = a & c; t3 = a | d; t4 = c ^ d; t5 = t1 & t3; t6 = t2 | t5; x2 = t4 ^ t6; t8 = b ^ t3; t9 = t6 ^ t8; t10 = t4 & t9; x0 = t1 ^ t10; t12 = x2 & x0; x1 = t9 ^ t12; x3 = (b | d) ^ (t4 ^ t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 4
		round = 4;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 5
		round = 5;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 6
		round = 6;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		// no linear transformation for the last round
		ciphertext0 = x0;// ciphertext1 = x1; ciphertext2 = x2; ciphertext3=x3;
		// Give difference
		x0 = plaintext0 ^ (0x00000010<<14);
		x1 = plaintext1;
		x2 = plaintext2 ^ (0x00000090<<14);
		x3 = plaintext3;
		// Our Difference
/*		x0 = plaintext0 ^ 0x00000002;
		x1 = plaintext1;
		x2 = plaintext2 ^ 0x00000012;
		x3 = plaintext3;*/
		// Round 2
		round = 2;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = b ^ d; t3 = c & t1; x0 = t2 ^ t3; t5 = c ^ t1; t6 = c ^ x0; t7 = b & t6; x3 = t5 ^ t7; x2 = a ^ ((d | t7) & (x0 | t5)); x1 = (t2 ^ x3) ^ (x2 ^ (d | t1));
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 3
		round = 3;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ b; t2 = a & c; t3 = a | d; t4 = c ^ d; t5 = t1 & t3; t6 = t2 | t5; x2 = t4 ^ t6; t8 = b ^ t3; t9 = t6 ^ t8; t10 = t4 & t9; x0 = t1 ^ t10; t12 = x2 & x0; x1 = t9 ^ t12; x3 = (b | d) ^ (t4 ^ t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 4
		round = 4;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 5
		round = 5;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 6
		round = 6;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		// no linear transformation for the last round

		// Check Difference
		x0 ^= ciphertext0;
		t1 = (x0 >> 7) & 0x1;
		if (t1 == 0) counter++;

		// Change plaintext
		plaintext0 = x0; plaintext1 = x1; plaintext2 = x2; plaintext3 = x3;
	}
	hit[blockIdx.x *blockDim.x + threadIdx.x] = counter;
}
__global__ void Serpent5round_eprint(bit32 p0, bit32 p1, bit32 p2, bit32 p3, bit64* hit, bit32* k_d) {
	__shared__ bit32  k[132];
	bit32 x0, x1, x2, x3, plaintext0 = p0, plaintext1 = p1, plaintext2 = p2, plaintext3 = p3 + blockIdx.x * blockDim.x + threadIdx.x;
	bit32 ciphertext0, ciphertext1, ciphertext2, ciphertext3;
	bit32 a, b, c, d;
	bit32 t0,t1, t2, t3, t4, t5, t6, t7, t8, t9, t10, t11, t12;
	bit32 round;
	bit64 counter = 0;
	if (threadIdx.x < 132) k[threadIdx.x] = k_d[threadIdx.x];
	__syncthreads();
	for (int i = 0; i < loop_d; i++) {
		x0 = plaintext0; x1 = plaintext1; x2 = plaintext2; x3 = plaintext3;
		// Round 2
		round = 2;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = b ^ d; t3 = c & t1; x0 = t2 ^ t3; t5 = c ^ t1; t6 = c ^ x0; t7 = b & t6; x3 = t5 ^ t7; x2 = a ^ ((d | t7) & (x0 | t5)); x1 = (t2 ^ x3) ^ (x2 ^ (d | t1));
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 3
		round = 3;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ b; t2 = a & c; t3 = a | d; t4 = c ^ d; t5 = t1 & t3; t6 = t2 | t5; x2 = t4 ^ t6; t8 = b ^ t3; t9 = t6 ^ t8; t10 = t4 & t9; x0 = t1 ^ t10; t12 = x2 & x0; x1 = t9 ^ t12; x3 = (b | d) ^ (t4 ^ t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 4
		round = 4;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 5
		round = 5;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 6
		round = 6;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		// no linear transformation for the last round

			// Last round linear transformation
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);

		ciphertext0 = x0;
		ciphertext1 = x1;
		ciphertext2 = x2;
		ciphertext3 = x3;
		// Give difference	(eprint)	
		
		x0 = plaintext0 ^ (0x01000000);
		x1 = plaintext1 ^ (0x01000000);
		x2 = plaintext2 ^ (0x09000000);
		x3 = plaintext3 ^ (0x00000000);

		// Round 2
		round = 2;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = b ^ d; t3 = c & t1; x0 = t2 ^ t3; t5 = c ^ t1; t6 = c ^ x0; t7 = b & t6; x3 = t5 ^ t7; x2 = a ^ ((d | t7) & (x0 | t5)); x1 = (t2 ^ x3) ^ (x2 ^ (d | t1));
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 3
		round = 3;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ b; t2 = a & c; t3 = a | d; t4 = c ^ d; t5 = t1 & t3; t6 = t2 | t5; x2 = t4 ^ t6; t8 = b ^ t3; t9 = t6 ^ t8; t10 = t4 & t9; x0 = t1 ^ t10; t12 = x2 & x0; x1 = t9 ^ t12; x3 = (b | d) ^ (t4 ^ t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 4
		round = 4;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 5
		round = 5;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 6
		round = 6;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		// no linear transformation for the last round

			// Last round linear transformation
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);

			// Check Difference
		x0 ^= ciphertext0;
		x1 ^= ciphertext1;
		x2 ^= ciphertext2;
		x3 ^= ciphertext3;

		// Masked bits (eprint)
/*		t0 = x0 & 0x84001080;
		t1 = x1 & 0x0c284084;
		t2 = x2 & 0x00090800;
		t3 = x3 & 0x20000084;*/

		// without the final transformation
/*		t0 = x0 & 0x00000004;
		t1 = x1 & 0x00000004;
		t2 = x2 & 0x00000000;
		t3 = x3 & 0x00000004;*/

		t0 = x0 & 0x42004000;
		t1 = x1 & 0x02100228;
		t2 = x2 & 0x00000005;
		t3 = x3 & 0x02181600;



		// Our best mask
/*		t0 = x0 & 0x02100000;
		t1 = x1 & 0x00000000;
		t2 = x2 & 0x00000000;
		t3 = x3 & 0x00000000;*/

		t0 ^= t0 >> 1;
		t0 ^= t0 >> 2;
		t0 = (t0 & 0x11111111U) * 0x11111111U;
		t0 = (t0 >> 28) & 1;

		t1 ^= t1 >> 1;
		t1 ^= t1 >> 2;
		t1 = (t1 & 0x11111111U) * 0x11111111U;
		t1 = (t1 >> 28) & 1;

		t2 ^= t2 >> 1;
		t2 ^= t2 >> 2;
		t2 = (t2 & 0x11111111U) * 0x11111111U;
		t2 = (t2 >> 28) & 1;

		t3 ^= t3 >> 1;
		t3 ^= t3 >> 2;
		t3 = (t3 & 0x11111111U) * 0x11111111U;
		t3 = (t3 >> 28) & 1;

		t0 = t0 ^ t1 ^ t2 ^ t3;

		if (t0 == 0) counter++;

		// Change plaintext
		plaintext0 = x0; plaintext1 = x1; plaintext2 = x2; plaintext3 = x3;
	}
	hit[blockIdx.x * blockDim.x + threadIdx.x] = counter;
}
__global__ void Serpent4round(bit32 p0, bit32 p1, bit32 p2, bit32 p3, bit64 *hit, bit32 *k_d) {
	__shared__ bit32  k[132];
	bit32 x0, x1, x2, x3, plaintext0 = p0, plaintext1 = p1, plaintext2 = p2, plaintext3 = p3 + blockIdx.x *blockDim.x + threadIdx.x;
	bit32 a, b, c, d;
	bit32 ciphertext0, ciphertext1, ciphertext2, ciphertext3;
	bit32 t1, t2, t3, t4, t5, t6, t7, t8, t9, t10, t11, t12;
	bit32 round;
	bit64 counter = 0;
	if (threadIdx.x<132) k[threadIdx.x] = k_d[threadIdx.x];
	__syncthreads();
	for (int i = 0; i < loop_d; i++) {
		x0 = plaintext0; x1 = plaintext1; x2 = plaintext2; x3 = plaintext3;
		// Round 2
		round = 2;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = b ^ d; t3 = c & t1; x0 = t2 ^ t3; t5 = c ^ t1; t6 = c ^ x0; t7 = b & t6; x3 = t5 ^ t7; x2 = a ^ ((d | t7) & (x0 | t5)); x1 = (t2 ^ x3) ^ (x2 ^ (d | t1));
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 3
		round = 3;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ b; t2 = a & c; t3 = a | d; t4 = c ^ d; t5 = t1 & t3; t6 = t2 | t5; x2 = t4 ^ t6; t8 = b ^ t3; t9 = t6 ^ t8; t10 = t4 & t9; x0 = t1 ^ t10; t12 = x2 & x0; x1 = t9 ^ t12; x3 = (b | d) ^ (t4 ^ t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 4
		round = 4;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 5
		round = 5;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		ciphertext0 = x0; ciphertext1 = x1; ciphertext2 = x2; ciphertext3 = x3;

		// Give difference
		x0 = plaintext0 ^ (0x00000010<<14);
		x1 = plaintext1;
		x2 = plaintext2 ^ (0x00000090<<14);
		x3 = plaintext3;
		// Round 2
		round = 2;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = b ^ d; t3 = c & t1; x0 = t2 ^ t3; t5 = c ^ t1; t6 = c ^ x0; t7 = b & t6; x3 = t5 ^ t7; x2 = a ^ ((d | t7) & (x0 | t5)); x1 = (t2 ^ x3) ^ (x2 ^ (d | t1));
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 3
		round = 3;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ b; t2 = a & c; t3 = a | d; t4 = c ^ d; t5 = t1 & t3; t6 = t2 | t5; x2 = t4 ^ t6; t8 = b ^ t3; t9 = t6 ^ t8; t10 = t4 & t9; x0 = t1 ^ t10; t12 = x2 & x0; x1 = t9 ^ t12; x3 = (b | d) ^ (t4 ^ t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 4
		round = 4;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 5
		round = 5;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);

		// Check Difference
		x2 ^= ciphertext2;
		x3 ^= ciphertext3;
		t1 = (x2 >> 29) & 0x1;
		t2 = x3 & 0x1;
		t1 ^= t2;
		if (t1 == 0) counter++;

		// Change plaintext
		plaintext0 = x0; plaintext1 = x1; plaintext2 = x2; plaintext3 = x3;
	}
	hit[blockIdx.x *blockDim.x + threadIdx.x] = counter;
}
__global__ void Serpent3round_PZWD24(bit32 p0, bit32 p1, bit32 p2, bit32 p3, bit64* hit, bit32* k_d) {
	__shared__ bit32  k[132];
	bit32 x0, x1, x2, x3, plaintext0 = p0, plaintext1 = p1, plaintext2 = p2, plaintext3 = p3 + blockIdx.x * blockDim.x + threadIdx.x;
	bit32 a, b, c, d;
	bit32 ciphertext0, ciphertext1, ciphertext2, ciphertext3;
	bit32 t0, t1, t2, t3, t4, t5, t6, t7, t8, t9, t10, t11, t12;
	bit32 round;
	bit64 counter = 0;
	if (threadIdx.x < 132) k[threadIdx.x] = k_d[threadIdx.x];
	__syncthreads();
	for (int i = 0; i < loop_d; i++) {
		x0 = plaintext0; x1 = plaintext1; x2 = plaintext2; x3 = plaintext3;
		// Round 5
		round = 5;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 6
		round = 6;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 7
		round = 7;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = b ^ c; t2 = c & t1; t3 = d ^ t2; t4 = a ^ t3; t5 = d | t1; t6 = t4 & t5; x1 = b ^ t6; t8 = t3 | x1; t9 = a & t4; x3 = t1 ^ t9; t11 = t4 ^ t8; t12 = x3 & t11; x2 = t3 ^ t12; x0 = (~t11) ^ (x3 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);

		// Give difference
		ciphertext0 = x0; ciphertext1 = x1; ciphertext2 = x2; ciphertext3 = x3;		
		x0 = plaintext0 ^ 0x01000000;
		x1 = plaintext1 ^ 0x01000000;
		x2 = plaintext2 ^ 0x00000000;
		x3 = plaintext3 ^ 0x00000000;

		// Round 5
		round = 5;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 6
		round = 6;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 7
		round = 7;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = b ^ c; t2 = c & t1; t3 = d ^ t2; t4 = a ^ t3; t5 = d | t1; t6 = t4 & t5; x1 = b ^ t6; t8 = t3 | x1; t9 = a & t4; x3 = t1 ^ t9; t11 = t4 ^ t8; t12 = x3 & t11; x2 = t3 ^ t12; x0 = (~t11) ^ (x3 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);

		// Check Difference
/*		t1 = x0 ^ ciphertext0;
		t2 = x1 ^ ciphertext1;
		t3 = x2 ^ ciphertext2;
		t4 = x3 ^ ciphertext3;

		bit32 result = 0;
		t1 = t1 & 0x08010812;
		t2 = t2 & 0xa00a2004;
		t3 = t3 & 0x00000010;
		t4 = t4 & 0x000a0000;
		for (int i = 0; i < 32; i++) {
			result ^= ((t1 >> i) & 0x1);
			result ^= ((t2 >> i) & 0x1);
			result ^= ((t3 >> i) & 0x1);
			result ^= ((t4 >> i) & 0x1);
		}
		if (result == 0) counter++;*/

/*		t0 = x0 & 0x44900010;
		t1 = x1 & 0x00010000;
		t2 = x2 & 0x01980010;
		t3 = x3 & 0x12000200;*/
		x0 ^= ciphertext0;
		x1 ^= ciphertext1;
		x2 ^= ciphertext2;
		x3 ^= ciphertext3;

		t0 = x0 & 0x12000200;
		t1 = x1 & 0x01980010;
		t2 = x2 & 0x00010000;
		t3 = x3 & 0x44900010;

		t0 ^= t0 >> 1;
		t0 ^= t0 >> 2;
		t0 = (t0 & 0x11111111U) * 0x11111111U;
		t0 = (t0 >> 28) & 1;

		t1 ^= t1 >> 1;
		t1 ^= t1 >> 2;
		t1 = (t1 & 0x11111111U) * 0x11111111U;
		t1 = (t1 >> 28) & 1;

		t2 ^= t2 >> 1;
		t2 ^= t2 >> 2;
		t2 = (t2 & 0x11111111U) * 0x11111111U;
		t2 = (t2 >> 28) & 1;

		t3 ^= t3 >> 1;
		t3 ^= t3 >> 2;
		t3 = (t3 & 0x11111111U) * 0x11111111U;
		t3 = (t3 >> 28) & 1;

		t0 = t0 ^ t1 ^ t2 ^ t3;

		if (t0 == 0) counter++;

		// Change plaintext
		plaintext0 = x0; plaintext1 = x1; plaintext2 = x2; plaintext3 = x3;
	}
	hit[blockIdx.x * blockDim.x + threadIdx.x] = counter;
}
__global__ void Serpent3round_PZWD24_new(bit32 p0, bit32 p1, bit32 p2, bit32 p3, bit64* hit, bit32* k_d) {
	__shared__ bit32  k[132];
	bit32 x0, x1, x2, x3, x4, plaintext0 = p0, plaintext1 = p1, plaintext2 = p2, plaintext3 = p3 + blockIdx.x * blockDim.x + threadIdx.x;
	bit32 a, b, c, d;
	bit32 ciphertext0, ciphertext1, ciphertext2, ciphertext3;
	bit32 t0, t1, t2, t3, t4, t5, t6, t7, t8, t9, t10, t11, t12;
	bit32 round;
	bit64 counter = 0;
	if (threadIdx.x < 132) k[threadIdx.x] = k_d[threadIdx.x];
	__syncthreads();
	for (int i = 0; i < loop_d; i++) {
		x0 = plaintext0; x1 = plaintext1; x2 = plaintext2; x3 = plaintext3;
		// Round 5
		round = 5;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		x4 = x1; x1 |= x0; x2 ^= x1;	x3 = ~x3;	x4 ^= x0; x0 ^= x2;	x1 &= x4;	x4 |= x3; x4 ^= x0;	x0 &= x3;	x1 ^= x3;	x3 ^= x2;	x0 ^= x1;	x2 &= x4; x1 ^= x2;	x2 &= x0; x3 ^= x2;
		ROL(x0, 13); ROL(x2, 3);	x1 ^= x0;	x4 = x0 << 3; x3 ^= x2;	x1 ^= x2; ROL(x1, 1);	x3 ^= x4; ROL(x3, 7);	x4 = x1; x0 ^= x1;	x4 <<= 7;	x2 ^= x3; x0 ^= x3;	x2 ^= x4;	ROL(x0, 5);	ROL(x2, 22); 
		// Round 6
		round = 6;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		x4 = x1; x3 ^= x0;	x1 ^= x2;	x2 ^= x0; x0 &= x3;	x1 |= x3;	x4 = ~x4; x0 ^= x1;	x1 ^= x2; x3 ^= x4;	x4 ^= x0;	x2 &= x0; x4 ^= x1;	x2 ^= x3;	x3 &= x1; x3 ^= x0;	x1 ^= x2;
		ROL(x0, 13); ROL(x2, 3);	x1 ^= x0;	x4 = x0 << 3; x3 ^= x2;	x1 ^= x2; ROL(x1, 1);	x3 ^= x4; ROL(x3, 7);	x4 = x1; x0 ^= x1;	x4 <<= 7;	x2 ^= x3; x0 ^= x3;	x2 ^= x4;	ROL(x0, 5);	ROL(x2, 22);
		// Round 7
		round = 7;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		x1 = ~x1; x4 = x1;	x0 = ~x0;	x1 &= x2;	x1 ^= x3;	x3 |= x4;	x4 ^= x2;	x2 ^= x3;	x3 ^= x0;	x0 |= x1;	x2 &= x0;	x0 ^= x4;	x4 ^= x3;	x3 &= x0;	x4 ^= x1; x2 ^= x4;	x3 ^= x1;	x4 |= x0; x4 ^= x1;
		ROL(x0, 13); ROL(x2, 3);	x1 ^= x0;	x4 = x0 << 3; x3 ^= x2;	x1 ^= x2; ROL(x1, 1);	x3 ^= x4; ROL(x3, 7);	x4 = x1; x0 ^= x1;	x4 <<= 7;	x2 ^= x3; x0 ^= x3;	x2 ^= x4;	ROL(x0, 5);	ROL(x2, 22);

		// Give difference
		ciphertext0 = x0; ciphertext1 = x1; ciphertext2 = x2; ciphertext3 = x3;
		x0 = plaintext0 ^ 0x01000000;
		x1 = plaintext1 ^ 0x01000000;
		x2 = plaintext2 ^ 0x00000000;
		x3 = plaintext3 ^ 0x00000000;

		// Round 5
		round = 5;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		x4 = x1; x1 |= x0; x2 ^= x1;	x3 = ~x3;	x4 ^= x0; x0 ^= x2;	x1 &= x4;	x4 |= x3; x4 ^= x0;	x0 &= x3;	x1 ^= x3;	x3 ^= x2;	x0 ^= x1;	x2 &= x4; x1 ^= x2;	x2 &= x0; x3 ^= x2;
		ROL(x0, 13); ROL(x2, 3);	x1 ^= x0;	x4 = x0 << 3; x3 ^= x2;	x1 ^= x2; ROL(x1, 1);	x3 ^= x4; ROL(x3, 7);	x4 = x1; x0 ^= x1;	x4 <<= 7;	x2 ^= x3; x0 ^= x3;	x2 ^= x4;	ROL(x0, 5);	ROL(x2, 22);
		// Round 6
		round = 6;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		x4 = x1; x3 ^= x0;	x1 ^= x2;	x2 ^= x0; x0 &= x3;	x1 |= x3;	x4 = ~x4; x0 ^= x1;	x1 ^= x2; x3 ^= x4;	x4 ^= x0;	x2 &= x0; x4 ^= x1;	x2 ^= x3;	x3 &= x1; x3 ^= x0;	x1 ^= x2;
		ROL(x0, 13); ROL(x2, 3);	x1 ^= x0;	x4 = x0 << 3; x3 ^= x2;	x1 ^= x2; ROL(x1, 1);	x3 ^= x4; ROL(x3, 7);	x4 = x1; x0 ^= x1;	x4 <<= 7;	x2 ^= x3; x0 ^= x3;	x2 ^= x4;	ROL(x0, 5);	ROL(x2, 22);
		// Round 7
		round = 7;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		x1 = ~x1; x4 = x1;	x0 = ~x0;	x1 &= x2;	x1 ^= x3;	x3 |= x4;	x4 ^= x2;	x2 ^= x3;	x3 ^= x0;	x0 |= x1;	x2 &= x0;	x0 ^= x4;	x4 ^= x3;	x3 &= x0;	x4 ^= x1; x2 ^= x4;	x3 ^= x1;	x4 |= x0; x4 ^= x1;
		ROL(x0, 13); ROL(x2, 3);	x1 ^= x0;	x4 = x0 << 3; x3 ^= x2;	x1 ^= x2; ROL(x1, 1);	x3 ^= x4; ROL(x3, 7);	x4 = x1; x0 ^= x1;	x4 <<= 7;	x2 ^= x3; x0 ^= x3;	x2 ^= x4;	ROL(x0, 5);	ROL(x2, 22);

		// Check Difference
/*		t1 = x0 ^ ciphertext0;
		t2 = x1 ^ ciphertext1;
		t3 = x2 ^ ciphertext2;
		t4 = x3 ^ ciphertext3;

		bit32 result = 0;
		t1 = t1 & 0x08010812;
		t2 = t2 & 0xa00a2004;
		t3 = t3 & 0x00000010;
		t4 = t4 & 0x000a0000;
		for (int i = 0; i < 32; i++) {
			result ^= ((t1 >> i) & 0x1);
			result ^= ((t2 >> i) & 0x1);
			result ^= ((t3 >> i) & 0x1);
			result ^= ((t4 >> i) & 0x1);
		}
		if (result == 0) counter++;*/
		t1 = x0 ^ ciphertext0;
		t2 = x1 ^ ciphertext1;
		t3 = x2 ^ ciphertext2;
		t4 = x3 ^ ciphertext3;
		t0 = x0 & 0x12000200;
		t1 = x1 & 0x01980010;
		t2 = x2 & 0x00010000;
		t3 = x3 & 0x44900010;





		t0 ^= t0 >> 1;
		t0 ^= t0 >> 2;
		t0 = (t0 & 0x11111111U) * 0x11111111U;
		t0 = (t0 >> 28) & 1;

		t1 ^= t1 >> 1;
		t1 ^= t1 >> 2;
		t1 = (t1 & 0x11111111U) * 0x11111111U;
		t1 = (t1 >> 28) & 1;

		t2 ^= t2 >> 1;
		t2 ^= t2 >> 2;
		t2 = (t2 & 0x11111111U) * 0x11111111U;
		t2 = (t2 >> 28) & 1;

		t3 ^= t3 >> 1;
		t3 ^= t3 >> 2;
		t3 = (t3 & 0x11111111U) * 0x11111111U;
		t3 = (t3 >> 28) & 1;

		t0 = t0 ^ t1 ^ t2 ^ t3;

		if (t0 == 0) counter++;

		// Change plaintext
		plaintext0 = x0; plaintext1 = x1; plaintext2 = x2; plaintext3 = x3;
	}
	hit[blockIdx.x * blockDim.x + threadIdx.x] = counter;
}
__global__ void Serpent3round_PZWD24_partof6(bit32 p0, bit32 p1, bit32 p2, bit32 p3, bit64* hit, bit32* k_d) {
	__shared__ bit32  k[132];
	bit32 x0, x1, x2, x3, plaintext0 = p0, plaintext1 = p1, plaintext2 = p2, plaintext3 = p3 + blockIdx.x * blockDim.x + threadIdx.x;
	bit32 a, b, c, d;
	bit32 ciphertext0, ciphertext1, ciphertext2, ciphertext3;
	bit32 t0, t1, t2, t3, t4, t5, t6, t7, t8, t9, t10, t11, t12;
	bit32 round;
	bit64 counter = 0;
	if (threadIdx.x < 132) k[threadIdx.x] = k_d[threadIdx.x];
	__syncthreads();
	for (int i = 0; i < loop_d; i++) {
		x0 = plaintext0; x1 = plaintext1; x2 = plaintext2; x3 = plaintext3;
		// Round 5
		round = 5;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 6
		round = 6;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 7
		round = 7;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = b ^ c; t2 = c & t1; t3 = d ^ t2; t4 = a ^ t3; t5 = d | t1; t6 = t4 & t5; x1 = b ^ t6; t8 = t3 | x1; t9 = a & t4; x3 = t1 ^ t9; t11 = t4 ^ t8; t12 = x3 & t11; x2 = t3 ^ t12; x0 = (~t11) ^ (x3 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);

		// Give difference
		ciphertext0 = x0; ciphertext1 = x1; ciphertext2 = x2; ciphertext3 = x3;
		x0 = plaintext0 ^ 0x00000004;
		x1 = plaintext1 ^ 0x00000000;
		x2 = plaintext2 ^ 0x00000000;
		x3 = plaintext3 ^ 0x00000000;
		// 00000000000000000000000000000004

		// Round 5
		round = 5;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 6
		round = 6;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 7
		round = 7;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = b ^ c; t2 = c & t1; t3 = d ^ t2; t4 = a ^ t3; t5 = d | t1; t6 = t4 & t5; x1 = b ^ t6; t8 = t3 | x1; t9 = a & t4; x3 = t1 ^ t9; t11 = t4 ^ t8; t12 = x3 & t11; x2 = t3 ^ t12; x0 = (~t11) ^ (x3 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);

		// Check Difference

		x0 ^= ciphertext0;
		x1 ^= ciphertext1;
		x2 ^= ciphertext2;
		x3 ^= ciphertext3;

		t0 = x0 & 0x00010000;
		t1 = x1 & 0x00000800;
		t2 = x2 & 0x00000000;
		t3 = x3 & 0x00000800;
		// 00000800000000000000080000010000

		t0 ^= t0 >> 1;
		t0 ^= t0 >> 2;
		t0 = (t0 & 0x11111111U) * 0x11111111U;
		t0 = (t0 >> 28) & 1;

		t1 ^= t1 >> 1;
		t1 ^= t1 >> 2;
		t1 = (t1 & 0x11111111U) * 0x11111111U;
		t1 = (t1 >> 28) & 1;

		t2 ^= t2 >> 1;
		t2 ^= t2 >> 2;
		t2 = (t2 & 0x11111111U) * 0x11111111U;
		t2 = (t2 >> 28) & 1;

		t3 ^= t3 >> 1;
		t3 ^= t3 >> 2;
		t3 = (t3 & 0x11111111U) * 0x11111111U;
		t3 = (t3 >> 28) & 1;

		t0 = t0 ^ t1 ^ t2 ^ t3;

		if (t0 == 0) counter++;

		// Change plaintext
		plaintext0 = x0; plaintext1 = x1; plaintext2 = x2; plaintext3 = x3;
	}
	hit[blockIdx.x * blockDim.x + threadIdx.x] = counter;
}
__global__ void Serpent4round_PZWD24_partof6(bit32 p0, bit32 p1, bit32 p2, bit32 p3, bit64* hit, bit32* k_d) {
	__shared__ bit32  k[132];
	bit32 x0, x1, x2, x3, plaintext0 = p0, plaintext1 = p1, plaintext2 = p2, plaintext3 = p3 + blockIdx.x * blockDim.x + threadIdx.x;
	bit32 a, b, c, d;
	bit32 ciphertext0, ciphertext1, ciphertext2, ciphertext3;
	bit32 t0, t1, t2, t3, t4, t5, t6, t7, t8, t9, t10, t11, t12;
	bit32 round;
	bit64 counter = 0;
	if (threadIdx.x < 132) k[threadIdx.x] = k_d[threadIdx.x];
	__syncthreads();
	for (int i = 0; i < loop_d; i++) {
		x0 = plaintext0; x1 = plaintext1; x2 = plaintext2; x3 = plaintext3;
		// Round 5
		round = 5;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 6
		round = 6;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 7
		round = 7;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = b ^ c; t2 = c & t1; t3 = d ^ t2; t4 = a ^ t3; t5 = d | t1; t6 = t4 & t5; x1 = b ^ t6; t8 = t3 | x1; t9 = a & t4; x3 = t1 ^ t9; t11 = t4 ^ t8; t12 = x3 & t11; x2 = t3 ^ t12; x0 = (~t11) ^ (x3 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 8
		x0 ^= k[32];	x1 ^= k[33];	x2 ^= k[34];	x3 ^= k[35];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t3 = c ^ t1; t4 = b ^ t3; x3 = (a & d) ^ t4; t7 = a ^ (b & t1); x2 = t4 ^ (c | t7); t12 = x3 & (t3 ^ t7); x1 = (~t3) ^ t12; x0 = t12 ^ (~t7);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);

		// Give difference
		ciphertext0 = x0; ciphertext1 = x1; ciphertext2 = x2; ciphertext3 = x3;
		x0 = plaintext0 ^ 0x00000004;
		x1 = plaintext1 ^ 0x00000000;
		x2 = plaintext2 ^ 0x00000000;
		x3 = plaintext3 ^ 0x00000000;
		// 00000000000000000000000000000004

		// Round 5
		round = 5;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 6
		round = 6;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 7
		round = 7;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = b ^ c; t2 = c & t1; t3 = d ^ t2; t4 = a ^ t3; t5 = d | t1; t6 = t4 & t5; x1 = b ^ t6; t8 = t3 | x1; t9 = a & t4; x3 = t1 ^ t9; t11 = t4 ^ t8; t12 = x3 & t11; x2 = t3 ^ t12; x0 = (~t11) ^ (x3 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 8
		x0 ^= k[32];	x1 ^= k[33];	x2 ^= k[34];	x3 ^= k[35];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t3 = c ^ t1; t4 = b ^ t3; x3 = (a & d) ^ t4; t7 = a ^ (b & t1); x2 = t4 ^ (c | t7); t12 = x3 & (t3 ^ t7); x1 = (~t3) ^ t12; x0 = t12 ^ (~t7);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);

		// Check Difference
		x0 ^= ciphertext0;
		x1 ^= ciphertext1;
		x2 ^= ciphertext2;
		x3 ^= ciphertext3;

		t0 = x0 & 0x00010000;
		t1 = x1 & 0x00000800;
		t2 = x2 & 0x00000000;
		t3 = x3 & 0x00000800;
		// 00000800000000000000080000010000

		t0 ^= t0 >> 1;
		t0 ^= t0 >> 2;
		t0 = (t0 & 0x11111111U) * 0x11111111U;
		t0 = (t0 >> 28) & 1;

		t1 ^= t1 >> 1;
		t1 ^= t1 >> 2;
		t1 = (t1 & 0x11111111U) * 0x11111111U;
		t1 = (t1 >> 28) & 1;

		t2 ^= t2 >> 1;
		t2 ^= t2 >> 2;
		t2 = (t2 & 0x11111111U) * 0x11111111U;
		t2 = (t2 >> 28) & 1;

		t3 ^= t3 >> 1;
		t3 ^= t3 >> 2;
		t3 = (t3 & 0x11111111U) * 0x11111111U;
		t3 = (t3 >> 28) & 1;

		t0 = t0 ^ t1 ^ t2 ^ t3;

		if (t0 == 0) counter++;

		// Change plaintext
		plaintext0 = x0; plaintext1 = x1; plaintext2 = x2; plaintext3 = x3;
	}
	hit[blockIdx.x * blockDim.x + threadIdx.x] = counter;
}
__global__ void Serpent4round_PZWD24_first4of6(bit32 p0, bit32 p1, bit32 p2, bit32 p3, bit64* hit, bit32* k_d) {
	__shared__ bit32  k[132];
	bit32 x0, x1, x2, x3, plaintext0 = p0, plaintext1 = p1, plaintext2 = p2, plaintext3 = p3 + blockIdx.x * blockDim.x + threadIdx.x;
	bit32 a, b, c, d;
	bit32 ciphertext0, ciphertext1, ciphertext2, ciphertext3;
	bit32 t0, t1, t2, t3, t4, t5, t6, t7, t8, t9, t10, t11, t12;
	bit32 round;
	bit64 counter = 0;
	if (threadIdx.x < 132) k[threadIdx.x] = k_d[threadIdx.x];
	__syncthreads();
	for (int i = 0; i < loop_d; i++) {
		x0 = plaintext0; x1 = plaintext1; x2 = plaintext2; x3 = plaintext3;
		// Round 4
		x0 ^= k[16];	x1 ^= k[17];	x2 ^= k[18];	x3 ^= k[19];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 5
		round = 5;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 6
		round = 6;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 7
		round = 7;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = b ^ c; t2 = c & t1; t3 = d ^ t2; t4 = a ^ t3; t5 = d | t1; t6 = t4 & t5; x1 = b ^ t6; t8 = t3 | x1; t9 = a & t4; x3 = t1 ^ t9; t11 = t4 ^ t8; t12 = x3 & t11; x2 = t3 ^ t12; x0 = (~t11) ^ (x3 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);

		// Give difference
		ciphertext0 = x0; ciphertext1 = x1; ciphertext2 = x2; ciphertext3 = x3;
		x0 = plaintext0 ^ 0x20010000;
		x1 = plaintext1 ^ 0x00010000;
		x2 = plaintext2 ^ 0x00000000;
		x3 = plaintext3 ^ 0x20010000;
		//20010000000000000001000020010000
		

		// Round 4
		x0 ^= k[16];	x1 ^= k[17];	x2 ^= k[18];	x3 ^= k[19];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 5
		round = 5;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 6
		round = 6;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 7
		round = 7;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = b ^ c; t2 = c & t1; t3 = d ^ t2; t4 = a ^ t3; t5 = d | t1; t6 = t4 & t5; x1 = b ^ t6; t8 = t3 | x1; t9 = a & t4; x3 = t1 ^ t9; t11 = t4 ^ t8; t12 = x3 & t11; x2 = t3 ^ t12; x0 = (~t11) ^ (x3 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);


		x0 ^= ciphertext0;
		x1 ^= ciphertext1;
		x2 ^= ciphertext2;
		x3 ^= ciphertext3;

		t0 = x0 & 0x00010000;
		t1 = x1 & 0x00000800;
		t2 = x2 & 0x00000000;
		t3 = x3 & 0x00000800;
		// 00000800000000000000080000010000

		t0 ^= t0 >> 1;
		t0 ^= t0 >> 2;
		t0 = (t0 & 0x11111111U) * 0x11111111U;
		t0 = (t0 >> 28) & 1;

		t1 ^= t1 >> 1;
		t1 ^= t1 >> 2;
		t1 = (t1 & 0x11111111U) * 0x11111111U;
		t1 = (t1 >> 28) & 1;

		t2 ^= t2 >> 1;
		t2 ^= t2 >> 2;
		t2 = (t2 & 0x11111111U) * 0x11111111U;
		t2 = (t2 >> 28) & 1;

		t3 ^= t3 >> 1;
		t3 ^= t3 >> 2;
		t3 = (t3 & 0x11111111U) * 0x11111111U;
		t3 = (t3 >> 28) & 1;

		t0 = t0 ^ t1 ^ t2 ^ t3;

		if (t0 == 0) counter++;

		// Change plaintext
		plaintext0 = x0; plaintext1 = x1; plaintext2 = x2; plaintext3 = x3;
	}
	hit[blockIdx.x * blockDim.x + threadIdx.x] = counter;
}
__global__ void Serpent5round_PZWD24(bit32 p0, bit32 p1, bit32 p2, bit32 p3, bit64* hit, bit32* k_d) {
	__shared__ bit32  k[132];
	bit32 x0, x1, x2, x3, plaintext0 = p0, plaintext1 = p1, plaintext2 = p2, plaintext3 = p3 + blockIdx.x * blockDim.x + threadIdx.x;
	bit32 a, b, c, d;
	bit32 ciphertext0, ciphertext1, ciphertext2, ciphertext3;
	bit32 t0, t1, t2, t3, t4, t5, t6, t7, t8, t9, t10, t11, t12;
	bit32 round;
	bit64 counter = 0;
	if (threadIdx.x < 132) k[threadIdx.x] = k_d[threadIdx.x];
	__syncthreads();
	for (int i = 0; i < loop_d; i++) {
		x0 = plaintext0; x1 = plaintext1; x2 = plaintext2; x3 = plaintext3;
		// Round 5
		round = 5;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 6
		round = 6;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 7
		round = 7;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = b ^ c; t2 = c & t1; t3 = d ^ t2; t4 = a ^ t3; t5 = d | t1; t6 = t4 & t5; x1 = b ^ t6; t8 = t3 | x1; t9 = a & t4; x3 = t1 ^ t9; t11 = t4 ^ t8; t12 = x3 & t11; x2 = t3 ^ t12; x0 = (~t11) ^ (x3 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 8
		x0 ^= k[32];	x1 ^= k[33];	x2 ^= k[34];	x3 ^= k[35];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t3 = c ^ t1; t4 = b ^ t3; x3 = (a & d) ^ t4; t7 = a ^ (b & t1); x2 = t4 ^ (c | t7); t12 = x3 & (t3 ^ t7); x1 = (~t3) ^ t12; x0 = t12 ^ (~t7);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 9
		x0 ^= k[36];	x1 ^= k[37];	x2 ^= k[38];	x3 ^= k[39];
		a = x0; b = x1; c = x2; d = x3;
		t2 = b ^ (~a);		t5 = c ^ (a | t2);		x2 = d ^ t5;		t7 = b ^ (d | t2);		t8 = t2 ^ x2;		x3 = t8 ^ (t5 & t7); 		t11 = t5 ^ t7; 		x1 = x3 ^ t11; 		x0 = t5 ^ (t8 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);

		// Give difference
		ciphertext0 = x0; ciphertext1 = x1; ciphertext2 = x2; ciphertext3 = x3;
		x0 = plaintext0 ^ 0x00000004;
		x1 = plaintext1 ^ 0x00000000;
		x2 = plaintext2 ^ 0x80000000;
		x3 = plaintext3 ^ 0x80000000;
		// 80000000800000000000000000000004
		
		// Round 5
		round = 5;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 6
		round = 6;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 7
		round = 7;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = b ^ c; t2 = c & t1; t3 = d ^ t2; t4 = a ^ t3; t5 = d | t1; t6 = t4 & t5; x1 = b ^ t6; t8 = t3 | x1; t9 = a & t4; x3 = t1 ^ t9; t11 = t4 ^ t8; t12 = x3 & t11; x2 = t3 ^ t12; x0 = (~t11) ^ (x3 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 8
		x0 ^= k[32];	x1 ^= k[33];	x2 ^= k[34];	x3 ^= k[35];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t3 = c ^ t1; t4 = b ^ t3; x3 = (a & d) ^ t4; t7 = a ^ (b & t1); x2 = t4 ^ (c | t7); t12 = x3 & (t3 ^ t7); x1 = (~t3) ^ t12; x0 = t12 ^ (~t7);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 9
		x0 ^= k[36];	x1 ^= k[37];	x2 ^= k[38];	x3 ^= k[39];
		a = x0; b = x1; c = x2; d = x3;
		t2 = b ^ (~a);		t5 = c ^ (a | t2);		x2 = d ^ t5;		t7 = b ^ (d | t2);		t8 = t2 ^ x2;		x3 = t8 ^ (t5 & t7); 		t11 = t5 ^ t7; 		x1 = x3 ^ t11; 		x0 = t5 ^ (t8 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);

		// Check Difference
/*		t1 = x0 ^ ciphertext0;
		t2 = x1 ^ ciphertext1;
		t3 = x2 ^ ciphertext2;
		t4 = x3 ^ ciphertext3;

		bit32 result = 0;
		t1 = t1 & 0x08010812;
		t2 = t2 & 0xa00a2004;
		t3 = t3 & 0x00000010;
		t4 = t4 & 0x000a0000;
		for (int i = 0; i < 32; i++) {
			result ^= ((t1 >> i) & 0x1);
			result ^= ((t2 >> i) & 0x1);
			result ^= ((t3 >> i) & 0x1);
			result ^= ((t4 >> i) & 0x1);
		}
		if (result == 0) counter++;*/

		/*		t0 = x0 & 0x44900010;
				t1 = x1 & 0x00010000;
				t2 = x2 & 0x01980010;
				t3 = x3 & 0x12000200;*/
		x0 ^= ciphertext0;
		x1 ^= ciphertext1;
		x2 ^= ciphertext2;
		x3 ^= ciphertext3;

		t0 = x0 & 0x08404200;
		t1 = x1 & 0x00420634;
		t2 = x2 & 0x84000000;
		t3 = x3 & 0x00420000;

		//00420000840000000042063408404200
		

		t0 ^= t0 >> 1;
		t0 ^= t0 >> 2;
		t0 = (t0 & 0x11111111U) * 0x11111111U;
		t0 = (t0 >> 28) & 1;

		t1 ^= t1 >> 1;
		t1 ^= t1 >> 2;
		t1 = (t1 & 0x11111111U) * 0x11111111U;
		t1 = (t1 >> 28) & 1;

		t2 ^= t2 >> 1;
		t2 ^= t2 >> 2;
		t2 = (t2 & 0x11111111U) * 0x11111111U;
		t2 = (t2 >> 28) & 1;

		t3 ^= t3 >> 1;
		t3 ^= t3 >> 2;
		t3 = (t3 & 0x11111111U) * 0x11111111U;
		t3 = (t3 >> 28) & 1;

		t0 = t0 ^ t1 ^ t2 ^ t3;

		if (t0 == 0) counter++;

		// Change plaintext
		plaintext0 = x0; plaintext1 = x1; plaintext2 = x2; plaintext3 = x3;
	}
	hit[blockIdx.x * blockDim.x + threadIdx.x] = counter;
}
__global__ void Serpent5round_PZWD24_second(bit32 p0, bit32 p1, bit32 p2, bit32 p3, bit64* hit, bit32* k_d) {
	__shared__ bit32  k[132];
	bit32 x0, x1, x2, x3, plaintext0 = p0, plaintext1 = p1, plaintext2 = p2, plaintext3 = p3 + blockIdx.x * blockDim.x + threadIdx.x;
	bit32 a, b, c, d;
	bit32 ciphertext0, ciphertext1, ciphertext2, ciphertext3;
	bit32 t0, t1, t2, t3, t4, t5, t6, t7, t8, t9, t10, t11, t12;
	bit32 round;
	bit64 counter = 0;
	if (threadIdx.x < 132) k[threadIdx.x] = k_d[threadIdx.x];
	__syncthreads();
	for (int i = 0; i < loop_d; i++) {
		x0 = plaintext0; x1 = plaintext1; x2 = plaintext2; x3 = plaintext3;
		// Round 5
		round = 5;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 6
		round = 6;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 7
		round = 7;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = b ^ c; t2 = c & t1; t3 = d ^ t2; t4 = a ^ t3; t5 = d | t1; t6 = t4 & t5; x1 = b ^ t6; t8 = t3 | x1; t9 = a & t4; x3 = t1 ^ t9; t11 = t4 ^ t8; t12 = x3 & t11; x2 = t3 ^ t12; x0 = (~t11) ^ (x3 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 8
		x0 ^= k[32];	x1 ^= k[33];	x2 ^= k[34];	x3 ^= k[35];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t3 = c ^ t1; t4 = b ^ t3; x3 = (a & d) ^ t4; t7 = a ^ (b & t1); x2 = t4 ^ (c | t7); t12 = x3 & (t3 ^ t7); x1 = (~t3) ^ t12; x0 = t12 ^ (~t7);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 9
		x0 ^= k[36];	x1 ^= k[37];	x2 ^= k[38];	x3 ^= k[39];
		a = x0; b = x1; c = x2; d = x3;
		t2 = b ^ (~a);		t5 = c ^ (a | t2);		x2 = d ^ t5;		t7 = b ^ (d | t2);		t8 = t2 ^ x2;		x3 = t8 ^ (t5 & t7); 		t11 = t5 ^ t7; 		x1 = x3 ^ t11; 		x0 = t5 ^ (t8 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);

		// Give difference
		ciphertext0 = x0; ciphertext1 = x1; ciphertext2 = x2; ciphertext3 = x3;
		x0 = plaintext0 ^ 0x00000008;
		x1 = plaintext1 ^ 0x00000000;
		x2 = plaintext2 ^ 0x00000001;
		x3 = plaintext3 ^ 0x00000001;
		//00000001000000010000000000000008


		// Round 5
		round = 5;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 6
		round = 6;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 7
		round = 7;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = b ^ c; t2 = c & t1; t3 = d ^ t2; t4 = a ^ t3; t5 = d | t1; t6 = t4 & t5; x1 = b ^ t6; t8 = t3 | x1; t9 = a & t4; x3 = t1 ^ t9; t11 = t4 ^ t8; t12 = x3 & t11; x2 = t3 ^ t12; x0 = (~t11) ^ (x3 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 8
		x0 ^= k[32];	x1 ^= k[33];	x2 ^= k[34];	x3 ^= k[35];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t3 = c ^ t1; t4 = b ^ t3; x3 = (a & d) ^ t4; t7 = a ^ (b & t1); x2 = t4 ^ (c | t7); t12 = x3 & (t3 ^ t7); x1 = (~t3) ^ t12; x0 = t12 ^ (~t7);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 9
		x0 ^= k[36];	x1 ^= k[37];	x2 ^= k[38];	x3 ^= k[39];
		a = x0; b = x1; c = x2; d = x3;
		t2 = b ^ (~a);		t5 = c ^ (a | t2);		x2 = d ^ t5;		t7 = b ^ (d | t2);		t8 = t2 ^ x2;		x3 = t8 ^ (t5 & t7); 		t11 = t5 ^ t7; 		x1 = x3 ^ t11; 		x0 = t5 ^ (t8 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);

		// Check Difference
		x0 ^= ciphertext0;
		x1 ^= ciphertext1;
		x2 ^= ciphertext2;
		x3 ^= ciphertext3;

		t0 = x0 & 0x10808400;
		t1 = x1 & 0x00840c68;
		t2 = x2 & 0x08000001;
		t3 = x3 & 0x00840000;

		// a8000008820000186a0000014c000008   Reported on the Crypto paper
		// 008400000800000100840c6810808400   My 4 bit rotated version

		t0 ^= t0 >> 1;
		t0 ^= t0 >> 2;
		t0 = (t0 & 0x11111111U) * 0x11111111U;
		t0 = (t0 >> 28) & 1;

		t1 ^= t1 >> 1;
		t1 ^= t1 >> 2;
		t1 = (t1 & 0x11111111U) * 0x11111111U;
		t1 = (t1 >> 28) & 1;

		t2 ^= t2 >> 1;
		t2 ^= t2 >> 2;
		t2 = (t2 & 0x11111111U) * 0x11111111U;
		t2 = (t2 >> 28) & 1;

		t3 ^= t3 >> 1;
		t3 ^= t3 >> 2;
		t3 = (t3 & 0x11111111U) * 0x11111111U;
		t3 = (t3 >> 28) & 1;

		t0 = t0 ^ t1 ^ t2 ^ t3;

		if (t0 == 0) counter++;

		// Change plaintext
		plaintext0 = x0; plaintext1 = x1; plaintext2 = x2; plaintext3 = x3;
	}
	hit[blockIdx.x * blockDim.x + threadIdx.x] = counter;
}
__global__ void Serpent5round_PZWD24_partof6(bit32 p0, bit32 p1, bit32 p2, bit32 p3, bit64* hit, bit32* k_d) {
	__shared__ bit32  k[132];
	bit32 x0, x1, x2, x3, plaintext0 = p0, plaintext1 = p1, plaintext2 = p2, plaintext3 = p3 + blockIdx.x * blockDim.x + threadIdx.x;
	bit32 a, b, c, d;
	bit32 ciphertext0, ciphertext1, ciphertext2, ciphertext3;
	bit32 t0, t1, t2, t3, t4, t5, t6, t7, t8, t9, t10, t11, t12;
	bit32 round;
	bit64 counter = 0;
	if (threadIdx.x < 132) k[threadIdx.x] = k_d[threadIdx.x];
	__syncthreads();
	for (int i = 0; i < loop_d; i++) {
		x0 = plaintext0; x1 = plaintext1; x2 = plaintext2; x3 = plaintext3;
		// Round 5
		round = 5;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 6
		round = 6;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 7
		round = 7;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = b ^ c; t2 = c & t1; t3 = d ^ t2; t4 = a ^ t3; t5 = d | t1; t6 = t4 & t5; x1 = b ^ t6; t8 = t3 | x1; t9 = a & t4; x3 = t1 ^ t9; t11 = t4 ^ t8; t12 = x3 & t11; x2 = t3 ^ t12; x0 = (~t11) ^ (x3 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 8
		x0 ^= k[32];	x1 ^= k[33];	x2 ^= k[34];	x3 ^= k[35];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t3 = c ^ t1; t4 = b ^ t3; x3 = (a & d) ^ t4; t7 = a ^ (b & t1); x2 = t4 ^ (c | t7); t12 = x3 & (t3 ^ t7); x1 = (~t3) ^ t12; x0 = t12 ^ (~t7);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 9
		x0 ^= k[36];	x1 ^= k[37];	x2 ^= k[38];	x3 ^= k[39];
		a = x0; b = x1; c = x2; d = x3;
		t2 = b ^ (~a);		t5 = c ^ (a | t2);		x2 = d ^ t5;		t7 = b ^ (d | t2);		t8 = t2 ^ x2;		x3 = t8 ^ (t5 & t7); 		t11 = t5 ^ t7; 		x1 = x3 ^ t11; 		x0 = t5 ^ (t8 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);

		// Give difference
		ciphertext0 = x0; ciphertext1 = x1; ciphertext2 = x2; ciphertext3 = x3;
		x0 = plaintext0 ^ 0x00000004;
		x1 = plaintext1 ^ 0x00000000;
		x2 = plaintext2 ^ 0x00000000;
		x3 = plaintext3 ^ 0x00000000;
		//00000000000000000000000000000004


		// Round 5
		round = 5;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 6
		round = 6;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 7
		round = 7;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = b ^ c; t2 = c & t1; t3 = d ^ t2; t4 = a ^ t3; t5 = d | t1; t6 = t4 & t5; x1 = b ^ t6; t8 = t3 | x1; t9 = a & t4; x3 = t1 ^ t9; t11 = t4 ^ t8; t12 = x3 & t11; x2 = t3 ^ t12; x0 = (~t11) ^ (x3 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 8
		x0 ^= k[32];	x1 ^= k[33];	x2 ^= k[34];	x3 ^= k[35];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t3 = c ^ t1; t4 = b ^ t3; x3 = (a & d) ^ t4; t7 = a ^ (b & t1); x2 = t4 ^ (c | t7); t12 = x3 & (t3 ^ t7); x1 = (~t3) ^ t12; x0 = t12 ^ (~t7);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 9
		x0 ^= k[36];	x1 ^= k[37];	x2 ^= k[38];	x3 ^= k[39];
		a = x0; b = x1; c = x2; d = x3;
		t2 = b ^ (~a);		t5 = c ^ (a | t2);		x2 = d ^ t5;		t7 = b ^ (d | t2);		t8 = t2 ^ x2;		x3 = t8 ^ (t5 & t7); 		t11 = t5 ^ t7; 		x1 = x3 ^ t11; 		x0 = t5 ^ (t8 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);

		// Check Difference
		x0 ^= ciphertext0;
		x1 ^= ciphertext1;
		x2 ^= ciphertext2;
		x3 ^= ciphertext3;

		t0 = x0 & 0x84808006;
		t1 = x1 & 0x4c040336;
		t2 = x2 & 0x00002008;
		t3 = x3 & 0x8724021e;
		// 8724021e000020084c04033684808006


		t0 ^= t0 >> 1;
		t0 ^= t0 >> 2;
		t0 = (t0 & 0x11111111U) * 0x11111111U;
		t0 = (t0 >> 28) & 1;

		t1 ^= t1 >> 1;
		t1 ^= t1 >> 2;
		t1 = (t1 & 0x11111111U) * 0x11111111U;
		t1 = (t1 >> 28) & 1;

		t2 ^= t2 >> 1;
		t2 ^= t2 >> 2;
		t2 = (t2 & 0x11111111U) * 0x11111111U;
		t2 = (t2 >> 28) & 1;

		t3 ^= t3 >> 1;
		t3 ^= t3 >> 2;
		t3 = (t3 & 0x11111111U) * 0x11111111U;
		t3 = (t3 >> 28) & 1;

		t0 = t0 ^ t1 ^ t2 ^ t3;

		if (t0 == 0) counter++;

		// Change plaintext
		plaintext0 = x0; plaintext1 = x1; plaintext2 = x2; plaintext3 = x3;
	}
	hit[blockIdx.x * blockDim.x + threadIdx.x] = counter;
}
__global__ void Serpent6round_PZWD24(bit32 p0, bit32 p1, bit32 p2, bit32 p3, bit64* hit, bit32* k_d) {
	__shared__ bit32  k[132];
	bit32 x0, x1, x2, x3, plaintext0 = p0, plaintext1 = p1, plaintext2 = p2, plaintext3 = p3 + blockIdx.x * blockDim.x + threadIdx.x;
	bit32 a, b, c, d;
	bit32 ciphertext0, ciphertext1, ciphertext2, ciphertext3;
	bit32 t0, t1, t2, t3, t4, t5, t6, t7, t8, t9, t10, t11, t12;
	bit32 round;
	bit64 counter = 0;
	if (threadIdx.x < 132) k[threadIdx.x] = k_d[threadIdx.x];
	__syncthreads();
	for (int i = 0; i < loop_d; i++) {
		x0 = plaintext0; x1 = plaintext1; x2 = plaintext2; x3 = plaintext3;
		// Round 4
		x0 ^= k[16];	x1 ^= k[17];	x2 ^= k[18];	x3 ^= k[19];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 5
		round = 5;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 6
		round = 6;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 7
		round = 7;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = b ^ c; t2 = c & t1; t3 = d ^ t2; t4 = a ^ t3; t5 = d | t1; t6 = t4 & t5; x1 = b ^ t6; t8 = t3 | x1; t9 = a & t4; x3 = t1 ^ t9; t11 = t4 ^ t8; t12 = x3 & t11; x2 = t3 ^ t12; x0 = (~t11) ^ (x3 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 8
		x0 ^= k[32];	x1 ^= k[33];	x2 ^= k[34];	x3 ^= k[35];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t3 = c ^ t1; t4 = b ^ t3; x3 = (a & d) ^ t4; t7 = a ^ (b & t1); x2 = t4 ^ (c | t7); t12 = x3 & (t3 ^ t7); x1 = (~t3) ^ t12; x0 = t12 ^ (~t7);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 9
		x0 ^= k[36];	x1 ^= k[37];	x2 ^= k[38];	x3 ^= k[39];
		a = x0; b = x1; c = x2; d = x3;
		t2 = b ^ (~a);		t5 = c ^ (a | t2);		x2 = d ^ t5;		t7 = b ^ (d | t2);		t8 = t2 ^ x2;		x3 = t8 ^ (t5 & t7); 		t11 = t5 ^ t7; 		x1 = x3 ^ t11; 		x0 = t5 ^ (t8 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);

		// Give difference
		ciphertext0 = x0; ciphertext1 = x1; ciphertext2 = x2; ciphertext3 = x3;
		x0 = plaintext0 ^ 0x20010000;
		x1 = plaintext1 ^ 0x00010000;
		x2 = plaintext2 ^ 0x00000000;
		x3 = plaintext3 ^ 0x20010000;
		//20010000000000000001000020010000
		
		// Round 4
		x0 ^= k[16];	x1 ^= k[17];	x2 ^= k[18];	x3 ^= k[19];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 5
		round = 5;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 6
		round = 6;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 7
		round = 7;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = b ^ c; t2 = c & t1; t3 = d ^ t2; t4 = a ^ t3; t5 = d | t1; t6 = t4 & t5; x1 = b ^ t6; t8 = t3 | x1; t9 = a & t4; x3 = t1 ^ t9; t11 = t4 ^ t8; t12 = x3 & t11; x2 = t3 ^ t12; x0 = (~t11) ^ (x3 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 8
		x0 ^= k[32];	x1 ^= k[33];	x2 ^= k[34];	x3 ^= k[35];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t3 = c ^ t1; t4 = b ^ t3; x3 = (a & d) ^ t4; t7 = a ^ (b & t1); x2 = t4 ^ (c | t7); t12 = x3 & (t3 ^ t7); x1 = (~t3) ^ t12; x0 = t12 ^ (~t7);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 9
		x0 ^= k[36];	x1 ^= k[37];	x2 ^= k[38];	x3 ^= k[39];
		a = x0; b = x1; c = x2; d = x3;
		t2 = b ^ (~a);		t5 = c ^ (a | t2);		x2 = d ^ t5;		t7 = b ^ (d | t2);		t8 = t2 ^ x2;		x3 = t8 ^ (t5 & t7); 		t11 = t5 ^ t7; 		x1 = x3 ^ t11; 		x0 = t5 ^ (t8 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);


		x0 ^= ciphertext0;
		x1 ^= ciphertext1;
		x2 ^= ciphertext2;
		x3 ^= ciphertext3;

		t0 = x0 & 0x01084802;
		t1 = x1 & 0x302842c0;
		t2 = x2 & 0x10040000;
		t3 = x3 & 0x00084200;
		// 8724021e000020084c04033684808006
		// corrected: 002600010020b4000b0010a022000010 => 0008420010040000302842c001084802

		t0 ^= t0 >> 1;
		t0 ^= t0 >> 2;
		t0 = (t0 & 0x11111111U) * 0x11111111U;
		t0 = (t0 >> 28) & 1;

		t1 ^= t1 >> 1;
		t1 ^= t1 >> 2;
		t1 = (t1 & 0x11111111U) * 0x11111111U;
		t1 = (t1 >> 28) & 1;

		t2 ^= t2 >> 1;
		t2 ^= t2 >> 2;
		t2 = (t2 & 0x11111111U) * 0x11111111U;
		t2 = (t2 >> 28) & 1;

		t3 ^= t3 >> 1;
		t3 ^= t3 >> 2;
		t3 = (t3 & 0x11111111U) * 0x11111111U;
		t3 = (t3 >> 28) & 1;

		t0 = t0 ^ t1 ^ t2 ^ t3;

		if (t0 == 0) counter++;

		// Change plaintext
		plaintext0 = x0; plaintext1 = x1; plaintext2 = x2; plaintext3 = x3;
	}
	hit[blockIdx.x * blockDim.x + threadIdx.x] = counter;
}
__global__ void Serpent6round_PZWD24_final(bit32 p0, bit32 p1, bit32 p2, bit32 p3, bit64* hit, bit32* k_d) {
	__shared__ bit32  k[132];
	bit32 x0, x1, x2, x3, plaintext0 = p0, plaintext1 = p1, plaintext2 = p2, plaintext3 = p3 + blockIdx.x * blockDim.x + threadIdx.x;
	bit32 a, b, c, d;
	bit32 ciphertext0, ciphertext1, ciphertext2, ciphertext3;
	bit32 t0, t1, t2, t3, t4, t5, t6, t7, t8, t9, t10, t11, t12;
	bit32 round;
	bit64 counter = 0;
	if (threadIdx.x < 132) k[threadIdx.x] = k_d[threadIdx.x];
	__syncthreads();
	for (int i = 0; i < loop_d; i++) {
		x0 = plaintext0; x1 = plaintext1; x2 = plaintext2; x3 = plaintext3;
		// Round 4
		x0 ^= k[16];	x1 ^= k[17];	x2 ^= k[18];	x3 ^= k[19];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 5
		round = 5;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 6
		round = 6;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 7
		round = 7;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = b ^ c; t2 = c & t1; t3 = d ^ t2; t4 = a ^ t3; t5 = d | t1; t6 = t4 & t5; x1 = b ^ t6; t8 = t3 | x1; t9 = a & t4; x3 = t1 ^ t9; t11 = t4 ^ t8; t12 = x3 & t11; x2 = t3 ^ t12; x0 = (~t11) ^ (x3 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 8
		x0 ^= k[32];	x1 ^= k[33];	x2 ^= k[34];	x3 ^= k[35];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t3 = c ^ t1; t4 = b ^ t3; x3 = (a & d) ^ t4; t7 = a ^ (b & t1); x2 = t4 ^ (c | t7); t12 = x3 & (t3 ^ t7); x1 = (~t3) ^ t12; x0 = t12 ^ (~t7);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 9
		x0 ^= k[36];	x1 ^= k[37];	x2 ^= k[38];	x3 ^= k[39];
		a = x0; b = x1; c = x2; d = x3;
		t2 = b ^ (~a);		t5 = c ^ (a | t2);		x2 = d ^ t5;		t7 = b ^ (d | t2);		t8 = t2 ^ x2;		x3 = t8 ^ (t5 & t7); 		t11 = t5 ^ t7; 		x1 = x3 ^ t11; 		x0 = t5 ^ (t8 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);

		// Give difference
		ciphertext0 = x0; ciphertext1 = x1; ciphertext2 = x2; ciphertext3 = x3;
		x0 = plaintext0 ^ 0x20010000;
		x1 = plaintext1 ^ 0x00010000;
		x2 = plaintext2 ^ 0x00000000;
		x3 = plaintext3 ^ 0x20010000;
		//20010000000000000001000020010000

		// Round 4
		x0 ^= k[16];	x1 ^= k[17];	x2 ^= k[18];	x3 ^= k[19];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 5
		round = 5;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 6
		round = 6;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 7
		round = 7;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = b ^ c; t2 = c & t1; t3 = d ^ t2; t4 = a ^ t3; t5 = d | t1; t6 = t4 & t5; x1 = b ^ t6; t8 = t3 | x1; t9 = a & t4; x3 = t1 ^ t9; t11 = t4 ^ t8; t12 = x3 & t11; x2 = t3 ^ t12; x0 = (~t11) ^ (x3 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 8
		x0 ^= k[32];	x1 ^= k[33];	x2 ^= k[34];	x3 ^= k[35];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t3 = c ^ t1; t4 = b ^ t3; x3 = (a & d) ^ t4; t7 = a ^ (b & t1); x2 = t4 ^ (c | t7); t12 = x3 & (t3 ^ t7); x1 = (~t3) ^ t12; x0 = t12 ^ (~t7);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 9
		x0 ^= k[36];	x1 ^= k[37];	x2 ^= k[38];	x3 ^= k[39];
		a = x0; b = x1; c = x2; d = x3;
		t2 = b ^ (~a);		t5 = c ^ (a | t2);		x2 = d ^ t5;		t7 = b ^ (d | t2);		t8 = t2 ^ x2;		x3 = t8 ^ (t5 & t7); 		t11 = t5 ^ t7; 		x1 = x3 ^ t11; 		x0 = t5 ^ (t8 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);


		x0 ^= ciphertext0;
		x1 ^= ciphertext1;
		x2 ^= ciphertext2;
		x3 ^= ciphertext3;

		t0 = x0 & 0x20108480;
		t1 = x1 & 0x0302842c;
		t2 = x2 & 0x01004000;
		t3 = x3 & 0x00008420;
		// 
		// corrected: 0010002600010020b4000b0010a02200 => 00008420010040000302842c20108480

		t0 ^= t0 >> 1;
		t0 ^= t0 >> 2;
		t0 = (t0 & 0x11111111U) * 0x11111111U;
		t0 = (t0 >> 28) & 1;

		t1 ^= t1 >> 1;
		t1 ^= t1 >> 2;
		t1 = (t1 & 0x11111111U) * 0x11111111U;
		t1 = (t1 >> 28) & 1;

		t2 ^= t2 >> 1;
		t2 ^= t2 >> 2;
		t2 = (t2 & 0x11111111U) * 0x11111111U;
		t2 = (t2 >> 28) & 1;

		t3 ^= t3 >> 1;
		t3 ^= t3 >> 2;
		t3 = (t3 & 0x11111111U) * 0x11111111U;
		t3 = (t3 >> 28) & 1;

		t0 = t0 ^ t1 ^ t2 ^ t3;

		if (t0 == 0) counter++;

		// Change plaintext
		plaintext0 = x0; plaintext1 = x1; plaintext2 = x2; plaintext3 = x3;
	}
	hit[blockIdx.x * blockDim.x + threadIdx.x] = counter;
}
__global__ void Serpent6round_PZWD24_second(bit32 p0, bit32 p1, bit32 p2, bit32 p3, bit64* hit, bit32* k_d) {
	__shared__ bit32  k[132];
	bit32 x0, x1, x2, x3, plaintext0 = p0, plaintext1 = p1, plaintext2 = p2, plaintext3 = p3 + blockIdx.x * blockDim.x + threadIdx.x;
	bit32 a, b, c, d;
	bit32 ciphertext0, ciphertext1, ciphertext2, ciphertext3;
	bit32 t0, t1, t2, t3, t4, t5, t6, t7, t8, t9, t10, t11, t12;
	bit32 round;
	bit64 counter = 0;
	if (threadIdx.x < 132) k[threadIdx.x] = k_d[threadIdx.x];
	__syncthreads();
	for (int i = 0; i < loop_d; i++) {
		x0 = plaintext0; x1 = plaintext1; x2 = plaintext2; x3 = plaintext3;
		// Round 4
		x0 ^= k[16];	x1 ^= k[17];	x2 ^= k[18];	x3 ^= k[19];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 5
		round = 5;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 6
		round = 6;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 7
		round = 7;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = b ^ c; t2 = c & t1; t3 = d ^ t2; t4 = a ^ t3; t5 = d | t1; t6 = t4 & t5; x1 = b ^ t6; t8 = t3 | x1; t9 = a & t4; x3 = t1 ^ t9; t11 = t4 ^ t8; t12 = x3 & t11; x2 = t3 ^ t12; x0 = (~t11) ^ (x3 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 8
		x0 ^= k[32];	x1 ^= k[33];	x2 ^= k[34];	x3 ^= k[35];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t3 = c ^ t1; t4 = b ^ t3; x3 = (a & d) ^ t4; t7 = a ^ (b & t1); x2 = t4 ^ (c | t7); t12 = x3 & (t3 ^ t7); x1 = (~t3) ^ t12; x0 = t12 ^ (~t7);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 9
		x0 ^= k[36];	x1 ^= k[37];	x2 ^= k[38];	x3 ^= k[39];
		a = x0; b = x1; c = x2; d = x3;
		t2 = b ^ (~a);		t5 = c ^ (a | t2);		x2 = d ^ t5;		t7 = b ^ (d | t2);		t8 = t2 ^ x2;		x3 = t8 ^ (t5 & t7); 		t11 = t5 ^ t7; 		x1 = x3 ^ t11; 		x0 = t5 ^ (t8 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);

		// Give difference
		ciphertext0 = x0; ciphertext1 = x1; ciphertext2 = x2; ciphertext3 = x3;
		x0 = plaintext0 ^ 0x40020000;
		x1 = plaintext1 ^ 0x00020000;
		x2 = plaintext2 ^ 0x00000000;
		x3 = plaintext3 ^ 0x40020000;
		// 40020000000000000002000040020000

		// Round 4
		x0 ^= k[16];	x1 ^= k[17];	x2 ^= k[18];	x3 ^= k[19];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 5
		round = 5;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 6
		round = 6;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 7
		round = 7;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = b ^ c; t2 = c & t1; t3 = d ^ t2; t4 = a ^ t3; t5 = d | t1; t6 = t4 & t5; x1 = b ^ t6; t8 = t3 | x1; t9 = a & t4; x3 = t1 ^ t9; t11 = t4 ^ t8; t12 = x3 & t11; x2 = t3 ^ t12; x0 = (~t11) ^ (x3 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 8
		x0 ^= k[32];	x1 ^= k[33];	x2 ^= k[34];	x3 ^= k[35];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t3 = c ^ t1; t4 = b ^ t3; x3 = (a & d) ^ t4; t7 = a ^ (b & t1); x2 = t4 ^ (c | t7); t12 = x3 & (t3 ^ t7); x1 = (~t3) ^ t12; x0 = t12 ^ (~t7);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 9
		x0 ^= k[36];	x1 ^= k[37];	x2 ^= k[38];	x3 ^= k[39];
		a = x0; b = x1; c = x2; d = x3;
		t2 = b ^ (~a);		t5 = c ^ (a | t2);		x2 = d ^ t5;		t7 = b ^ (d | t2);		t8 = t2 ^ x2;		x3 = t8 ^ (t5 & t7); 		t11 = t5 ^ t7; 		x1 = x3 ^ t11; 		x0 = t5 ^ (t8 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);

	
		x0 ^= ciphertext0;
		x1 ^= ciphertext1;
		x2 ^= ciphertext2;
		x3 ^= ciphertext3;

		t0 = x0 & 0x58181200;
		t1 = x1 & 0xc948142c;
		t2 = x2 & 0x00008020;
		t3 = x3 & 0x42481008;
		//4248100800008020c948142c58181200


		t0 ^= t0 >> 1;
		t0 ^= t0 >> 2;
		t0 = (t0 & 0x11111111U) * 0x11111111U;
		t0 = (t0 >> 28) & 1;

		t1 ^= t1 >> 1;
		t1 ^= t1 >> 2;
		t1 = (t1 & 0x11111111U) * 0x11111111U;
		t1 = (t1 >> 28) & 1;

		t2 ^= t2 >> 1;
		t2 ^= t2 >> 2;
		t2 = (t2 & 0x11111111U) * 0x11111111U;
		t2 = (t2 >> 28) & 1;

		t3 ^= t3 >> 1;
		t3 ^= t3 >> 2;
		t3 = (t3 & 0x11111111U) * 0x11111111U;
		t3 = (t3 >> 28) & 1;

		t0 = t0 ^ t1 ^ t2 ^ t3;

		if (t0 == 0) counter++;

		// Change plaintext
		plaintext0 = x0; plaintext1 = x1; plaintext2 = x2; plaintext3 = x3;
	}
	hit[blockIdx.x * blockDim.x + threadIdx.x] = counter;
}

__global__ void Serpent6round_PZWD24_sanity(bit32 p0, bit32 p1, bit32 p2, bit32 p3, bit64* hit, bit32* k_d) {
	__shared__ bit32  k[132];
	bit32 x0, x1, x2, x3, plaintext0 = p0, plaintext1 = p1, plaintext2 = p2, plaintext3 = p3 + blockIdx.x * blockDim.x + threadIdx.x;
	bit32 a, b, c, d;
	bit32 ciphertext0, ciphertext1, ciphertext2, ciphertext3;
	bit32 t0, t1, t2, t3, t4, t5, t6, t7, t8, t9, t10, t11, t12;
	bit32 round;
	bit64 counter = 0;
	if (threadIdx.x < 132) k[threadIdx.x] = k_d[threadIdx.x];
	__syncthreads();
	for (int i = 0; i < loop_d; i++) {
		x0 = plaintext0; x1 = plaintext1; x2 = plaintext2; x3 = plaintext3;
		// Round 4
		x0 ^= k[16];	x1 ^= k[17];	x2 ^= k[18];	x3 ^= k[19];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);

		// Round 5
		round = 5;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 6
		round = 6;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 7
		round = 7;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = b ^ c; t2 = c & t1; t3 = d ^ t2; t4 = a ^ t3; t5 = d | t1; t6 = t4 & t5; x1 = b ^ t6; t8 = t3 | x1; t9 = a & t4; x3 = t1 ^ t9; t11 = t4 ^ t8; t12 = x3 & t11; x2 = t3 ^ t12; x0 = (~t11) ^ (x3 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 8
		x0 ^= k[32];	x1 ^= k[33];	x2 ^= k[34];	x3 ^= k[35];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t3 = c ^ t1; t4 = b ^ t3; x3 = (a & d) ^ t4; t7 = a ^ (b & t1); x2 = t4 ^ (c | t7); t12 = x3 & (t3 ^ t7); x1 = (~t3) ^ t12; x0 = t12 ^ (~t7);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 9
		x0 ^= k[36];	x1 ^= k[37];	x2 ^= k[38];	x3 ^= k[39];
		a = x0; b = x1; c = x2; d = x3;
		t2 = b ^ (~a);		t5 = c ^ (a | t2);		x2 = d ^ t5;		t7 = b ^ (d | t2);		t8 = t2 ^ x2;		x3 = t8 ^ (t5 & t7); 		t11 = t5 ^ t7; 		x1 = x3 ^ t11; 		x0 = t5 ^ (t8 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);

		// Give difference
		ciphertext0 = x0; ciphertext1 = x1; ciphertext2 = x2; ciphertext3 = x3;
		x0 = plaintext0 ^ 0x20010000;
		x1 = plaintext1 ^ 0x00010000;
		x2 = plaintext2 ^ 0x00000000;
		x3 = plaintext3 ^ 0x20010000;
		//20010000000000000001000020010000

		// Round 4
		x0 ^= k[16];	x1 ^= k[17];	x2 ^= k[18];	x3 ^= k[19];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);

		// Round 5
		round = 5;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 6
		round = 6;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 7
		round = 7;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = b ^ c; t2 = c & t1; t3 = d ^ t2; t4 = a ^ t3; t5 = d | t1; t6 = t4 & t5; x1 = b ^ t6; t8 = t3 | x1; t9 = a & t4; x3 = t1 ^ t9; t11 = t4 ^ t8; t12 = x3 & t11; x2 = t3 ^ t12; x0 = (~t11) ^ (x3 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 8
		x0 ^= k[32];	x1 ^= k[33];	x2 ^= k[34];	x3 ^= k[35];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t3 = c ^ t1; t4 = b ^ t3; x3 = (a & d) ^ t4; t7 = a ^ (b & t1); x2 = t4 ^ (c | t7); t12 = x3 & (t3 ^ t7); x1 = (~t3) ^ t12; x0 = t12 ^ (~t7);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 9
		x0 ^= k[36];	x1 ^= k[37];	x2 ^= k[38];	x3 ^= k[39];
		a = x0; b = x1; c = x2; d = x3;
		t2 = b ^ (~a);		t5 = c ^ (a | t2);		x2 = d ^ t5;		t7 = b ^ (d | t2);		t8 = t2 ^ x2;		x3 = t8 ^ (t5 & t7); 		t11 = t5 ^ t7; 		x1 = x3 ^ t11; 		x0 = t5 ^ (t8 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);

		x0 ^= ciphertext0;
		x1 ^= ciphertext1;
		x2 ^= ciphertext2;
		x3 ^= ciphertext3;

		t0 = x0 & 0x84808006;
		t1 = x1 & 0x4c040336;
		t2 = x2 & 0x00002008;
		t3 = x3 & 0x8724021e;
		// 8724021e000020084c04033684808006

		t0 ^= t0 >> 1;
		t0 ^= t0 >> 2;
		t0 = (t0 & 0x11111111U) * 0x11111111U;
		t0 = (t0 >> 28) & 1;

		t1 ^= t1 >> 1;
		t1 ^= t1 >> 2;
		t1 = (t1 & 0x11111111U) * 0x11111111U;
		t1 = (t1 >> 28) & 1;

		t2 ^= t2 >> 1;
		t2 ^= t2 >> 2;
		t2 = (t2 & 0x11111111U) * 0x11111111U;
		t2 = (t2 >> 28) & 1;

		t3 ^= t3 >> 1;
		t3 ^= t3 >> 2;
		t3 = (t3 & 0x11111111U) * 0x11111111U;
		t3 = (t3 >> 28) & 1;

		t0 = t0 ^ t1 ^ t2 ^ t3;

		if (t0 == 0) counter++;

		// Change plaintext
		plaintext0 = x0; plaintext1 = x1; plaintext2 = x2; plaintext3 = x3;
	}
	hit[blockIdx.x * blockDim.x + threadIdx.x] = counter;
}
__global__ void Serpent7round_PZWD24_last6(bit32 p0, bit32 p1, bit32 p2, bit32 p3, bit64* hit, bit32* k_d) {
	__shared__ bit32  k[132];
	bit32 x0, x1, x2, x3, plaintext0 = p0, plaintext1 = p1, plaintext2 = p2, plaintext3 = p3 + blockIdx.x * blockDim.x + threadIdx.x;
	bit32 a, b, c, d;
	bit32 ciphertext0, ciphertext1, ciphertext2, ciphertext3;
	bit32 t0, t1, t2, t3, t4, t5, t6, t7, t8, t9, t10, t11, t12;
	bit32 round;
	bit64 counter = 0;
	if (threadIdx.x < 132) k[threadIdx.x] = k_d[threadIdx.x];
	__syncthreads();
	for (int i = 0; i < loop_d; i++) {
		x0 = plaintext0; x1 = plaintext1; x2 = plaintext2; x3 = plaintext3;
		// Round 2
		x0 ^= k[8];	x1 ^= k[9];	x2 ^= k[19];	x3 ^= k[11];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = b ^ d; t3 = c & t1; x0 = t2 ^ t3; t5 = c ^ t1; t6 = c ^ x0; t7 = b & t6; x3 = t5 ^ t7; x2 = a ^ ((d | t7) & (x0 | t5)); x1 = (t2 ^ x3) ^ (x2 ^ (d | t1));
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 3
		x0 ^= k[12];	x1 ^= k[13];	x2 ^= k[14];	x3 ^= k[15];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ b; t2 = a & c; t3 = a | d; t4 = c ^ d; t5 = t1 & t3; t6 = t2 | t5; x2 = t4 ^ t6; t8 = b ^ t3; t9 = t6 ^ t8; t10 = t4 & t9; x0 = t1 ^ t10; t12 = x2 & x0; x1 = t9 ^ t12; x3 = (b | d) ^ (t4 ^ t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 4
		x0 ^= k[16];	x1 ^= k[17];	x2 ^= k[18];	x3 ^= k[19];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 5
		round = 5;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 6
		round = 6;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 7
		round = 7;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = b ^ c; t2 = c & t1; t3 = d ^ t2; t4 = a ^ t3; t5 = d | t1; t6 = t4 & t5; x1 = b ^ t6; t8 = t3 | x1; t9 = a & t4; x3 = t1 ^ t9; t11 = t4 ^ t8; t12 = x3 & t11; x2 = t3 ^ t12; x0 = (~t11) ^ (x3 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);

		// Give difference
		ciphertext0 = x0; ciphertext1 = x1; ciphertext2 = x2; ciphertext3 = x3;
		x0 = plaintext0 ^ 0x00000000;
		x1 = plaintext1 ^ 0x00000000;
		x2 = plaintext2 ^ 0x00010000;
		x3 = plaintext3 ^ 0x00000000;
		//00000000000100000000000000000000

		// Round 2
		x0 ^= k[8];	x1 ^= k[9];	x2 ^= k[19];	x3 ^= k[11];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = b ^ d; t3 = c & t1; x0 = t2 ^ t3; t5 = c ^ t1; t6 = c ^ x0; t7 = b & t6; x3 = t5 ^ t7; x2 = a ^ ((d | t7) & (x0 | t5)); x1 = (t2 ^ x3) ^ (x2 ^ (d | t1));
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 3
		x0 ^= k[12];	x1 ^= k[13];	x2 ^= k[14];	x3 ^= k[15];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ b; t2 = a & c; t3 = a | d; t4 = c ^ d; t5 = t1 & t3; t6 = t2 | t5; x2 = t4 ^ t6; t8 = b ^ t3; t9 = t6 ^ t8; t10 = t4 & t9; x0 = t1 ^ t10; t12 = x2 & x0; x1 = t9 ^ t12; x3 = (b | d) ^ (t4 ^ t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 4
		x0 ^= k[16];	x1 ^= k[17];	x2 ^= k[18];	x3 ^= k[19];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 5
		round = 5;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 6
		round = 6;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 7
		round = 7;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = b ^ c; t2 = c & t1; t3 = d ^ t2; t4 = a ^ t3; t5 = d | t1; t6 = t4 & t5; x1 = b ^ t6; t8 = t3 | x1; t9 = a & t4; x3 = t1 ^ t9; t11 = t4 ^ t8; t12 = x3 & t11; x2 = t3 ^ t12; x0 = (~t11) ^ (x3 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);


		x0 ^= ciphertext0;
		x1 ^= ciphertext1;
		x2 ^= ciphertext2;
		x3 ^= ciphertext3;

		t0 = x0 & 0x42000840;
		t1 = x1 & 0x06142042;
		t2 = x2 & 0x00048400;
		t3 = x3 & 0x10000042;
		// 10000042000484000614204242000840

		t0 ^= t0 >> 1;
		t0 ^= t0 >> 2;
		t0 = (t0 & 0x11111111U) * 0x11111111U;
		t0 = (t0 >> 28) & 1;

		t1 ^= t1 >> 1;
		t1 ^= t1 >> 2;
		t1 = (t1 & 0x11111111U) * 0x11111111U;
		t1 = (t1 >> 28) & 1;

		t2 ^= t2 >> 1;
		t2 ^= t2 >> 2;
		t2 = (t2 & 0x11111111U) * 0x11111111U;
		t2 = (t2 >> 28) & 1;

		t3 ^= t3 >> 1;
		t3 ^= t3 >> 2;
		t3 = (t3 & 0x11111111U) * 0x11111111U;
		t3 = (t3 >> 28) & 1;

		t0 = t0 ^ t1 ^ t2 ^ t3;

		if (t0 == 0) counter++;

		// Change plaintext
		plaintext0 = x0; plaintext1 = x1; plaintext2 = x2; plaintext3 = x3;
	}
	hit[blockIdx.x * blockDim.x + threadIdx.x] = counter;
}
__global__ void Serpent7round_PZWD24_last6_maybe5(bit32 p0, bit32 p1, bit32 p2, bit32 p3, bit64* hit, bit32* k_d) {
	__shared__ bit32  k[132];
	bit32 x0, x1, x2, x3, plaintext0 = p0, plaintext1 = p1, plaintext2 = p2, plaintext3 = p3 + blockIdx.x * blockDim.x + threadIdx.x;
	bit32 a, b, c, d;
	bit32 ciphertext0, ciphertext1, ciphertext2, ciphertext3;
	bit32 t0, t1, t2, t3, t4, t5, t6, t7, t8, t9, t10, t11, t12;
	bit32 round;
	bit64 counter = 0;
	if (threadIdx.x < 132) k[threadIdx.x] = k_d[threadIdx.x];
	__syncthreads();
	for (int i = 0; i < loop_d; i++) {
		x0 = plaintext0; x1 = plaintext1; x2 = plaintext2; x3 = plaintext3;
		// Round 2
		x0 ^= k[8];	x1 ^= k[9];	x2 ^= k[19];	x3 ^= k[11];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = b ^ d; t3 = c & t1; x0 = t2 ^ t3; t5 = c ^ t1; t6 = c ^ x0; t7 = b & t6; x3 = t5 ^ t7; x2 = a ^ ((d | t7) & (x0 | t5)); x1 = (t2 ^ x3) ^ (x2 ^ (d | t1));
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 3
		x0 ^= k[12];	x1 ^= k[13];	x2 ^= k[14];	x3 ^= k[15];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ b; t2 = a & c; t3 = a | d; t4 = c ^ d; t5 = t1 & t3; t6 = t2 | t5; x2 = t4 ^ t6; t8 = b ^ t3; t9 = t6 ^ t8; t10 = t4 & t9; x0 = t1 ^ t10; t12 = x2 & x0; x1 = t9 ^ t12; x3 = (b | d) ^ (t4 ^ t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 4
		x0 ^= k[16];	x1 ^= k[17];	x2 ^= k[18];	x3 ^= k[19];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 5
		round = 5;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 6
		round = 6;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);

		// Give difference
		ciphertext0 = x0; ciphertext1 = x1; ciphertext2 = x2; ciphertext3 = x3;
		x0 = plaintext0 ^ 0x00000000;
		x1 = plaintext1 ^ 0x00000000;
		x2 = plaintext2 ^ 0x00010000;
		x3 = plaintext3 ^ 0x00000000;
		//00000000000100000000000000000000

		// Round 2
		x0 ^= k[8];	x1 ^= k[9];	x2 ^= k[19];	x3 ^= k[11];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = b ^ d; t3 = c & t1; x0 = t2 ^ t3; t5 = c ^ t1; t6 = c ^ x0; t7 = b & t6; x3 = t5 ^ t7; x2 = a ^ ((d | t7) & (x0 | t5)); x1 = (t2 ^ x3) ^ (x2 ^ (d | t1));
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 3
		x0 ^= k[12];	x1 ^= k[13];	x2 ^= k[14];	x3 ^= k[15];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ b; t2 = a & c; t3 = a | d; t4 = c ^ d; t5 = t1 & t3; t6 = t2 | t5; x2 = t4 ^ t6; t8 = b ^ t3; t9 = t6 ^ t8; t10 = t4 & t9; x0 = t1 ^ t10; t12 = x2 & x0; x1 = t9 ^ t12; x3 = (b | d) ^ (t4 ^ t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 4
		x0 ^= k[16];	x1 ^= k[17];	x2 ^= k[18];	x3 ^= k[19];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 5
		round = 5;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 6
		round = 6;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);


		x0 ^= ciphertext0;
		x1 ^= ciphertext1;
		x2 ^= ciphertext2;
		x3 ^= ciphertext3;

		t0 = x0 & 0x42000840;
		t1 = x1 & 0x06142042;
		t2 = x2 & 0x00048400;
		t3 = x3 & 0x10000042;
		// 10000042000484000614204242000840

		t0 ^= t0 >> 1;
		t0 ^= t0 >> 2;
		t0 = (t0 & 0x11111111U) * 0x11111111U;
		t0 = (t0 >> 28) & 1;

		t1 ^= t1 >> 1;
		t1 ^= t1 >> 2;
		t1 = (t1 & 0x11111111U) * 0x11111111U;
		t1 = (t1 >> 28) & 1;

		t2 ^= t2 >> 1;
		t2 ^= t2 >> 2;
		t2 = (t2 & 0x11111111U) * 0x11111111U;
		t2 = (t2 >> 28) & 1;

		t3 ^= t3 >> 1;
		t3 ^= t3 >> 2;
		t3 = (t3 & 0x11111111U) * 0x11111111U;
		t3 = (t3 >> 28) & 1;

		t0 = t0 ^ t1 ^ t2 ^ t3;

		if (t0 == 0) counter++;

		// Change plaintext
		plaintext0 = x0; plaintext1 = x1; plaintext2 = x2; plaintext3 = x3;
	}
	hit[blockIdx.x * blockDim.x + threadIdx.x] = counter;
}

__global__ void Serpent7round_PZWD24_middle3(bit32 p0, bit32 p1, bit32 p2, bit32 p3, bit64* hit, bit32* k_d) {
	__shared__ bit32  k[132];
	bit32 x0, x1, x2, x3, plaintext0 = p0, plaintext1 = p1, plaintext2 = p2, plaintext3 = p3 + blockIdx.x * blockDim.x + threadIdx.x;
	bit32 a, b, c, d;
	bit32 ciphertext0, ciphertext1, ciphertext2, ciphertext3;
	bit32 t0, t1, t2, t3, t4, t5, t6, t7, t8, t9, t10, t11, t12;
	bit32 round;
	bit64 counter = 0;
	if (threadIdx.x < 132) k[threadIdx.x] = k_d[threadIdx.x];
	__syncthreads();
	for (int i = 0; i < loop_d; i++) {
		x0 = plaintext0; x1 = plaintext1; x2 = plaintext2; x3 = plaintext3;
		// Round 2
		x0 ^= k[8];	x1 ^= k[9];	x2 ^= k[19];	x3 ^= k[11];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = b ^ d; t3 = c & t1; x0 = t2 ^ t3; t5 = c ^ t1; t6 = c ^ x0; t7 = b & t6; x3 = t5 ^ t7; x2 = a ^ ((d | t7) & (x0 | t5)); x1 = (t2 ^ x3) ^ (x2 ^ (d | t1));
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 3
		x0 ^= k[12];	x1 ^= k[13];	x2 ^= k[14];	x3 ^= k[15];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ b; t2 = a & c; t3 = a | d; t4 = c ^ d; t5 = t1 & t3; t6 = t2 | t5; x2 = t4 ^ t6; t8 = b ^ t3; t9 = t6 ^ t8; t10 = t4 & t9; x0 = t1 ^ t10; t12 = x2 & x0; x1 = t9 ^ t12; x3 = (b | d) ^ (t4 ^ t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 4
		x0 ^= k[16];	x1 ^= k[17];	x2 ^= k[18];	x3 ^= k[19];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);

		// Give difference
		ciphertext0 = x0; ciphertext1 = x1; ciphertext2 = x2; ciphertext3 = x3;
		x0 = plaintext0 ^ 0x00000000;
		x1 = plaintext1 ^ 0x00000000;
		x2 = plaintext2 ^ 0x00010000;
		x3 = plaintext3 ^ 0x00000000;
		//00000000000100000000000000000000

		// Round 2
		x0 ^= k[8];	x1 ^= k[9];	x2 ^= k[19];	x3 ^= k[11];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = b ^ d; t3 = c & t1; x0 = t2 ^ t3; t5 = c ^ t1; t6 = c ^ x0; t7 = b & t6; x3 = t5 ^ t7; x2 = a ^ ((d | t7) & (x0 | t5)); x1 = (t2 ^ x3) ^ (x2 ^ (d | t1));
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 3
		x0 ^= k[12];	x1 ^= k[13];	x2 ^= k[14];	x3 ^= k[15];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ b; t2 = a & c; t3 = a | d; t4 = c ^ d; t5 = t1 & t3; t6 = t2 | t5; x2 = t4 ^ t6; t8 = b ^ t3; t9 = t6 ^ t8; t10 = t4 & t9; x0 = t1 ^ t10; t12 = x2 & x0; x1 = t9 ^ t12; x3 = (b | d) ^ (t4 ^ t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 4
		x0 ^= k[16];	x1 ^= k[17];	x2 ^= k[18];	x3 ^= k[19];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);


		x0 ^= ciphertext0;
		x1 ^= ciphertext1;
		x2 ^= ciphertext2;
		x3 ^= ciphertext3;

		t0 = x0 & 0x00000000;
		t1 = x1 & 0x00000001;
		t2 = x2 & 0x20000000;
		t3 = x3 & 0x00000000;
		// 00000000200000000000000100000000

		t0 ^= t0 >> 1;
		t0 ^= t0 >> 2;
		t0 = (t0 & 0x11111111U) * 0x11111111U;
		t0 = (t0 >> 28) & 1;

		t1 ^= t1 >> 1;
		t1 ^= t1 >> 2;
		t1 = (t1 & 0x11111111U) * 0x11111111U;
		t1 = (t1 >> 28) & 1;

		t2 ^= t2 >> 1;
		t2 ^= t2 >> 2;
		t2 = (t2 & 0x11111111U) * 0x11111111U;
		t2 = (t2 >> 28) & 1;

		t3 ^= t3 >> 1;
		t3 ^= t3 >> 2;
		t3 = (t3 & 0x11111111U) * 0x11111111U;
		t3 = (t3 >> 28) & 1;

		t0 = t0 ^ t1 ^ t2 ^ t3;

		if (t0 == 0) counter++;

		// Change plaintext
		plaintext0 = x0; plaintext1 = x1; plaintext2 = x2; plaintext3 = x3;
	}
	hit[blockIdx.x * blockDim.x + threadIdx.x] = counter;
}
__global__ void Serpent7round_PZWD24_middle3_final(bit32 p0, bit32 p1, bit32 p2, bit32 p3, bit64* hit, bit32* k_d) {
	__shared__ bit32  k[132];
	bit32 x0, x1, x2, x3, plaintext0 = p0, plaintext1 = p1, plaintext2 = p2, plaintext3 = p3 + blockIdx.x * blockDim.x + threadIdx.x;
	bit32 a, b, c, d;
	bit32 ciphertext0, ciphertext1, ciphertext2, ciphertext3;
	bit32 t0, t1, t2, t3, t4, t5, t6, t7, t8, t9, t10, t11, t12;
	bit32 round;
	bit64 counter = 0;
	if (threadIdx.x < 132) k[threadIdx.x] = k_d[threadIdx.x];
	__syncthreads();
	for (int i = 0; i < loop_d; i++) {
		x0 = plaintext0; x1 = plaintext1; x2 = plaintext2; x3 = plaintext3;
		// Round 2
		x0 ^= k[8];	x1 ^= k[9];	x2 ^= k[19];	x3 ^= k[11];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = b ^ d; t3 = c & t1; x0 = t2 ^ t3; t5 = c ^ t1; t6 = c ^ x0; t7 = b & t6; x3 = t5 ^ t7; x2 = a ^ ((d | t7) & (x0 | t5)); x1 = (t2 ^ x3) ^ (x2 ^ (d | t1));
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 3
		x0 ^= k[12];	x1 ^= k[13];	x2 ^= k[14];	x3 ^= k[15];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ b; t2 = a & c; t3 = a | d; t4 = c ^ d; t5 = t1 & t3; t6 = t2 | t5; x2 = t4 ^ t6; t8 = b ^ t3; t9 = t6 ^ t8; t10 = t4 & t9; x0 = t1 ^ t10; t12 = x2 & x0; x1 = t9 ^ t12; x3 = (b | d) ^ (t4 ^ t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 4
		x0 ^= k[16];	x1 ^= k[17];	x2 ^= k[18];	x3 ^= k[19];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);

		// Give difference
		ciphertext0 = x0; ciphertext1 = x1; ciphertext2 = x2; ciphertext3 = x3;
		x0 = plaintext0 ^ 0x00000000;
		x1 = plaintext1 ^ 0x00000000;
		x2 = plaintext2 ^ 0x00010000;
		x3 = plaintext3 ^ 0x00000000;
		//00000000000100000000000000000000

		// Round 2
		x0 ^= k[8];	x1 ^= k[9];	x2 ^= k[19];	x3 ^= k[11];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = b ^ d; t3 = c & t1; x0 = t2 ^ t3; t5 = c ^ t1; t6 = c ^ x0; t7 = b & t6; x3 = t5 ^ t7; x2 = a ^ ((d | t7) & (x0 | t5)); x1 = (t2 ^ x3) ^ (x2 ^ (d | t1));
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 3
		x0 ^= k[12];	x1 ^= k[13];	x2 ^= k[14];	x3 ^= k[15];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ b; t2 = a & c; t3 = a | d; t4 = c ^ d; t5 = t1 & t3; t6 = t2 | t5; x2 = t4 ^ t6; t8 = b ^ t3; t9 = t6 ^ t8; t10 = t4 & t9; x0 = t1 ^ t10; t12 = x2 & x0; x1 = t9 ^ t12; x3 = (b | d) ^ (t4 ^ t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 4
		x0 ^= k[16];	x1 ^= k[17];	x2 ^= k[18];	x3 ^= k[19];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);


		x0 ^= ciphertext0;
		x1 ^= ciphertext1;
		x2 ^= ciphertext2;
		x3 ^= ciphertext3;

		t0 = x0 & 0x00000000;
		t1 = x1 & 0x00000002;
		t2 = x2 & 0x40000000;
		t3 = x3 & 0x00000000;
		//00000000800000000000000400000000
		//04000000000000000000000000000020
		//  

/*		t0 = x0 & 0x00000000;
		t1 = x1 & 0x00000000;
		t2 = x2 & 0x00000000;
		t3 = x3 & 0x00000100;*/
		//00000100000000000000000000000000
		

		t0 ^= t0 >> 1;
		t0 ^= t0 >> 2;
		t0 = (t0 & 0x11111111U) * 0x11111111U;
		t0 = (t0 >> 28) & 1;

		t1 ^= t1 >> 1;
		t1 ^= t1 >> 2;
		t1 = (t1 & 0x11111111U) * 0x11111111U;
		t1 = (t1 >> 28) & 1;

		t2 ^= t2 >> 1;
		t2 ^= t2 >> 2;
		t2 = (t2 & 0x11111111U) * 0x11111111U;
		t2 = (t2 >> 28) & 1;

		t3 ^= t3 >> 1;
		t3 ^= t3 >> 2;
		t3 = (t3 & 0x11111111U) * 0x11111111U;
		t3 = (t3 >> 28) & 1;

		t0 = t0 ^ t1 ^ t2 ^ t3;

		if (t0 == 0) counter++;

		// Change plaintext
		plaintext0 = x0; plaintext1 = x1; plaintext2 = x2; plaintext3 = x3;
	}
	hit[blockIdx.x * blockDim.x + threadIdx.x] = counter;
}
__global__ void Serpent7round_PZWD24_middle3_old(bit32 p0, bit32 p1, bit32 p2, bit32 p3, bit64* hit, bit32* k_d) {
	__shared__ bit32  k[132];
	bit32 x0, x1, x2, x3, plaintext0 = p0, plaintext1 = p1, plaintext2 = p2, plaintext3 = p3 + blockIdx.x * blockDim.x + threadIdx.x;
	bit32 a, b, c, d;
	bit32 ciphertext0, ciphertext1, ciphertext2, ciphertext3;
	bit32 t0, t1, t2, t3, t4, t5, t6, t7, t8, t9, t10, t11, t12;
	bit32 round;
	bit64 counter = 0;
	if (threadIdx.x < 132) k[threadIdx.x] = k_d[threadIdx.x];
	__syncthreads();
	for (int i = 0; i < loop_d; i++) {
		x0 = plaintext0; x1 = plaintext1; x2 = plaintext2; x3 = plaintext3;
		// Round 2
		x0 ^= k[8];	x1 ^= k[9];	x2 ^= k[19];	x3 ^= k[11];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = b ^ d; t3 = c & t1; x0 = t2 ^ t3; t5 = c ^ t1; t6 = c ^ x0; t7 = b & t6; x3 = t5 ^ t7; x2 = a ^ ((d | t7) & (x0 | t5)); x1 = (t2 ^ x3) ^ (x2 ^ (d | t1));
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 3
		x0 ^= k[12];	x1 ^= k[13];	x2 ^= k[14];	x3 ^= k[15];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ b; t2 = a & c; t3 = a | d; t4 = c ^ d; t5 = t1 & t3; t6 = t2 | t5; x2 = t4 ^ t6; t8 = b ^ t3; t9 = t6 ^ t8; t10 = t4 & t9; x0 = t1 ^ t10; t12 = x2 & x0; x1 = t9 ^ t12; x3 = (b | d) ^ (t4 ^ t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 4
		x0 ^= k[16];	x1 ^= k[17];	x2 ^= k[18];	x3 ^= k[19];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);

		// Give difference
		ciphertext0 = x0; ciphertext1 = x1; ciphertext2 = x2; ciphertext3 = x3;
		x0 = plaintext0 ^ 0x00000000;
		x1 = plaintext1 ^ 0x00000000;
		x2 = plaintext2 ^ 0x00010000;
		x3 = plaintext3 ^ 0x00000000;
		//00000000000100000000000000000000

		// Round 2
		x0 ^= k[8];	x1 ^= k[9];	x2 ^= k[19];	x3 ^= k[11];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = b ^ d; t3 = c & t1; x0 = t2 ^ t3; t5 = c ^ t1; t6 = c ^ x0; t7 = b & t6; x3 = t5 ^ t7; x2 = a ^ ((d | t7) & (x0 | t5)); x1 = (t2 ^ x3) ^ (x2 ^ (d | t1));
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 3
		x0 ^= k[12];	x1 ^= k[13];	x2 ^= k[14];	x3 ^= k[15];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ b; t2 = a & c; t3 = a | d; t4 = c ^ d; t5 = t1 & t3; t6 = t2 | t5; x2 = t4 ^ t6; t8 = b ^ t3; t9 = t6 ^ t8; t10 = t4 & t9; x0 = t1 ^ t10; t12 = x2 & x0; x1 = t9 ^ t12; x3 = (b | d) ^ (t4 ^ t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 4
		x0 ^= k[16];	x1 ^= k[17];	x2 ^= k[18];	x3 ^= k[19];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);


		x0 ^= ciphertext0;
		x1 ^= ciphertext1;
		x2 ^= ciphertext2;
		x3 ^= ciphertext3;

		t0 = x0 & 0x00000000;
		t1 = x1 & 0x00000000;
		t2 = x2 & 0x00000000;
		t3 = x3 & 0x00000100;
		// 00000100000000000000000000000000

		t0 ^= t0 >> 1;
		t0 ^= t0 >> 2;
		t0 = (t0 & 0x11111111U) * 0x11111111U;
		t0 = (t0 >> 28) & 1;

		t1 ^= t1 >> 1;
		t1 ^= t1 >> 2;
		t1 = (t1 & 0x11111111U) * 0x11111111U;
		t1 = (t1 >> 28) & 1;

		t2 ^= t2 >> 1;
		t2 ^= t2 >> 2;
		t2 = (t2 & 0x11111111U) * 0x11111111U;
		t2 = (t2 >> 28) & 1;

		t3 ^= t3 >> 1;
		t3 ^= t3 >> 2;
		t3 = (t3 & 0x11111111U) * 0x11111111U;
		t3 = (t3 >> 28) & 1;

		t0 = t0 ^ t1 ^ t2 ^ t3;

		if (t0 == 0) counter++;

		// Change plaintext
		plaintext0 = x0; plaintext1 = x1; plaintext2 = x2; plaintext3 = x3;
	}
	hit[blockIdx.x * blockDim.x + threadIdx.x] = counter;
}
__global__ void Serpent7round_PZWD24_middle4(bit32 p0, bit32 p1, bit32 p2, bit32 p3, bit64* hit, bit32* k_d) {
	__shared__ bit32  k[132];
	bit32 x0, x1, x2, x3, plaintext0 = p0, plaintext1 = p1, plaintext2 = p2, plaintext3 = p3 + blockIdx.x * blockDim.x + threadIdx.x;
	bit32 a, b, c, d;
	bit32 ciphertext0, ciphertext1, ciphertext2, ciphertext3;
	bit32 t0, t1, t2, t3, t4, t5, t6, t7, t8, t9, t10, t11, t12;
	bit32 round;
	bit64 counter = 0;
	if (threadIdx.x < 132) k[threadIdx.x] = k_d[threadIdx.x];
	__syncthreads();
	for (int i = 0; i < loop_d; i++) {
		x0 = plaintext0; x1 = plaintext1; x2 = plaintext2; x3 = plaintext3;
		// Round 2
		x0 ^= k[8];	x1 ^= k[9];	x2 ^= k[19];	x3 ^= k[11];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = b ^ d; t3 = c & t1; x0 = t2 ^ t3; t5 = c ^ t1; t6 = c ^ x0; t7 = b & t6; x3 = t5 ^ t7; x2 = a ^ ((d | t7) & (x0 | t5)); x1 = (t2 ^ x3) ^ (x2 ^ (d | t1));
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 3
		x0 ^= k[12];	x1 ^= k[13];	x2 ^= k[14];	x3 ^= k[15];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ b; t2 = a & c; t3 = a | d; t4 = c ^ d; t5 = t1 & t3; t6 = t2 | t5; x2 = t4 ^ t6; t8 = b ^ t3; t9 = t6 ^ t8; t10 = t4 & t9; x0 = t1 ^ t10; t12 = x2 & x0; x1 = t9 ^ t12; x3 = (b | d) ^ (t4 ^ t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 4
		x0 ^= k[16];	x1 ^= k[17];	x2 ^= k[18];	x3 ^= k[19];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 5
		x0 ^= k[20];	x1 ^= k[21];	x2 ^= k[22];	x3 ^= k[23];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);

		// Give difference
		ciphertext0 = x0; ciphertext1 = x1; ciphertext2 = x2; ciphertext3 = x3;
		x0 = plaintext0 ^ 0x00000000;
		x1 = plaintext1 ^ 0x00000000;
		x2 = plaintext2 ^ 0x00010000;
		x3 = plaintext3 ^ 0x00000000;
		//00000000000100000000000000000000

		// Round 2
		x0 ^= k[8];	x1 ^= k[9];	x2 ^= k[19];	x3 ^= k[11];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = b ^ d; t3 = c & t1; x0 = t2 ^ t3; t5 = c ^ t1; t6 = c ^ x0; t7 = b & t6; x3 = t5 ^ t7; x2 = a ^ ((d | t7) & (x0 | t5)); x1 = (t2 ^ x3) ^ (x2 ^ (d | t1));
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 3
		x0 ^= k[12];	x1 ^= k[13];	x2 ^= k[14];	x3 ^= k[15];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ b; t2 = a & c; t3 = a | d; t4 = c ^ d; t5 = t1 & t3; t6 = t2 | t5; x2 = t4 ^ t6; t8 = b ^ t3; t9 = t6 ^ t8; t10 = t4 & t9; x0 = t1 ^ t10; t12 = x2 & x0; x1 = t9 ^ t12; x3 = (b | d) ^ (t4 ^ t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 4
		x0 ^= k[16];	x1 ^= k[17];	x2 ^= k[18];	x3 ^= k[19];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 5
		x0 ^= k[20];	x1 ^= k[21];	x2 ^= k[22];	x3 ^= k[23];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);


		x0 ^= ciphertext0;
		x1 ^= ciphertext1;
		x2 ^= ciphertext2;
		x3 ^= ciphertext3;

		t0 = x0 & 0x00000000;
		t1 = x1 & 0x00000000;
		t2 = x2 & 0x00000000;
		t3 = x3 & 0x00000100;
		//00000100000000000000000000000000


		t0 ^= t0 >> 1;
		t0 ^= t0 >> 2;
		t0 = (t0 & 0x11111111U) * 0x11111111U;
		t0 = (t0 >> 28) & 1;

		t1 ^= t1 >> 1;
		t1 ^= t1 >> 2;
		t1 = (t1 & 0x11111111U) * 0x11111111U;
		t1 = (t1 >> 28) & 1;

		t2 ^= t2 >> 1;
		t2 ^= t2 >> 2;
		t2 = (t2 & 0x11111111U) * 0x11111111U;
		t2 = (t2 >> 28) & 1;

		t3 ^= t3 >> 1;
		t3 ^= t3 >> 2;
		t3 = (t3 & 0x11111111U) * 0x11111111U;
		t3 = (t3 >> 28) & 1;

		t0 = t0 ^ t1 ^ t2 ^ t3;

		if (t0 == 0) counter++;

		// Change plaintext
		plaintext0 = x0; plaintext1 = x1; plaintext2 = x2; plaintext3 = x3;
	}
	hit[blockIdx.x * blockDim.x + threadIdx.x] = counter;
}
__global__ void Serpent7round_PZWD24_middle5(bit32 p0, bit32 p1, bit32 p2, bit32 p3, bit64* hit, bit32* k_d) {
	__shared__ bit32  k[132];
	bit32 x0, x1, x2, x3, plaintext0 = p0, plaintext1 = p1, plaintext2 = p2, plaintext3 = p3 + blockIdx.x * blockDim.x + threadIdx.x;
	bit32 a, b, c, d;
	bit32 ciphertext0, ciphertext1, ciphertext2, ciphertext3;
	bit32 t0, t1, t2, t3, t4, t5, t6, t7, t8, t9, t10, t11, t12;
	bit32 round;
	bit64 counter = 0;
	if (threadIdx.x < 132) k[threadIdx.x] = k_d[threadIdx.x];
	__syncthreads();
	for (int i = 0; i < loop_d; i++) {
		x0 = plaintext0; x1 = plaintext1; x2 = plaintext2; x3 = plaintext3;
		// Round 2
		x0 ^= k[8];	x1 ^= k[9];	x2 ^= k[19];	x3 ^= k[11];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = b ^ d; t3 = c & t1; x0 = t2 ^ t3; t5 = c ^ t1; t6 = c ^ x0; t7 = b & t6; x3 = t5 ^ t7; x2 = a ^ ((d | t7) & (x0 | t5)); x1 = (t2 ^ x3) ^ (x2 ^ (d | t1));
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 3
		x0 ^= k[12];	x1 ^= k[13];	x2 ^= k[14];	x3 ^= k[15];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ b; t2 = a & c; t3 = a | d; t4 = c ^ d; t5 = t1 & t3; t6 = t2 | t5; x2 = t4 ^ t6; t8 = b ^ t3; t9 = t6 ^ t8; t10 = t4 & t9; x0 = t1 ^ t10; t12 = x2 & x0; x1 = t9 ^ t12; x3 = (b | d) ^ (t4 ^ t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 4
		x0 ^= k[16];	x1 ^= k[17];	x2 ^= k[18];	x3 ^= k[19];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 5
		x0 ^= k[20];	x1 ^= k[21];	x2 ^= k[22];	x3 ^= k[23];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 6
		x0 ^= k[24];	x1 ^= k[25];	x2 ^= k[26];	x3 ^= k[27];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);

		// Give difference
		ciphertext0 = x0; ciphertext1 = x1; ciphertext2 = x2; ciphertext3 = x3;
		x0 = plaintext0 ^ 0x00000000;
		x1 = plaintext1 ^ 0x00000000;
		x2 = plaintext2 ^ 0x00010000;
		x3 = plaintext3 ^ 0x00000000;
		//00000000000100000000000000000000

		// Round 2
		x0 ^= k[8];	x1 ^= k[9];	x2 ^= k[19];	x3 ^= k[11];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = b ^ d; t3 = c & t1; x0 = t2 ^ t3; t5 = c ^ t1; t6 = c ^ x0; t7 = b & t6; x3 = t5 ^ t7; x2 = a ^ ((d | t7) & (x0 | t5)); x1 = (t2 ^ x3) ^ (x2 ^ (d | t1));
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 3
		x0 ^= k[12];	x1 ^= k[13];	x2 ^= k[14];	x3 ^= k[15];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ b; t2 = a & c; t3 = a | d; t4 = c ^ d; t5 = t1 & t3; t6 = t2 | t5; x2 = t4 ^ t6; t8 = b ^ t3; t9 = t6 ^ t8; t10 = t4 & t9; x0 = t1 ^ t10; t12 = x2 & x0; x1 = t9 ^ t12; x3 = (b | d) ^ (t4 ^ t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 4
		x0 ^= k[16];	x1 ^= k[17];	x2 ^= k[18];	x3 ^= k[19];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 5
		x0 ^= k[20];	x1 ^= k[21];	x2 ^= k[22];	x3 ^= k[23];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 6
		x0 ^= k[24];	x1 ^= k[25];	x2 ^= k[26];	x3 ^= k[27];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);


		x0 ^= ciphertext0;
		x1 ^= ciphertext1;
		x2 ^= ciphertext2;
		x3 ^= ciphertext3;

		t0 = x0 & 0x04000000;
		t1 = x1 & 0x00200000;
		t2 = x2 & 0x00000000;
		t3 = x3 & 0x00200000;
		//00200000000000000020000004000000

		t0 ^= t0 >> 1;
		t0 ^= t0 >> 2;
		t0 = (t0 & 0x11111111U) * 0x11111111U;
		t0 = (t0 >> 28) & 1;

		t1 ^= t1 >> 1;
		t1 ^= t1 >> 2;
		t1 = (t1 & 0x11111111U) * 0x11111111U;
		t1 = (t1 >> 28) & 1;

		t2 ^= t2 >> 1;
		t2 ^= t2 >> 2;
		t2 = (t2 & 0x11111111U) * 0x11111111U;
		t2 = (t2 >> 28) & 1;

		t3 ^= t3 >> 1;
		t3 ^= t3 >> 2;
		t3 = (t3 & 0x11111111U) * 0x11111111U;
		t3 = (t3 >> 28) & 1;

		t0 = t0 ^ t1 ^ t2 ^ t3;

		if (t0 == 0) counter++;

		// Change plaintext
		plaintext0 = x0; plaintext1 = x1; plaintext2 = x2; plaintext3 = x3;
	}
	hit[blockIdx.x * blockDim.x + threadIdx.x] = counter;
}
__global__ void Serpent7round_PZWD24_middle6(bit32 p0, bit32 p1, bit32 p2, bit32 p3, bit64* hit, bit32* k_d) {
	__shared__ bit32  k[132];
	bit32 x0, x1, x2, x3, plaintext0 = p0, plaintext1 = p1, plaintext2 = p2, plaintext3 = p3 + blockIdx.x * blockDim.x + threadIdx.x;
	bit32 a, b, c, d;
	bit32 ciphertext0, ciphertext1, ciphertext2, ciphertext3;
	bit32 t0, t1, t2, t3, t4, t5, t6, t7, t8, t9, t10, t11, t12;
	bit32 round;
	bit64 counter = 0;
	if (threadIdx.x < 132) k[threadIdx.x] = k_d[threadIdx.x];
	__syncthreads();
	for (int i = 0; i < loop_d; i++) {
		x0 = plaintext0; x1 = plaintext1; x2 = plaintext2; x3 = plaintext3;
		// Round 2
		x0 ^= k[8];	x1 ^= k[9];	x2 ^= k[19];	x3 ^= k[11];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = b ^ d; t3 = c & t1; x0 = t2 ^ t3; t5 = c ^ t1; t6 = c ^ x0; t7 = b & t6; x3 = t5 ^ t7; x2 = a ^ ((d | t7) & (x0 | t5)); x1 = (t2 ^ x3) ^ (x2 ^ (d | t1));
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 3
		x0 ^= k[12];	x1 ^= k[13];	x2 ^= k[14];	x3 ^= k[15];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ b; t2 = a & c; t3 = a | d; t4 = c ^ d; t5 = t1 & t3; t6 = t2 | t5; x2 = t4 ^ t6; t8 = b ^ t3; t9 = t6 ^ t8; t10 = t4 & t9; x0 = t1 ^ t10; t12 = x2 & x0; x1 = t9 ^ t12; x3 = (b | d) ^ (t4 ^ t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 4
		x0 ^= k[16];	x1 ^= k[17];	x2 ^= k[18];	x3 ^= k[19];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 5
		x0 ^= k[20];	x1 ^= k[21];	x2 ^= k[22];	x3 ^= k[23];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 6
		x0 ^= k[24];	x1 ^= k[25];	x2 ^= k[26];	x3 ^= k[27];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 7
		x0 ^= k[28];	x1 ^= k[29];	x2 ^= k[30];	x3 ^= k[31];
		a = x0; b = x1; c = x2; d = x3;
		t1 = b ^ c; t2 = c & t1; t3 = d ^ t2; t4 = a ^ t3; t5 = d | t1; t6 = t4 & t5; x1 = b ^ t6; t8 = t3 | x1; t9 = a & t4; x3 = t1 ^ t9; t11 = t4 ^ t8; t12 = x3 & t11; x2 = t3 ^ t12; x0 = (~t11) ^ (x3 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);

		// Give difference
		ciphertext0 = x0; ciphertext1 = x1; ciphertext2 = x2; ciphertext3 = x3;
		x0 = plaintext0 ^ 0x00000000;
		x1 = plaintext1 ^ 0x00000000;
		x2 = plaintext2 ^ 0x00010000;
		x3 = plaintext3 ^ 0x00000000;
		//00000000000100000000000000000000

		// Round 2
		x0 ^= k[8];	x1 ^= k[9];	x2 ^= k[19];	x3 ^= k[11];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = b ^ d; t3 = c & t1; x0 = t2 ^ t3; t5 = c ^ t1; t6 = c ^ x0; t7 = b & t6; x3 = t5 ^ t7; x2 = a ^ ((d | t7) & (x0 | t5)); x1 = (t2 ^ x3) ^ (x2 ^ (d | t1));
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 3
		x0 ^= k[12];	x1 ^= k[13];	x2 ^= k[14];	x3 ^= k[15];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ b; t2 = a & c; t3 = a | d; t4 = c ^ d; t5 = t1 & t3; t6 = t2 | t5; x2 = t4 ^ t6; t8 = b ^ t3; t9 = t6 ^ t8; t10 = t4 & t9; x0 = t1 ^ t10; t12 = x2 & x0; x1 = t9 ^ t12; x3 = (b | d) ^ (t4 ^ t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 4
		x0 ^= k[16];	x1 ^= k[17];	x2 ^= k[18];	x3 ^= k[19];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 5
		x0 ^= k[20];	x1 ^= k[21];	x2 ^= k[22];	x3 ^= k[23];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 6
		x0 ^= k[24];	x1 ^= k[25];	x2 ^= k[26];	x3 ^= k[27];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 7
		x0 ^= k[28];	x1 ^= k[29];	x2 ^= k[30];	x3 ^= k[31];
		a = x0; b = x1; c = x2; d = x3;
		t1 = b ^ c; t2 = c & t1; t3 = d ^ t2; t4 = a ^ t3; t5 = d | t1; t6 = t4 & t5; x1 = b ^ t6; t8 = t3 | x1; t9 = a & t4; x3 = t1 ^ t9; t11 = t4 ^ t8; t12 = x3 & t11; x2 = t3 ^ t12; x0 = (~t11) ^ (x3 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);


		x0 ^= ciphertext0;
		x1 ^= ciphertext1;
		x2 ^= ciphertext2;
		x3 ^= ciphertext3;

		t0 = x0 & 0x84001080;
		t1 = x1 & 0x0c284084;
		t2 = x2 & 0x00090800;
		t3 = x3 & 0x20000084;
		//20000084000908000c28408484001080

		t0 ^= t0 >> 1;
		t0 ^= t0 >> 2;
		t0 = (t0 & 0x11111111U) * 0x11111111U;
		t0 = (t0 >> 28) & 1;

		t1 ^= t1 >> 1;
		t1 ^= t1 >> 2;
		t1 = (t1 & 0x11111111U) * 0x11111111U;
		t1 = (t1 >> 28) & 1;

		t2 ^= t2 >> 1;
		t2 ^= t2 >> 2;
		t2 = (t2 & 0x11111111U) * 0x11111111U;
		t2 = (t2 >> 28) & 1;

		t3 ^= t3 >> 1;
		t3 ^= t3 >> 2;
		t3 = (t3 & 0x11111111U) * 0x11111111U;
		t3 = (t3 >> 28) & 1;

		t0 = t0 ^ t1 ^ t2 ^ t3;

		if (t0 == 0) counter++;

		// Change plaintext
		plaintext0 = x0; plaintext1 = x1; plaintext2 = x2; plaintext3 = x3;
	}
	hit[blockIdx.x * blockDim.x + threadIdx.x] = counter;
}


__global__ void Serpent_benchmark(bit32 p0, bit32 p1, bit32 p2, bit32 p3, bit64* hit, bit32* k_d) {
	__shared__ bit32  k[132];
	bit32 x0, x1, x2, x3, plaintext0 = p0, plaintext1 = p1, plaintext2 = p2, plaintext3 = p3 + blockIdx.x * blockDim.x + threadIdx.x;
	bit32 ciphertext0;
	bit32 a, b, c, d;
	bit32 t1, t2, t3, t4, t5, t6, t7, t8, t9, t10, t11, t12;
	bit64 counter = 0;
	if (threadIdx.x < 132) k[threadIdx.x] = k_d[threadIdx.x];
	__syncthreads();
	for (int i = 0; i < loop_d; i++) {
		x0 = plaintext0; x1 = plaintext1; x2 = plaintext2; x3 = plaintext3;
//#pragma unroll
/*		for (int i = 0; i < 32; i++) {
			x0 ^= k[i*4];	x1 ^= k[i * 4 + 1];	x2 ^= k[i * 4 + 2]; x3 ^= k[i * 4 + 3];
			a = x0; b = x1; c = x2; d = x3;
			t1 = ~a; t2 = b ^ d; t3 = c & t1; x0 = t2 ^ t3; t5 = c ^ t1; t6 = c ^ x0; t7 = b & t6; x3 = t5 ^ t7; x2 = a ^ ((d | t7) & (x0 | t5)); x1 = (t2 ^ x3) ^ (x2 ^ (d | t1));
			x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		}*/

		// Round 0
		x0 ^= k[0];	x1 ^= k[1];	x2 ^= k[2];	x3 ^= k[3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t3 = c ^ t1; t4 = b ^ t3; x3 = (a & d) ^ t4; t7 = a ^ (b & t1); x2 = t4 ^ (c | t7); t12 = x3 & (t3 ^ t7); x1 = (~t3) ^ t12; x0 = t12 ^ (~t7);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 1
		x0 ^= k[4];	x1 ^= k[5];	x2 ^= k[6];	x3 ^= k[7];
		a = x0; b = x1; c = x2; d = x3;
		t2 = b ^ (~a);		t5 = c ^ (a | t2);		x2 = d ^ t5;		t7 = b ^ (d | t2);		t8 = t2 ^ x2;		x3 = t8 ^ (t5 & t7); 		t11 = t5 ^ t7; 		x1 = x3 ^ t11; 		x0 = t5 ^ (t8 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 2
		x0 ^= k[8];	x1 ^= k[9];	x2 ^= k[19];	x3 ^= k[11];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = b ^ d; t3 = c & t1; x0 = t2 ^ t3; t5 = c ^ t1; t6 = c ^ x0; t7 = b & t6; x3 = t5 ^ t7; x2 = a ^ ((d | t7) & (x0 | t5)); x1 = (t2 ^ x3) ^ (x2 ^ (d | t1));
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 3
		x0 ^= k[12];	x1 ^= k[13];	x2 ^= k[14];	x3 ^= k[15];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ b; t2 = a & c; t3 = a | d; t4 = c ^ d; t5 = t1 & t3; t6 = t2 | t5; x2 = t4 ^ t6; t8 = b ^ t3; t9 = t6 ^ t8; t10 = t4 & t9; x0 = t1 ^ t10; t12 = x2 & x0; x1 = t9 ^ t12; x3 = (b | d) ^ (t4 ^ t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 4
		x0 ^= k[16];	x1 ^= k[17];	x2 ^= k[18];	x3 ^= k[19];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 5
		x0 ^= k[20];	x1 ^= k[21];	x2 ^= k[22];	x3 ^= k[23];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 6
		x0 ^= k[24];	x1 ^= k[25];	x2 ^= k[26];	x3 ^= k[27];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 7
		x0 ^= k[28];	x1 ^= k[29];	x2 ^= k[30];	x3 ^= k[31];
		a = x0; b = x1; c = x2; d = x3;
		t1 = b ^ c; t2 = c & t1; t3 = d ^ t2; t4 = a ^ t3; t5 = d | t1; t6 = t4 & t5; x1 = b ^ t6; t8 = t3 | x1; t9 = a & t4; x3 = t1 ^ t9; t11 = t4 ^ t8; t12 = x3 & t11; x2 = t3 ^ t12; x0 = (~t11) ^ (x3 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 8
		x0 ^= k[32];	x1 ^= k[33];	x2 ^= k[34];	x3 ^= k[35];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t3 = c ^ t1; t4 = b ^ t3; x3 = (a & d) ^ t4; t7 = a ^ (b & t1); x2 = t4 ^ (c | t7); t12 = x3 & (t3 ^ t7); x1 = (~t3) ^ t12; x0 = t12 ^ (~t7);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 9
		x0 ^= k[36];	x1 ^= k[37];	x2 ^= k[38];	x3 ^= k[39];
		a = x0; b = x1; c = x2; d = x3;
		t2 = b ^ (~a);		t5 = c ^ (a | t2);		x2 = d ^ t5;		t7 = b ^ (d | t2);		t8 = t2 ^ x2;		x3 = t8 ^ (t5 & t7); 		t11 = t5 ^ t7; 		x1 = x3 ^ t11; 		x0 = t5 ^ (t8 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 10
		x0 ^= k[40];	x1 ^= k[41];	x2 ^= k[42];	x3 ^= k[43];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = b ^ d; t3 = c & t1; x0 = t2 ^ t3; t5 = c ^ t1; t6 = c ^ x0; t7 = b & t6; x3 = t5 ^ t7; x2 = a ^ ((d | t7) & (x0 | t5)); x1 = (t2 ^ x3) ^ (x2 ^ (d | t1));
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 11
		x0 ^= k[44];	x1 ^= k[45];	x2 ^= k[46];	x3 ^= k[47];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ b; t2 = a & c; t3 = a | d; t4 = c ^ d; t5 = t1 & t3; t6 = t2 | t5; x2 = t4 ^ t6; t8 = b ^ t3; t9 = t6 ^ t8; t10 = t4 & t9; x0 = t1 ^ t10; t12 = x2 & x0; x1 = t9 ^ t12; x3 = (b | d) ^ (t4 ^ t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 12
		x0 ^= k[48];	x1 ^= k[49];	x2 ^= k[50];	x3 ^= k[51];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 13
		x0 ^= k[52];	x1 ^= k[53];	x2 ^= k[54];	x3 ^= k[55];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 14
		x0 ^= k[56];	x1 ^= k[57];	x2 ^= k[58];	x3 ^= k[59];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 15
		x0 ^= k[60];	x1 ^= k[61];	x2 ^= k[62];	x3 ^= k[63];
		a = x0; b = x1; c = x2; d = x3;
		t1 = b ^ c; t2 = c & t1; t3 = d ^ t2; t4 = a ^ t3; t5 = d | t1; t6 = t4 & t5; x1 = b ^ t6; t8 = t3 | x1; t9 = a & t4; x3 = t1 ^ t9; t11 = t4 ^ t8; t12 = x3 & t11; x2 = t3 ^ t12; x0 = (~t11) ^ (x3 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);

		// Round 16
		x0 ^= k[64];	x1 ^= k[65];	x2 ^= k[66];	x3 ^= k[67];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d;		t3 = c ^ t1;		t4 = b ^ t3;		x3 = (a & d) ^ t4;		t7 = a ^ (b & t1);		x2 = t4 ^ (c | t7);		t12 = x3 & (t3 ^ t7);		x1 = (~t3) ^ t12;		x0 = t12 ^ (~t7);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 17
		x0 ^= k[68];	x1 ^= k[69];	x2 ^= k[70];	x3 ^= k[71];
		a = x0; b = x1; c = x2; d = x3;
		t2 = b ^ (~a);		t5 = c ^ (a | t2);		x2 = d ^ t5;		t7 = b ^ (d | t2);		t8 = t2 ^ x2;		x3 = t8 ^ (t5 & t7); 		t11 = t5 ^ t7; 		x1 = x3 ^ t11; 		x0 = t5 ^ (t8 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 18
		x0 ^= k[72];	x1 ^= k[73];	x2 ^= k[74];	x3 ^= k[75];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = b ^ d; t3 = c & t1; x0 = t2 ^ t3; t5 = c ^ t1; t6 = c ^ x0; t7 = b & t6; x3 = t5 ^ t7; x2 = a ^ ((d | t7) & (x0 | t5)); x1 = (t2 ^ x3) ^ (x2 ^ (d | t1));
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 19
		x0 ^= k[76];	x1 ^= k[77];	x2 ^= k[78];	x3 ^= k[79];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ b; t2 = a & c; t3 = a | d; t4 = c ^ d; t5 = t1 & t3; t6 = t2 | t5; x2 = t4 ^ t6; t8 = b ^ t3; t9 = t6 ^ t8; t10 = t4 & t9; x0 = t1 ^ t10; t12 = x2 & x0; x1 = t9 ^ t12; x3 = (b | d) ^ (t4 ^ t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 20
		x0 ^= k[80];	x1 ^= k[81];	x2 ^= k[82];	x3 ^= k[83];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 21
		x0 ^= k[84];	x1 ^= k[85];	x2 ^= k[86];	x3 ^= k[87];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 22
		x0 ^= k[88];	x1 ^= k[89];	x2 ^= k[90];	x3 ^= k[91];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 23
		x0 ^= k[92];	x1 ^= k[93];	x2 ^= k[94];	x3 ^= k[95];
		a = x0; b = x1; c = x2; d = x3;
		t1 = b ^ c; t2 = c & t1; t3 = d ^ t2; t4 = a ^ t3; t5 = d | t1; t6 = t4 & t5; x1 = b ^ t6; t8 = t3 | x1; t9 = a & t4; x3 = t1 ^ t9; t11 = t4 ^ t8; t12 = x3 & t11; x2 = t3 ^ t12; x0 = (~t11) ^ (x3 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 24
		x0 ^= k[96];	x1 ^= k[97];	x2 ^= k[98];	x3 ^= k[99];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t3 = c ^ t1; t4 = b ^ t3; x3 = (a & d) ^ t4; t7 = a ^ (b & t1); x2 = t4 ^ (c | t7); t12 = x3 & (t3 ^ t7); x1 = (~t3) ^ t12; x0 = t12 ^ (~t7);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 25
		x0 ^= k[100];	x1 ^= k[101];	x2 ^= k[102];	x3 ^= k[103];
		a = x0; b = x1; c = x2; d = x3;
		t2 = b ^ (~a);		t5 = c ^ (a | t2);		x2 = d ^ t5;		t7 = b ^ (d | t2);		t8 = t2 ^ x2;		x3 = t8 ^ (t5 & t7); 		t11 = t5 ^ t7; 		x1 = x3 ^ t11; 		x0 = t5 ^ (t8 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 26
		x0 ^= k[104];	x1 ^= k[105];	x2 ^= k[106];	x3 ^= k[107];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = b ^ d; t3 = c & t1; x0 = t2 ^ t3; t5 = c ^ t1; t6 = c ^ x0; t7 = b & t6; x3 = t5 ^ t7; x2 = a ^ ((d | t7) & (x0 | t5)); x1 = (t2 ^ x3) ^ (x2 ^ (d | t1));
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 27
		x0 ^= k[108];	x1 ^= k[109];	x2 ^= k[110];	x3 ^= k[111];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ b; t2 = a & c; t3 = a | d; t4 = c ^ d; t5 = t1 & t3; t6 = t2 | t5; x2 = t4 ^ t6; t8 = b ^ t3; t9 = t6 ^ t8; t10 = t4 & t9; x0 = t1 ^ t10; t12 = x2 & x0; x1 = t9 ^ t12; x3 = (b | d) ^ (t4 ^ t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 28
		x0 ^= k[112];	x1 ^= k[113];	x2 ^= k[114];	x3 ^= k[115];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 29
		x0 ^= k[116];	x1 ^= k[117];	x2 ^= k[118];	x3 ^= k[119];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 30
		x0 ^= k[120];	x1 ^= k[121];	x2 ^= k[122];	x3 ^= k[123];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 31
		x0 ^= k[124];	x1 ^= k[125];	x2 ^= k[126];	x3 ^= k[127];
		a = x0; b = x1; c = x2; d = x3;
		t1 = b ^ c; t2 = c & t1; t3 = d ^ t2; t4 = a ^ t3; t5 = d | t1; t6 = t4 & t5; x1 = b ^ t6; t8 = t3 | x1; t9 = a & t4; x3 = t1 ^ t9; t11 = t4 ^ t8; t12 = x3 & t11; x2 = t3 ^ t12; x0 = (~t11) ^ (x3 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 31
		x0 ^= k[128];	x1 ^= k[129];	x2 ^= k[130];	x3 ^= k[131];

		if (x0 == 0)
			if (x1 == 0)
				if (x2 == 0)
					if (x3 == 0)
						printf("Hello world\n");

		plaintext0 = x0; plaintext1 = x1; plaintext2 = x2; plaintext3 = x3;
	}
	hit[blockIdx.x * blockDim.x + threadIdx.x] = counter;
}
int main_old() {
	FILE *fp3, *fp,*fp2,*fp4,*fp5;
	bit64 *hit_d;
	__int64 temp, hit, bias = 0, cumulative_bias[100] = { 0 }, average_bias;
	bit32 *k_d, keylist[100][4];
	bit64 hit_test[65536] = { 0 };
	int t, j, i;
	hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
	get_user_inputs();
	hipSetDevice(devicenumber);
	hipMalloc((void **)&hit_d, 65536 * sizeof(bit64));
	hipMalloc((void **)&k_d, 132 * sizeof(bit32));
	if (numberofrounds == 4) fp = fopen("4round_cumulative.txt", "ab");
	else if (numberofrounds == 5) fp = fopen("5round_cumulative.txt", "ab");
	else if (numberofrounds == 6) fp = fopen("6round_cumulative.txt", "ab");
	else if (numberofrounds == 7) fp = fopen("7round_cumulative.txt", "ab");
	else exit(1);
	fprintf(fp, "Pairs: %I64d\n", blocksize*threadsize*loop*pairs);
//	plaintext[0] = 0xD095576F; plaintext[1] = 0xCEA3E3A7; plaintext[2] = 0xED98D9F2; plaintext[3] = 0x9073D78E;
//	plaintext[0] = 0xf05e1f01; plaintext[1] = 0x3acc879d; plaintext[2] = 0xa3aa601c; plaintext[3] = 0xcc14000b;
	plaintext[0] = 0xc7675a6e; plaintext[1] = 0xe3d21628; plaintext[2] = 0x9f090c02; plaintext[3] = 0x5eeaf89d;
	fp3 = fopen("keys.txt", "r");
	for (i = 0; i < 100; i++)	{ fscanf(fp3, "%x", &keylist[i][0]);	fscanf(fp3, "%x", &keylist[i][1]);	fscanf(fp3, "%x", &keylist[i][2]);	fscanf(fp3, "%x", &keylist[i][3]); }
	fclose(fp3);
	if (startingpoint) {
		fp4 = fopen("startingpoint.txt", "r");	for (i = 0; i < 100; i++)  { fscanf(fp4, "%I64d", &cumulative_bias[i]); printf("%d: %I64d\n",i,cumulative_bias[i]); }	fclose(fp4);
		for (i = 0; i < startingpoint; i++) { plaintext[0]++; plaintext[1]++; plaintext[2]++; plaintext[3]++; }
	}
	for (j = 0; j < pairs; j++){
		StartCounter();
		for (t = 0; t<experiment; t++) {
			hit = 0;
			key[0] = keylist[t][0]; key[1] = keylist[t][1]; key[2] = keylist[t][2]; key[3] = keylist[t][3];
			key_schedule();
//			for (i = 0; i < 65536; i++) hit_test[i] = 0;
			hipMemcpy(k_d, k, 132 * sizeof(bit32), hipMemcpyHostToDevice);
			hipMemcpy(hit_d, hit_test, 65536 * sizeof(bit64), hipMemcpyHostToDevice);
			if (numberofrounds == 4) Serpent4round << <blocksize, threadsize >> >(plaintext[0], plaintext[1], plaintext[2], plaintext[3], hit_d, k_d);
			else if (numberofrounds == 5) Serpent5round << <blocksize, threadsize >> >(plaintext[0], plaintext[1], plaintext[2], plaintext[3], hit_d, k_d);
			else if (numberofrounds == 6) Serpent6round_optimized << <blocksize, threadsize >> >(plaintext[0], plaintext[1], plaintext[2], plaintext[3], hit_d, k_d);
			else if (numberofrounds == 7) Serpent7round_optimized << <blocksize, threadsize >> >(plaintext[0], plaintext[1], plaintext[2], plaintext[3], hit_d, k_d);
			hipMemcpy(hit_test, hit_d, 65536 * sizeof(bit64), hipMemcpyDeviceToHost);
			for (i = 0; i < 65536; i++) hit += hit_test[i];
			cumulative_bias[t] += (hit - 34359738368);
//			printf("%2d: Key bias %I64d\n", t, cumulative_bias[t]);
			fprintf(fp,"%I64d\n",cumulative_bias[t]);
		}
		average_bias = 0;
		for (i = 0; i < 100; i++) average_bias += cumulative_bias[i];
		average_bias /= 100;
		printf("%5d: Time: %lf Average Bias %I64d\n", j+startingpoint, GetCounter(), average_bias);
		fprintf(fp, "%5d: Time: %lf Average Bias %I64d\n", j+startingpoint, GetCounter(), average_bias);
		if (numberofrounds == 4) fp2 = fopen("4round_cumulative_bias.txt", "ab");
		else if (numberofrounds == 5) fp2 = fopen("5round_cumulative_bias.txt", "ab");
		else if (numberofrounds == 6) fp2 = fopen("6round_cumulative_bias.txt", "ab");
		else if (numberofrounds == 7) fp2 = fopen("7round_cumulative_bias.txt", "ab");
		fprintf(fp2, "%5d: Time: %lf Average Bias %I64d\n", j+startingpoint, GetCounter(), average_bias);
		fclose(fp2);
		fp5 = fopen("6round_cumulative_bias_shortened.txt", "ab");
		fprintf(fp5, "%I64d\n",average_bias);
		fclose(fp5);
		plaintext[0]++; plaintext[1]++; plaintext[2]++; plaintext[3]++;
//		printf("%2d: Time: %lf ", t, GetCounter());	fprintf(fp, "%2d: Time: %lf ", t, GetCounter());
//		temp = blocksize*threadsize*loop*pairs/2;
//		bias += hit - temp;
//		printf("Hits: %I64d, Bias: %I64d\n", hit, hit - temp); fprintf(fp,"Hits: %I64d, Bias: %I64d\n", hit, hit - temp);
	}
//	bias = bias / experiment;
//	printf("Average Bias: %I64d\n",bias);
//	fprintf(fp,"Average Bias: %I64d\n", bias);
	hipFree(hit_d); hipFree(k_d);
	fclose(fp);   
//	system("PAUSE");
    return 0;
}
int main() {	
	FILE* fp3, * fp, * fp2, * fp4, * fp5;
	bit64* hit_d;
	bit64* hit_test;
	__int64 temp, hit, bias = 0, cumulative_bias[100] = { 0 }, average_bias;
	bit32* k_d, keylist[100][4];
//	bit64 hit_test[65536] = { 0 };
	hit_test = (bit64*)calloc(totalthreads, sizeof(bit64));
	int t, j, i;
	hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
	get_user_inputs();
	hipSetDevice(devicenumber);
	hipMalloc((void**)&hit_d, totalthreads * sizeof(bit64));
	hipMalloc((void**)&k_d, 132 * sizeof(bit32));
	if (numberofrounds == 3) fp = fopen("3round_cumulative.txt", "ab");
	else if (numberofrounds == 36) fp = fopen("36round_cumulative.txt", "ab");
	else if (numberofrounds == 4) fp = fopen("4round_cumulative.txt", "ab");
	else if (numberofrounds == 46) fp = fopen("46round_cumulative.txt", "ab");
	else if (numberofrounds == 466) fp = fopen("46bround_cumulative.txt", "ab");
	else if (numberofrounds == 5) fp = fopen("5round_cumulative.txt", "ab");
	else if (numberofrounds == 51) fp = fopen("51round_cumulative.txt", "ab");
	else if (numberofrounds == 56) fp = fopen("56round_cumulative.txt", "ab");
	else if (numberofrounds == 566) fp = fopen("566round_cumulative.txt", "ab");
	else if (numberofrounds == 6) fp = fopen("6round_cumulative.txt", "ab");
	else if (numberofrounds == 61) fp = fopen("61round_cumulative.txt", "ab");
	else if (numberofrounds == 66) fp = fopen("66round_cumulative.txt", "ab");
	else if (numberofrounds == 62) fp = fopen("62round_cumulative.txt", "ab");
	else if (numberofrounds == 63) fp = fopen("63round_cumulative.txt", "ab");
	else if (numberofrounds == 7) fp = fopen("7round_cumulative.txt", "ab");
	else if (numberofrounds == 76) fp = fopen("76round_cumulative.txt", "ab");
	else if (numberofrounds == 765) fp = fopen("765round_cumulative.txt", "ab");
	else if (numberofrounds == 73) fp = fopen("73round_cumulative.txt", "ab");
	else if (numberofrounds == 74) fp = fopen("74round_cumulative.txt", "ab");
	else if (numberofrounds == 75) fp = fopen("75round_cumulative.txt", "ab");
	else if (numberofrounds == 766) fp = fopen("766round_cumulative.txt", "ab");
	else if (numberofrounds == 732) fp = fopen("732round_cumulative.txt", "ab");
	else if (numberofrounds == 733) fp = fopen("733round_cumulative.txt", "ab");
	else if (numberofrounds == 32) fp = fopen("32round_benchmark.txt", "ab");
	else exit(1);
	fprintf(fp, "Pairs: %I64d\n", blocksize * threadsize * loop * pairs);
	//	plaintext[0] = 0xD095576F; plaintext[1] = 0xCEA3E3A7; plaintext[2] = 0xED98D9F2; plaintext[3] = 0x9073D78E;
	//	plaintext[0] = 0xf05e1f01; plaintext[1] = 0x3acc879d; plaintext[2] = 0xa3aa601c; plaintext[3] = 0xcc14000b;
	plaintext[0] = 0xc7675a6e; plaintext[1] = 0xe3d21628; plaintext[2] = 0x9f090c02; plaintext[3] = 0x5eeaf89d;
	fp3 = fopen("keys.txt", "r");
	for (i = 0; i < 100; i++) { fscanf(fp3, "%x", &keylist[i][0]);	fscanf(fp3, "%x", &keylist[i][1]);	fscanf(fp3, "%x", &keylist[i][2]);	fscanf(fp3, "%x", &keylist[i][3]); }
	fclose(fp3);
	if (startingpoint) {
		fp4 = fopen("startingpoint.txt", "r");	for (i = 0; i < experiment; i++) { fscanf(fp4, "%I64d", &cumulative_bias[i]); printf("%d: %I64d\n", i, cumulative_bias[i]); }	fclose(fp4);
		for (i = 0; i < startingpoint; i++) { plaintext[0]++; plaintext[1]++; plaintext[2]++; plaintext[3]++; }
	}
	for (j = 0; j < pairs; j++) {
		StartCounter();
		for (t = 0; t < experiment; t++) {
			hit = 0;
			key[0] = keylist[t][0]; key[1] = keylist[t][1]; key[2] = keylist[t][2]; key[3] = keylist[t][3];
			key_schedule();
			//			for (i = 0; i < 65536; i++) hit_test[i] = 0;
			hipMemcpy(k_d, k, 132 * sizeof(bit32), hipMemcpyHostToDevice);
			hipMemcpy(hit_d, hit_test, totalthreads * sizeof(bit64), hipMemcpyHostToDevice);
			
			if (numberofrounds == 3) Serpent3round_PZWD24 << <blocksize, threadsize >> > (plaintext[0], plaintext[1], plaintext[2], plaintext[3], hit_d, k_d);
			else if (numberofrounds == 36) Serpent3round_PZWD24_partof6 << <blocksize, threadsize >> > (plaintext[0], plaintext[1], plaintext[2], plaintext[3], hit_d, k_d);
			else if (numberofrounds == 4) Serpent4round << <blocksize, threadsize >> > (plaintext[0], plaintext[1], plaintext[2], plaintext[3], hit_d, k_d);
			else if (numberofrounds == 46) Serpent4round_PZWD24_partof6 << <blocksize, threadsize >> > (plaintext[0], plaintext[1], plaintext[2], plaintext[3], hit_d, k_d);
			else if (numberofrounds == 466) Serpent4round_PZWD24_first4of6 << <blocksize, threadsize >> > (plaintext[0], plaintext[1], plaintext[2], plaintext[3], hit_d, k_d);
			else if (numberofrounds == 5) Serpent5round_eprint << <blocksize, threadsize >> > (plaintext[0], plaintext[1], plaintext[2], plaintext[3], hit_d, k_d);
			else if (numberofrounds == 51) Serpent5round_PZWD24 << <blocksize, threadsize >> > (plaintext[0], plaintext[1], plaintext[2], plaintext[3], hit_d, k_d);
			else if (numberofrounds == 56) Serpent5round_PZWD24_partof6 << <blocksize, threadsize >> > (plaintext[0], plaintext[1], plaintext[2], plaintext[3], hit_d, k_d);
			else if (numberofrounds == 566) Serpent5round_PZWD24_second << <blocksize, threadsize >> > (plaintext[0], plaintext[1], plaintext[2], plaintext[3], hit_d, k_d);
			else if (numberofrounds == 61) Serpent6round_PZWD24 << <blocksize, threadsize >> > (plaintext[0], plaintext[1], plaintext[2], plaintext[3], hit_d, k_d);
			else if (numberofrounds == 66) Serpent6round_PZWD24_final << <blocksize, threadsize >> > (plaintext[0], plaintext[1], plaintext[2], plaintext[3], hit_d, k_d);
			else if (numberofrounds == 62) Serpent6round_PZWD24_second << <blocksize, threadsize >> > (plaintext[0], plaintext[1], plaintext[2], plaintext[3], hit_d, k_d);
			else if (numberofrounds == 63) Serpent6round_PZWD24_sanity << <blocksize, threadsize >> > (plaintext[0], plaintext[1], plaintext[2], plaintext[3], hit_d, k_d);
			else if (numberofrounds == 6) Serpent6round_optimized_eprint << <blocksize, threadsize >> > (plaintext[0], plaintext[1], plaintext[2], plaintext[3], hit_d, k_d);
			else if (numberofrounds == 7) Serpent7round_optimized << <blocksize, threadsize >> > (plaintext[0], plaintext[1], plaintext[2], plaintext[3], hit_d, k_d);
			else if (numberofrounds == 76) Serpent7round_PZWD24_last6 << <blocksize, threadsize >> > (plaintext[0], plaintext[1], plaintext[2], plaintext[3], hit_d, k_d);
			else if (numberofrounds == 765) Serpent7round_PZWD24_last6_maybe5 << <blocksize, threadsize >> > (plaintext[0], plaintext[1], plaintext[2], plaintext[3], hit_d, k_d);
			else if (numberofrounds == 73) Serpent7round_PZWD24_middle3 << <blocksize, threadsize >> > (plaintext[0], plaintext[1], plaintext[2], plaintext[3], hit_d, k_d);
			else if (numberofrounds == 732) Serpent7round_PZWD24_middle3_old << <blocksize, threadsize >> > (plaintext[0], plaintext[1], plaintext[2], plaintext[3], hit_d, k_d);
			else if (numberofrounds == 733) Serpent7round_PZWD24_middle3_final << <blocksize, threadsize >> > (plaintext[0], plaintext[1], plaintext[2], plaintext[3], hit_d, k_d);
			else if (numberofrounds == 74) Serpent7round_PZWD24_middle4 << <blocksize, threadsize >> > (plaintext[0], plaintext[1], plaintext[2], plaintext[3], hit_d, k_d);
			else if (numberofrounds == 75) Serpent7round_PZWD24_middle5 << <blocksize, threadsize >> > (plaintext[0], plaintext[1], plaintext[2], plaintext[3], hit_d, k_d);
			else if (numberofrounds == 766) Serpent7round_PZWD24_middle6 << <blocksize, threadsize >> > (plaintext[0], plaintext[1], plaintext[2], plaintext[3], hit_d, k_d);
			else if (numberofrounds == 32) {
				float milliseconds = 0;
				hipDeviceSynchronize();
				hipEvent_t start, stop;	hipEventCreate(&start);	hipEventCreate(&stop);	hipEventRecord(start);
				Serpent_benchmark << <blocksize, threadsize >> > (plaintext[0], plaintext[1], plaintext[2], plaintext[3], hit_d, k_d);
				hipEventRecord(stop);	hipEventSynchronize(stop);
				hipEventElapsedTime(&milliseconds, start, stop);	printf("Time elapsed: %f milliseconds\n", milliseconds);
			}
			hipMemcpy(hit_test, hit_d, totalthreads * sizeof(bit64), hipMemcpyDeviceToHost);
			for (i = 0; i < totalthreads; i++) hit += hit_test[i];
			cumulative_bias[t] += (hit - totalthreads*loop_d/2);
			//			printf("%2d: Key bias %I64d\n", t, cumulative_bias[t]);
			fprintf(fp, "%I64d\n", cumulative_bias[t]);
		}
		average_bias = 0;
		for (i = 0; i < experiment; i++) average_bias += cumulative_bias[i];
		average_bias /= experiment;
		printf("%5d: Time: %lf Average Bias %I64d\n", j + startingpoint, GetCounter(), average_bias);
		fprintf(fp, "%5d: Time: %lf Average Bias %I64d\n", j + startingpoint, GetCounter(), average_bias);
		if (numberofrounds == 3) fp2 = fopen("3round_cumulative_bias.txt", "ab");
		else if (numberofrounds == 36) fp2 = fopen("36round_cumulative_bias.txt", "ab");
		else if (numberofrounds == 4) fp2 = fopen("4round_cumulative_bias.txt", "ab");
		else if (numberofrounds == 46) fp2 = fopen("46round_cumulative_bias.txt", "ab");
		else if (numberofrounds == 466) fp2 = fopen("466round_cumulative_bias.txt", "ab");
		else if (numberofrounds == 5) fp2 = fopen("5round_cumulative_bias.txt", "ab");
		else if (numberofrounds == 51) fp2 = fopen("51round_cumulative_bias.txt", "ab");
		else if (numberofrounds == 56) fp2 = fopen("56round_cumulative_bias.txt", "ab");
		else if (numberofrounds == 566) fp2 = fopen("566round_cumulative_bias.txt", "ab");
		else if (numberofrounds == 61) fp2 = fopen("61round_cumulative_bias.txt", "ab");
		else if (numberofrounds == 62) fp2 = fopen("62round_cumulative_bias.txt", "ab");
		else if (numberofrounds == 63) fp2 = fopen("63round_cumulative_bias.txt", "ab");
		else if (numberofrounds == 6) fp2 = fopen("6round_cumulative_bias.txt", "ab");
		else if (numberofrounds == 66) fp2 = fopen("66round_cumulative_bias.txt", "ab");
		else if (numberofrounds == 7) fp2 = fopen("7round_cumulative_bias.txt", "ab");
		else if (numberofrounds == 76) fp2 = fopen("76round_cumulative_bias.txt", "ab");
		else if (numberofrounds == 765) fp2 = fopen("765round_cumulative_bias.txt", "ab");
		else if (numberofrounds == 73) fp2 = fopen("73round_cumulative_bias.txt", "ab");
		else if (numberofrounds == 74) fp2 = fopen("74round_cumulative_bias.txt", "ab");
		else if (numberofrounds == 75) fp2 = fopen("75round_cumulative_bias.txt", "ab");
		else if (numberofrounds == 766) fp2 = fopen("766round_cumulative_bias.txt", "ab");
		else if (numberofrounds == 732) fp2 = fopen("732round_cumulative_bias.txt", "ab");
		else if (numberofrounds == 733) fp2 = fopen("733round_cumulative_bias.txt", "ab");
		fprintf(fp2, "%5d: Time: %lf Average Bias %I64d\n", j + startingpoint, GetCounter(), average_bias);
		fclose(fp2);
		fp5 = fopen("6round_cumulative_bias_shortened.txt", "ab");
		fprintf(fp5, "%I64d\n", average_bias);
		fclose(fp5);
		plaintext[0]++; plaintext[1]++; plaintext[2]++; plaintext[3]++;
		//		printf("%2d: Time: %lf ", t, GetCounter());	fprintf(fp, "%2d: Time: %lf ", t, GetCounter());
		//		temp = blocksize*threadsize*loop*pairs/2;
		//		bias += hit - temp;
		//		printf("Hits: %I64d, Bias: %I64d\n", hit, hit - temp); fprintf(fp,"Hits: %I64d, Bias: %I64d\n", hit, hit - temp);
	}
	//	bias = bias / experiment;
	//	printf("Average Bias: %I64d\n",bias);
	//	fprintf(fp,"Average Bias: %I64d\n", bias);
	hipFree(hit_d); hipFree(k_d);
	fclose(fp);
	free(hit_test);
	printf("%s\n", hipGetErrorString(hipGetLastError()));
	//	system("PAUSE");
	return 0;
}

