#include "hip/hip_runtime.h"
// This code requires two files: keys.txt and startingpoint.txt
// Random 100 keys are stored in "keys.txt".
// Number of data of an experiment can be increased after an experiment is completed by storing the bias results in the "startingpoint.txt" file.
#include <Windows.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
typedef unsigned __int64 bit64;
typedef unsigned long bit32;
typedef unsigned char bit8;
#define ROL(x,r) ((x) = ((x) << (r)) | ((x) >> (32-(r))))
double PCFreq = 0.0;
__int64 CounterStart = 0;
__int64 blocksize = 512, threadsize = 1024, loop = 1024 * 256;
__int64 totalthreads = blocksize * threadsize;
//#define loop_d 128*1024
#define loop_d 1024*128
bit32 plaintext[4], ciphertext[4], key[8], w[132], k[132], phi = 0x9e3779b9;
bit32 x[4]; //plaintext words
int difference, devicenumber,userloop,pairs=1,numberofrounds, startingpoint=0;
float experiment=1; // This value should be an integer in [1,100] and determines the number of repeatition of the experiment with different keys
// key.txt contains 100 random keys (as 400 lines containing random 32-bit values in each line)

/* Serpent's S-boxes */
bit32 S[8][16] = {
	{ 0x3, 0x8, 0xf, 0x1, 0xa, 0x6, 0x5, 0xb, 0xe, 0xd, 0x4, 0x2, 0x7, 0x0, 0x9, 0xc },
	{ 0xf, 0xc, 0x2, 0x7, 0x9, 0x0, 0x5, 0xa, 0x1, 0xb, 0xe, 0x8, 0x6, 0xd, 0x3, 0x4 },
	{ 0x8, 0x6, 0x7, 0x9, 0x3, 0xc, 0xa, 0xf, 0xd, 0x1, 0xe, 0x4, 0x0, 0xb, 0x5, 0x2 },
	{ 0x0, 0xf, 0xb, 0x8, 0xc, 0x9, 0x6, 0x3, 0xd, 0x1, 0x2, 0x4, 0xa, 0x7, 0x5, 0xe },
	{ 0x1, 0xf, 0x8, 0x3, 0xc, 0x0, 0xb, 0x6, 0x2, 0x5, 0x4, 0xa, 0x9, 0xe, 0x7, 0xd },
	{ 0xf, 0x5, 0x2, 0xb, 0x4, 0xa, 0x9, 0xc, 0x0, 0x3, 0xe, 0x8, 0xd, 0x6, 0x7, 0x1 },
	{ 0x7, 0x2, 0xc, 0x5, 0x8, 0x4, 0x6, 0xb, 0xe, 0x9, 0x1, 0xf, 0xd, 0x3, 0xa, 0x0 },
	{ 0x1, 0xd, 0xf, 0x0, 0xe, 0x8, 0x2, 0xb, 0x7, 0x4, 0xc, 0xa, 0x9, 0x3, 0x5, 0x6 }
};
void StartCounter(){
	LARGE_INTEGER li;
	if (!QueryPerformanceFrequency(&li))
		printf("QueryPerformanceFrequency failed!\n");

	PCFreq = double(li.QuadPart) / 1000.0;

	QueryPerformanceCounter(&li);
	CounterStart = li.QuadPart;
}
double GetCounter(){
	LARGE_INTEGER li;
	QueryPerformanceCounter(&li);
	return double(li.QuadPart - CounterStart) / PCFreq;
}
void get_user_inputs() {
	key[4] = 0x80000000; key[5] = 0x00000000; key[6] = 0x00000000; key[7] = 0x00000000;
	printf("Select Cuda Device (0,1,2): ");
	scanf("%d", &devicenumber);
	printf("Choose an experiment:\n");
	printf(
		"(3) 3-round DL Experiment for [PZWD24]\n"
		"(32) SERPENT Benchmark\n"
		"(4) 4-round DL Experiment for our distinguisher (14 bit left rotation of [DIK08])\n"
		"(5) 5-round DL Experiment for [HDE24]\n"
		"(66) 6-round DL Experiment for [PZWD24] (final correction at eprint)\n"
		"(74) 4-round DL Experiment: Middle 4 rounds of 7-round DL of [PZWD24] \n"
		"(75) 5-round DL Experiment: Middle 5 rounds of 7-round DL of [PZWD24] \n"
		"(76) 6-round DL Experiment: Last 6 rounds of 7-round DL of [PZWD24] \n"
		"Choice: "
	);
	scanf("%d", &numberofrounds);
	printf("Number of Pairs: 2^36+");
	scanf("%d", &userloop);
	printf("Starting point (default 0): ");
	scanf("%d", &startingpoint);
	for (int i = 0; i<userloop; i++) pairs *= 2;
}
void key_s(int counter, int sbox) {
	int i, j;
	bit32 temp, x_temp[4];
	for (i = 0; i<4; i++) x_temp[i] = 0;
	for (i = 0; i<32; i++) {
		temp = 0;
		for (j = 0; j<4; j++) temp = temp | (((w[4 * counter + j] >> i) & 0x1) << j);
		temp = S[sbox][temp];
		for (j = 0; j<4; j++) x_temp[j] = x_temp[j] | (((temp >> j) & 0x1) << i);
	}
	for (i = 0; i<4; i++) k[4 * counter + i] = x_temp[i];
}
void key_schedule() {
	int i;
	for (i = 0; i<8; i++) w[i] = key[i];
	for (i = 8; i<16; i++) {
		w[i] = w[i - 8] ^ w[i - 5] ^ w[i - 3] ^ w[i - 1] ^ phi ^ (i - 8);
		w[i] = (w[i] << 11) | (w[i] >> 21);
	}
	for (i = 0; i<8; i++) w[i] = w[i + 8];
	// Generate w[i]'s
	for (i = 8; i<132; i++) {
		w[i] = (w[i - 8] ^ w[i - 5] ^ w[i - 3] ^ w[i - 1] ^ phi^i);
		// w[i] = w[i] <<< 11
		w[i] = (w[i] << 11) | (w[i] >> 21);
	}
	// Generate k[i]'s
	key_s(0, 3);
	key_s(1, 2);	key_s(2, 1);	key_s(3, 0);	key_s(4, 7);	key_s(5, 6);	key_s(6, 5);	key_s(7, 4);
	key_s(8, 3);	key_s(9, 2);	key_s(10, 1); key_s(11, 0); key_s(12, 7); key_s(13, 6); key_s(14, 5); key_s(15, 4);
	key_s(16, 3); key_s(17, 2); key_s(18, 1); key_s(19, 0); key_s(20, 7); key_s(21, 6); key_s(22, 5); key_s(23, 4);
	key_s(24, 3); key_s(25, 2); key_s(26, 1); key_s(27, 0); key_s(28, 7); key_s(29, 6); key_s(30, 5); key_s(31, 4); key_s(32, 3);
}
void s(int sbox) {
	int i, j;
	bit32 temp, x_temp[4];
	for (i = 0; i<4; i++) x_temp[i] = 0;
	for (i = 0; i<32; i++) {
		temp = 0;
		for (j = 0; j<4; j++) temp = temp | (((x[j] >> i) & 0x1) << j);
		temp = S[sbox][temp];
		for (j = 0; j<4; j++) x_temp[j] = x_temp[j] | (((temp >> j) & 0x1) << i);
	}
	for (i = 0; i<4; i++) x[i] = x_temp[i];
}
void Sb0(bit32 a, bit32 b, bit32 c, bit32 d) {
	bit32 t1, t3, t4, t7, t12;
	t1 = a ^ d;
	t3 = c ^ t1;
	t4 = b ^ t3;
	x[3] = (a & d) ^ t4;
	t7 = a ^ (b & t1);
	x[2] = t4 ^ (c | t7);
	t12 = x[3] & (t3 ^ t7);
	x[1] = (~t3) ^ t12;
	x[0] = t12 ^ (~t7);
}
void Sb1(bit32 a, bit32 b, bit32 c, bit32 d) {
	bit32 t2, t5, t7, t8, t11;
	t2 = b ^ (~a);
	t5 = c ^ (a | t2);
	x[2] = d ^ t5;
	t7 = b ^ (d | t2);
	t8 = t2 ^ x[2];
	x[3] = t8 ^ (t5 & t7);
	t11 = t5 ^ t7;
	x[1] = x[3] ^ t11;
	x[0] = t5 ^ (t8 & t11);
}
void Sb2(bit32 a, bit32 b, bit32 c, bit32 d)   {
	bit32 t1, t2, t3, t5, t6, t7;
	t1 = ~a;
	t2 = b ^ d;
	t3 = c & t1;
	x[0] = t2 ^ t3;
	t5 = c ^ t1;
	t6 = c ^ x[0];
	t7 = b & t6;
	x[3] = t5 ^ t7;
	x[2] = a ^ ((d | t7) & (x[0] | t5));
	x[1] = (t2 ^ x[3]) ^ (x[2] ^ (d | t1));
}
void Sb3(bit32 a, bit32 b, bit32 c, bit32 d) {
	bit32 t1, t2, t3, t4, t5, t6, t8, t9, t10, t12;
	t1 = a ^ b;
	t2 = a & c;
	t3 = a | d;
	t4 = c ^ d;
	t5 = t1 & t3;
	t6 = t2 | t5;
	x[2] = t4 ^ t6;
	t8 = b ^ t3;
	t9 = t6 ^ t8;
	t10 = t4 & t9;
	x[0] = t1 ^ t10;
	t12 = x[2] & x[0];
	x[1] = t9 ^ t12;
	x[3] = (b | d) ^ (t4 ^ t12);
}
void Sb4(bit32 a, bit32 b, bit32 c, bit32 d){
	bit32 t1, t2, t3, t4, t6, t7, t9, t10, t11;
	t1 = a ^ d;
	t2 = d & t1;
	t3 = c ^ t2;
	t4 = b | t3;
	x[3] = t1 ^ t4;
	t6 = ~b;
	t7 = t1 | t6;
	x[0] = t3 ^ t7;
	t9 = a & x[0];
	t10 = t1 ^ t6;
	t11 = t4 & t10;
	x[2] = t9 ^ t11;
	x[1] = (a ^ t3) ^ (t10 & x[2]);
}
void Sb5(bit32 a, bit32 b, bit32 c, bit32 d) {
	bit32 t1, t2, t3, t4, t5, t7, t8, t10, t11, t12;
	t1 = ~a;
	t2 = a ^ b;
	t3 = a ^ d;
	t4 = c ^ t1;
	t5 = t2 | t3;
	x[0] = t4 ^ t5;
	t7 = d & x[0];
	t8 = t2 ^ x[0];
	x[1] = t7 ^ t8;
	t10 = t1 | x[0];
	t11 = t2 | t7;
	t12 = t3 ^ t10;
	x[2] = t11 ^ t12;
	x[3] = (b ^ t7) ^ (x[1] & t12);
}
void Sb6(bit32 a, bit32 b, bit32 c, bit32 d) {
	bit32 t1, t2, t3, t4, t5, t7, t8, t9, t11;
	t1 = ~a;
	t2 = a ^ d;
	t3 = b ^ t2;
	t4 = t1 | t2;
	t5 = c ^ t4;
	x[1] = b ^ t5;
	t7 = t2 | x[1];
	t8 = d ^ t7;
	t9 = t5 & t8;
	x[2] = t3 ^ t9;
	t11 = t5 ^ t8;
	x[0] = x[2] ^ t11;
	x[3] = (~t5) ^ (t3 & t11);
}
void Sb7(bit32 a, bit32 b, bit32 c, bit32 d) {
	bit32 t1, t2, t3, t4, t5, t6, t8, t9, t11, t12;
	t1 = b ^ c;
	t2 = c & t1;
	t3 = d ^ t2;
	t4 = a ^ t3;
	t5 = d | t1;
	t6 = t4 & t5;
	x[1] = b ^ t6;
	t8 = t3 | x[1];
	t9 = a & t4;
	x[3] = t1 ^ t9;
	t11 = t4 ^ t8;
	t12 = x[3] & t11;
	x[2] = t3 ^ t12;
	x[0] = (~t11) ^ (x[3] & x[2]);
}
void key_addition(int round) {	for (int i = 0; i<4; i++) x[i] = x[i] ^ k[4 * round + i];}
void linear_transformation() {
	// x0 = x0 <<< 13
	x[0] = (x[0] << 13) | x[0] >> 19;
	// x2 <<< 3
	x[2] = (x[2] << 3) | (x[2] >> 29);
	// x1 = x1^x0^x2
	x[1] = x[1] ^ x[0] ^ x[2];
	// x3 = x3^x2^(x0 << 3)
	x[3] = x[3] ^ x[2] ^ (x[0] << 3);
	// x1 = x1 <<< 1
	x[1] = (x[1] << 1) | (x[1] >> 31);
	// x3 = x3 <<< 7
	x[3] = (x[3] << 7) | (x[3] >> 25);
	// x0 = x0 ^ x1 ^ x3
	x[0] = x[0] ^ x[1] ^ x[3];
	// x2 = x2 ^ x3 ^ (x1 << 7)
	x[2] = x[2] ^ x[3] ^ (x[1] << 7);
	// x0 = x0 <<< 5
	x[0] = (x[0] << 5) | (x[0] >> 27);
	// x2 = x2 <<< 22
	x[2] = (x[2] << 22) | (x[2] >> 10);
}
__global__ void Serpent4round(bit32 p0, bit32 p1, bit32 p2, bit32 p3, bit64 *hit, bit32 *k_d) {
	__shared__ bit32  k[132];
	bit32 x0, x1, x2, x3, plaintext0 = p0, plaintext1 = p1, plaintext2 = p2, plaintext3 = p3 + blockIdx.x *blockDim.x + threadIdx.x;
	bit32 a, b, c, d;
	bit32 ciphertext0, ciphertext1, ciphertext2, ciphertext3;
	bit32 t1, t2, t3, t4, t5, t6, t7, t8, t9, t10, t11, t12;
	bit32 round;
	bit64 counter = 0;
	if (threadIdx.x<132) k[threadIdx.x] = k_d[threadIdx.x];
	__syncthreads();
	for (int i = 0; i < loop_d; i++) {
		x0 = plaintext0; x1 = plaintext1; x2 = plaintext2; x3 = plaintext3;
		// Round 2
		round = 2;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = b ^ d; t3 = c & t1; x0 = t2 ^ t3; t5 = c ^ t1; t6 = c ^ x0; t7 = b & t6; x3 = t5 ^ t7; x2 = a ^ ((d | t7) & (x0 | t5)); x1 = (t2 ^ x3) ^ (x2 ^ (d | t1));
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 3
		round = 3;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ b; t2 = a & c; t3 = a | d; t4 = c ^ d; t5 = t1 & t3; t6 = t2 | t5; x2 = t4 ^ t6; t8 = b ^ t3; t9 = t6 ^ t8; t10 = t4 & t9; x0 = t1 ^ t10; t12 = x2 & x0; x1 = t9 ^ t12; x3 = (b | d) ^ (t4 ^ t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 4
		round = 4;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 5
		round = 5;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		ciphertext0 = x0; ciphertext1 = x1; ciphertext2 = x2; ciphertext3 = x3;

		// Give difference
		x0 = plaintext0 ^ (0x00000010<<14);
		x1 = plaintext1;
		x2 = plaintext2 ^ (0x00000090<<14);
		x3 = plaintext3;
		// Round 2
		round = 2;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = b ^ d; t3 = c & t1; x0 = t2 ^ t3; t5 = c ^ t1; t6 = c ^ x0; t7 = b & t6; x3 = t5 ^ t7; x2 = a ^ ((d | t7) & (x0 | t5)); x1 = (t2 ^ x3) ^ (x2 ^ (d | t1));
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 3
		round = 3;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ b; t2 = a & c; t3 = a | d; t4 = c ^ d; t5 = t1 & t3; t6 = t2 | t5; x2 = t4 ^ t6; t8 = b ^ t3; t9 = t6 ^ t8; t10 = t4 & t9; x0 = t1 ^ t10; t12 = x2 & x0; x1 = t9 ^ t12; x3 = (b | d) ^ (t4 ^ t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 4
		round = 4;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 5
		round = 5;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);

		// Check Difference
		x2 ^= ciphertext2;
		x3 ^= ciphertext3;
		t1 = (x2 >> 29) & 0x1;
		t2 = x3 & 0x1;
		t1 ^= t2;
		if (t1 == 0) counter++;

		// Change plaintext
		plaintext0 = x0; plaintext1 = x1; plaintext2 = x2; plaintext3 = x3;
	}
	hit[blockIdx.x *blockDim.x + threadIdx.x] = counter;
}
__global__ void Serpent3round_PZWD24(bit32 p0, bit32 p1, bit32 p2, bit32 p3, bit64* hit, bit32* k_d) {
	__shared__ bit32  k[132];
	bit32 x0, x1, x2, x3, plaintext0 = p0, plaintext1 = p1, plaintext2 = p2, plaintext3 = p3 + blockIdx.x * blockDim.x + threadIdx.x;
	bit32 a, b, c, d;
	bit32 ciphertext0, ciphertext1, ciphertext2, ciphertext3;
	bit32 t0, t1, t2, t3, t4, t5, t6, t7, t8, t9, t10, t11, t12;
	bit32 round;
	bit64 counter = 0;
	if (threadIdx.x < 132) k[threadIdx.x] = k_d[threadIdx.x];
	__syncthreads();
	for (int i = 0; i < loop_d; i++) {
		x0 = plaintext0; x1 = plaintext1; x2 = plaintext2; x3 = plaintext3;
		// Round 5
		round = 5;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 6
		round = 6;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 7
		round = 7;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = b ^ c; t2 = c & t1; t3 = d ^ t2; t4 = a ^ t3; t5 = d | t1; t6 = t4 & t5; x1 = b ^ t6; t8 = t3 | x1; t9 = a & t4; x3 = t1 ^ t9; t11 = t4 ^ t8; t12 = x3 & t11; x2 = t3 ^ t12; x0 = (~t11) ^ (x3 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);

		// Give difference
		ciphertext0 = x0; ciphertext1 = x1; ciphertext2 = x2; ciphertext3 = x3;		
		x0 = plaintext0 ^ 0x01000000;
		x1 = plaintext1 ^ 0x01000000;
		x2 = plaintext2 ^ 0x00000000;
		x3 = plaintext3 ^ 0x00000000;

		// Round 5
		round = 5;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 6
		round = 6;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 7
		round = 7;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = b ^ c; t2 = c & t1; t3 = d ^ t2; t4 = a ^ t3; t5 = d | t1; t6 = t4 & t5; x1 = b ^ t6; t8 = t3 | x1; t9 = a & t4; x3 = t1 ^ t9; t11 = t4 ^ t8; t12 = x3 & t11; x2 = t3 ^ t12; x0 = (~t11) ^ (x3 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);

		// Check Difference
/*		t1 = x0 ^ ciphertext0;
		t2 = x1 ^ ciphertext1;
		t3 = x2 ^ ciphertext2;
		t4 = x3 ^ ciphertext3;

		bit32 result = 0;
		t1 = t1 & 0x08010812;
		t2 = t2 & 0xa00a2004;
		t3 = t3 & 0x00000010;
		t4 = t4 & 0x000a0000;
		for (int i = 0; i < 32; i++) {
			result ^= ((t1 >> i) & 0x1);
			result ^= ((t2 >> i) & 0x1);
			result ^= ((t3 >> i) & 0x1);
			result ^= ((t4 >> i) & 0x1);
		}
		if (result == 0) counter++;*/

/*		t0 = x0 & 0x44900010;
		t1 = x1 & 0x00010000;
		t2 = x2 & 0x01980010;
		t3 = x3 & 0x12000200;*/
		x0 ^= ciphertext0;
		x1 ^= ciphertext1;
		x2 ^= ciphertext2;
		x3 ^= ciphertext3;

		t0 = x0 & 0x12000200;
		t1 = x1 & 0x01980010;
		t2 = x2 & 0x00010000;
		t3 = x3 & 0x44900010;

		t0 ^= t0 >> 1;
		t0 ^= t0 >> 2;
		t0 = (t0 & 0x11111111U) * 0x11111111U;
		t0 = (t0 >> 28) & 1;

		t1 ^= t1 >> 1;
		t1 ^= t1 >> 2;
		t1 = (t1 & 0x11111111U) * 0x11111111U;
		t1 = (t1 >> 28) & 1;

		t2 ^= t2 >> 1;
		t2 ^= t2 >> 2;
		t2 = (t2 & 0x11111111U) * 0x11111111U;
		t2 = (t2 >> 28) & 1;

		t3 ^= t3 >> 1;
		t3 ^= t3 >> 2;
		t3 = (t3 & 0x11111111U) * 0x11111111U;
		t3 = (t3 >> 28) & 1;

		t0 = t0 ^ t1 ^ t2 ^ t3;

		if (t0 == 0) counter++;

		// Change plaintext
		plaintext0 = x0; plaintext1 = x1; plaintext2 = x2; plaintext3 = x3;
	}
	hit[blockIdx.x * blockDim.x + threadIdx.x] = counter;
}
__global__ void Serpent6round_PZWD24_final(bit32 p0, bit32 p1, bit32 p2, bit32 p3, bit64* hit, bit32* k_d) {
	__shared__ bit32  k[132];
	bit32 x0, x1, x2, x3, plaintext0 = p0, plaintext1 = p1, plaintext2 = p2, plaintext3 = p3 + blockIdx.x * blockDim.x + threadIdx.x;
	bit32 a, b, c, d;
	bit32 ciphertext0, ciphertext1, ciphertext2, ciphertext3;
	bit32 t0, t1, t2, t3, t4, t5, t6, t7, t8, t9, t10, t11, t12;
	bit32 round;
	bit64 counter = 0;
	if (threadIdx.x < 132) k[threadIdx.x] = k_d[threadIdx.x];
	__syncthreads();
	for (int i = 0; i < loop_d; i++) {
		x0 = plaintext0; x1 = plaintext1; x2 = plaintext2; x3 = plaintext3;
		// Round 4
		x0 ^= k[16];	x1 ^= k[17];	x2 ^= k[18];	x3 ^= k[19];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 5
		round = 5;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 6
		round = 6;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 7
		round = 7;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = b ^ c; t2 = c & t1; t3 = d ^ t2; t4 = a ^ t3; t5 = d | t1; t6 = t4 & t5; x1 = b ^ t6; t8 = t3 | x1; t9 = a & t4; x3 = t1 ^ t9; t11 = t4 ^ t8; t12 = x3 & t11; x2 = t3 ^ t12; x0 = (~t11) ^ (x3 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 8
		x0 ^= k[32];	x1 ^= k[33];	x2 ^= k[34];	x3 ^= k[35];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t3 = c ^ t1; t4 = b ^ t3; x3 = (a & d) ^ t4; t7 = a ^ (b & t1); x2 = t4 ^ (c | t7); t12 = x3 & (t3 ^ t7); x1 = (~t3) ^ t12; x0 = t12 ^ (~t7);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 9
		x0 ^= k[36];	x1 ^= k[37];	x2 ^= k[38];	x3 ^= k[39];
		a = x0; b = x1; c = x2; d = x3;
		t2 = b ^ (~a);		t5 = c ^ (a | t2);		x2 = d ^ t5;		t7 = b ^ (d | t2);		t8 = t2 ^ x2;		x3 = t8 ^ (t5 & t7); 		t11 = t5 ^ t7; 		x1 = x3 ^ t11; 		x0 = t5 ^ (t8 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);

		// Give difference
		ciphertext0 = x0; ciphertext1 = x1; ciphertext2 = x2; ciphertext3 = x3;
		x0 = plaintext0 ^ 0x20010000;
		x1 = plaintext1 ^ 0x00010000;
		x2 = plaintext2 ^ 0x00000000;
		x3 = plaintext3 ^ 0x20010000;
		//20010000000000000001000020010000

		// Round 4
		x0 ^= k[16];	x1 ^= k[17];	x2 ^= k[18];	x3 ^= k[19];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 5
		round = 5;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 6
		round = 6;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 7
		round = 7;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = b ^ c; t2 = c & t1; t3 = d ^ t2; t4 = a ^ t3; t5 = d | t1; t6 = t4 & t5; x1 = b ^ t6; t8 = t3 | x1; t9 = a & t4; x3 = t1 ^ t9; t11 = t4 ^ t8; t12 = x3 & t11; x2 = t3 ^ t12; x0 = (~t11) ^ (x3 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 8
		x0 ^= k[32];	x1 ^= k[33];	x2 ^= k[34];	x3 ^= k[35];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t3 = c ^ t1; t4 = b ^ t3; x3 = (a & d) ^ t4; t7 = a ^ (b & t1); x2 = t4 ^ (c | t7); t12 = x3 & (t3 ^ t7); x1 = (~t3) ^ t12; x0 = t12 ^ (~t7);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 9
		x0 ^= k[36];	x1 ^= k[37];	x2 ^= k[38];	x3 ^= k[39];
		a = x0; b = x1; c = x2; d = x3;
		t2 = b ^ (~a);		t5 = c ^ (a | t2);		x2 = d ^ t5;		t7 = b ^ (d | t2);		t8 = t2 ^ x2;		x3 = t8 ^ (t5 & t7); 		t11 = t5 ^ t7; 		x1 = x3 ^ t11; 		x0 = t5 ^ (t8 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);


		x0 ^= ciphertext0;
		x1 ^= ciphertext1;
		x2 ^= ciphertext2;
		x3 ^= ciphertext3;

		t0 = x0 & 0x20108480;
		t1 = x1 & 0x0302842c;
		t2 = x2 & 0x01004000;
		t3 = x3 & 0x00008420;
		// 
		// corrected: 0010002600010020b4000b0010a02200 => 00008420010040000302842c20108480

		t0 ^= t0 >> 1;
		t0 ^= t0 >> 2;
		t0 = (t0 & 0x11111111U) * 0x11111111U;
		t0 = (t0 >> 28) & 1;

		t1 ^= t1 >> 1;
		t1 ^= t1 >> 2;
		t1 = (t1 & 0x11111111U) * 0x11111111U;
		t1 = (t1 >> 28) & 1;

		t2 ^= t2 >> 1;
		t2 ^= t2 >> 2;
		t2 = (t2 & 0x11111111U) * 0x11111111U;
		t2 = (t2 >> 28) & 1;

		t3 ^= t3 >> 1;
		t3 ^= t3 >> 2;
		t3 = (t3 & 0x11111111U) * 0x11111111U;
		t3 = (t3 >> 28) & 1;

		t0 = t0 ^ t1 ^ t2 ^ t3;

		if (t0 == 0) counter++;

		// Change plaintext
		plaintext0 = x0; plaintext1 = x1; plaintext2 = x2; plaintext3 = x3;
	}
	hit[blockIdx.x * blockDim.x + threadIdx.x] = counter;
}
__global__ void Serpent7round_PZWD24_middle4(bit32 p0, bit32 p1, bit32 p2, bit32 p3, bit64* hit, bit32* k_d) {
	__shared__ bit32  k[132];
	bit32 x0, x1, x2, x3, plaintext0 = p0, plaintext1 = p1, plaintext2 = p2, plaintext3 = p3 + blockIdx.x * blockDim.x + threadIdx.x;
	bit32 a, b, c, d;
	bit32 ciphertext0, ciphertext1, ciphertext2, ciphertext3;
	bit32 t0, t1, t2, t3, t4, t5, t6, t7, t8, t9, t10, t11, t12;
	bit32 round;
	bit64 counter = 0;
	if (threadIdx.x < 132) k[threadIdx.x] = k_d[threadIdx.x];
	__syncthreads();
	for (int i = 0; i < loop_d; i++) {
		x0 = plaintext0; x1 = plaintext1; x2 = plaintext2; x3 = plaintext3;
		// Round 2
		x0 ^= k[8];	x1 ^= k[9];	x2 ^= k[19];	x3 ^= k[11];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = b ^ d; t3 = c & t1; x0 = t2 ^ t3; t5 = c ^ t1; t6 = c ^ x0; t7 = b & t6; x3 = t5 ^ t7; x2 = a ^ ((d | t7) & (x0 | t5)); x1 = (t2 ^ x3) ^ (x2 ^ (d | t1));
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 3
		x0 ^= k[12];	x1 ^= k[13];	x2 ^= k[14];	x3 ^= k[15];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ b; t2 = a & c; t3 = a | d; t4 = c ^ d; t5 = t1 & t3; t6 = t2 | t5; x2 = t4 ^ t6; t8 = b ^ t3; t9 = t6 ^ t8; t10 = t4 & t9; x0 = t1 ^ t10; t12 = x2 & x0; x1 = t9 ^ t12; x3 = (b | d) ^ (t4 ^ t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 4
		x0 ^= k[16];	x1 ^= k[17];	x2 ^= k[18];	x3 ^= k[19];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 5
		x0 ^= k[20];	x1 ^= k[21];	x2 ^= k[22];	x3 ^= k[23];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);

		// Give difference
		ciphertext0 = x0; ciphertext1 = x1; ciphertext2 = x2; ciphertext3 = x3;
		x0 = plaintext0 ^ 0x00000000;
		x1 = plaintext1 ^ 0x00000000;
		x2 = plaintext2 ^ 0x00010000;
		x3 = plaintext3 ^ 0x00000000;
		//00000000000100000000000000000000

		// Round 2
		x0 ^= k[8];	x1 ^= k[9];	x2 ^= k[19];	x3 ^= k[11];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = b ^ d; t3 = c & t1; x0 = t2 ^ t3; t5 = c ^ t1; t6 = c ^ x0; t7 = b & t6; x3 = t5 ^ t7; x2 = a ^ ((d | t7) & (x0 | t5)); x1 = (t2 ^ x3) ^ (x2 ^ (d | t1));
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 3
		x0 ^= k[12];	x1 ^= k[13];	x2 ^= k[14];	x3 ^= k[15];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ b; t2 = a & c; t3 = a | d; t4 = c ^ d; t5 = t1 & t3; t6 = t2 | t5; x2 = t4 ^ t6; t8 = b ^ t3; t9 = t6 ^ t8; t10 = t4 & t9; x0 = t1 ^ t10; t12 = x2 & x0; x1 = t9 ^ t12; x3 = (b | d) ^ (t4 ^ t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 4
		x0 ^= k[16];	x1 ^= k[17];	x2 ^= k[18];	x3 ^= k[19];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 5
		x0 ^= k[20];	x1 ^= k[21];	x2 ^= k[22];	x3 ^= k[23];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);


		x0 ^= ciphertext0;
		x1 ^= ciphertext1;
		x2 ^= ciphertext2;
		x3 ^= ciphertext3;

		t0 = x0 & 0x00000000;
		t1 = x1 & 0x00000000;
		t2 = x2 & 0x00000000;
		t3 = x3 & 0x00000100;
		//00000100000000000000000000000000


		t0 ^= t0 >> 1;
		t0 ^= t0 >> 2;
		t0 = (t0 & 0x11111111U) * 0x11111111U;
		t0 = (t0 >> 28) & 1;

		t1 ^= t1 >> 1;
		t1 ^= t1 >> 2;
		t1 = (t1 & 0x11111111U) * 0x11111111U;
		t1 = (t1 >> 28) & 1;

		t2 ^= t2 >> 1;
		t2 ^= t2 >> 2;
		t2 = (t2 & 0x11111111U) * 0x11111111U;
		t2 = (t2 >> 28) & 1;

		t3 ^= t3 >> 1;
		t3 ^= t3 >> 2;
		t3 = (t3 & 0x11111111U) * 0x11111111U;
		t3 = (t3 >> 28) & 1;

		t0 = t0 ^ t1 ^ t2 ^ t3;

		if (t0 == 0) counter++;

		// Change plaintext
		plaintext0 = x0; plaintext1 = x1; plaintext2 = x2; plaintext3 = x3;
	}
	hit[blockIdx.x * blockDim.x + threadIdx.x] = counter;
}
__global__ void Serpent7round_PZWD24_middle5(bit32 p0, bit32 p1, bit32 p2, bit32 p3, bit64* hit, bit32* k_d) {
	__shared__ bit32  k[132];
	bit32 x0, x1, x2, x3, plaintext0 = p0, plaintext1 = p1, plaintext2 = p2, plaintext3 = p3 + blockIdx.x * blockDim.x + threadIdx.x;
	bit32 a, b, c, d;
	bit32 ciphertext0, ciphertext1, ciphertext2, ciphertext3;
	bit32 t0, t1, t2, t3, t4, t5, t6, t7, t8, t9, t10, t11, t12;
	bit32 round;
	bit64 counter = 0;
	if (threadIdx.x < 132) k[threadIdx.x] = k_d[threadIdx.x];
	__syncthreads();
	for (int i = 0; i < loop_d; i++) {
		x0 = plaintext0; x1 = plaintext1; x2 = plaintext2; x3 = plaintext3;
		// Round 2
		x0 ^= k[8];	x1 ^= k[9];	x2 ^= k[19];	x3 ^= k[11];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = b ^ d; t3 = c & t1; x0 = t2 ^ t3; t5 = c ^ t1; t6 = c ^ x0; t7 = b & t6; x3 = t5 ^ t7; x2 = a ^ ((d | t7) & (x0 | t5)); x1 = (t2 ^ x3) ^ (x2 ^ (d | t1));
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 3
		x0 ^= k[12];	x1 ^= k[13];	x2 ^= k[14];	x3 ^= k[15];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ b; t2 = a & c; t3 = a | d; t4 = c ^ d; t5 = t1 & t3; t6 = t2 | t5; x2 = t4 ^ t6; t8 = b ^ t3; t9 = t6 ^ t8; t10 = t4 & t9; x0 = t1 ^ t10; t12 = x2 & x0; x1 = t9 ^ t12; x3 = (b | d) ^ (t4 ^ t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 4
		x0 ^= k[16];	x1 ^= k[17];	x2 ^= k[18];	x3 ^= k[19];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 5
		x0 ^= k[20];	x1 ^= k[21];	x2 ^= k[22];	x3 ^= k[23];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 6
		x0 ^= k[24];	x1 ^= k[25];	x2 ^= k[26];	x3 ^= k[27];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);

		// Give difference
		ciphertext0 = x0; ciphertext1 = x1; ciphertext2 = x2; ciphertext3 = x3;
		x0 = plaintext0 ^ 0x00000000;
		x1 = plaintext1 ^ 0x00000000;
		x2 = plaintext2 ^ 0x00010000;
		x3 = plaintext3 ^ 0x00000000;
		//00000000000100000000000000000000

		// Round 2
		x0 ^= k[8];	x1 ^= k[9];	x2 ^= k[19];	x3 ^= k[11];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = b ^ d; t3 = c & t1; x0 = t2 ^ t3; t5 = c ^ t1; t6 = c ^ x0; t7 = b & t6; x3 = t5 ^ t7; x2 = a ^ ((d | t7) & (x0 | t5)); x1 = (t2 ^ x3) ^ (x2 ^ (d | t1));
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 3
		x0 ^= k[12];	x1 ^= k[13];	x2 ^= k[14];	x3 ^= k[15];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ b; t2 = a & c; t3 = a | d; t4 = c ^ d; t5 = t1 & t3; t6 = t2 | t5; x2 = t4 ^ t6; t8 = b ^ t3; t9 = t6 ^ t8; t10 = t4 & t9; x0 = t1 ^ t10; t12 = x2 & x0; x1 = t9 ^ t12; x3 = (b | d) ^ (t4 ^ t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 4
		x0 ^= k[16];	x1 ^= k[17];	x2 ^= k[18];	x3 ^= k[19];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 5
		x0 ^= k[20];	x1 ^= k[21];	x2 ^= k[22];	x3 ^= k[23];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 6
		x0 ^= k[24];	x1 ^= k[25];	x2 ^= k[26];	x3 ^= k[27];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);


		x0 ^= ciphertext0;
		x1 ^= ciphertext1;
		x2 ^= ciphertext2;
		x3 ^= ciphertext3;

		t0 = x0 & 0x04000000;
		t1 = x1 & 0x00200000;
		t2 = x2 & 0x00000000;
		t3 = x3 & 0x00200000;
		//00200000000000000020000004000000

		t0 ^= t0 >> 1;
		t0 ^= t0 >> 2;
		t0 = (t0 & 0x11111111U) * 0x11111111U;
		t0 = (t0 >> 28) & 1;

		t1 ^= t1 >> 1;
		t1 ^= t1 >> 2;
		t1 = (t1 & 0x11111111U) * 0x11111111U;
		t1 = (t1 >> 28) & 1;

		t2 ^= t2 >> 1;
		t2 ^= t2 >> 2;
		t2 = (t2 & 0x11111111U) * 0x11111111U;
		t2 = (t2 >> 28) & 1;

		t3 ^= t3 >> 1;
		t3 ^= t3 >> 2;
		t3 = (t3 & 0x11111111U) * 0x11111111U;
		t3 = (t3 >> 28) & 1;

		t0 = t0 ^ t1 ^ t2 ^ t3;

		if (t0 == 0) counter++;

		// Change plaintext
		plaintext0 = x0; plaintext1 = x1; plaintext2 = x2; plaintext3 = x3;
	}
	hit[blockIdx.x * blockDim.x + threadIdx.x] = counter;
}
__global__ void Serpent7round_PZWD24_middle6(bit32 p0, bit32 p1, bit32 p2, bit32 p3, bit64* hit, bit32* k_d) {
	__shared__ bit32  k[132];
	bit32 x0, x1, x2, x3, plaintext0 = p0, plaintext1 = p1, plaintext2 = p2, plaintext3 = p3 + blockIdx.x * blockDim.x + threadIdx.x;
	bit32 a, b, c, d;
	bit32 ciphertext0, ciphertext1, ciphertext2, ciphertext3;
	bit32 t0, t1, t2, t3, t4, t5, t6, t7, t8, t9, t10, t11, t12;
	bit32 round;
	bit64 counter = 0;
	if (threadIdx.x < 132) k[threadIdx.x] = k_d[threadIdx.x];
	__syncthreads();
	for (int i = 0; i < loop_d; i++) {
		x0 = plaintext0; x1 = plaintext1; x2 = plaintext2; x3 = plaintext3;
		// Round 2
		x0 ^= k[8];	x1 ^= k[9];	x2 ^= k[19];	x3 ^= k[11];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = b ^ d; t3 = c & t1; x0 = t2 ^ t3; t5 = c ^ t1; t6 = c ^ x0; t7 = b & t6; x3 = t5 ^ t7; x2 = a ^ ((d | t7) & (x0 | t5)); x1 = (t2 ^ x3) ^ (x2 ^ (d | t1));
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 3
		x0 ^= k[12];	x1 ^= k[13];	x2 ^= k[14];	x3 ^= k[15];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ b; t2 = a & c; t3 = a | d; t4 = c ^ d; t5 = t1 & t3; t6 = t2 | t5; x2 = t4 ^ t6; t8 = b ^ t3; t9 = t6 ^ t8; t10 = t4 & t9; x0 = t1 ^ t10; t12 = x2 & x0; x1 = t9 ^ t12; x3 = (b | d) ^ (t4 ^ t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 4
		x0 ^= k[16];	x1 ^= k[17];	x2 ^= k[18];	x3 ^= k[19];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 5
		x0 ^= k[20];	x1 ^= k[21];	x2 ^= k[22];	x3 ^= k[23];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 6
		x0 ^= k[24];	x1 ^= k[25];	x2 ^= k[26];	x3 ^= k[27];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 7
		x0 ^= k[28];	x1 ^= k[29];	x2 ^= k[30];	x3 ^= k[31];
		a = x0; b = x1; c = x2; d = x3;
		t1 = b ^ c; t2 = c & t1; t3 = d ^ t2; t4 = a ^ t3; t5 = d | t1; t6 = t4 & t5; x1 = b ^ t6; t8 = t3 | x1; t9 = a & t4; x3 = t1 ^ t9; t11 = t4 ^ t8; t12 = x3 & t11; x2 = t3 ^ t12; x0 = (~t11) ^ (x3 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);

		// Give difference
		ciphertext0 = x0; ciphertext1 = x1; ciphertext2 = x2; ciphertext3 = x3;
		x0 = plaintext0 ^ 0x00000000;
		x1 = plaintext1 ^ 0x00000000;
		x2 = plaintext2 ^ 0x00010000;
		x3 = plaintext3 ^ 0x00000000;
		//00000000000100000000000000000000

		// Round 2
		x0 ^= k[8];	x1 ^= k[9];	x2 ^= k[19];	x3 ^= k[11];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = b ^ d; t3 = c & t1; x0 = t2 ^ t3; t5 = c ^ t1; t6 = c ^ x0; t7 = b & t6; x3 = t5 ^ t7; x2 = a ^ ((d | t7) & (x0 | t5)); x1 = (t2 ^ x3) ^ (x2 ^ (d | t1));
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 3
		x0 ^= k[12];	x1 ^= k[13];	x2 ^= k[14];	x3 ^= k[15];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ b; t2 = a & c; t3 = a | d; t4 = c ^ d; t5 = t1 & t3; t6 = t2 | t5; x2 = t4 ^ t6; t8 = b ^ t3; t9 = t6 ^ t8; t10 = t4 & t9; x0 = t1 ^ t10; t12 = x2 & x0; x1 = t9 ^ t12; x3 = (b | d) ^ (t4 ^ t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 4
		x0 ^= k[16];	x1 ^= k[17];	x2 ^= k[18];	x3 ^= k[19];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 5
		x0 ^= k[20];	x1 ^= k[21];	x2 ^= k[22];	x3 ^= k[23];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 6
		x0 ^= k[24];	x1 ^= k[25];	x2 ^= k[26];	x3 ^= k[27];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 7
		x0 ^= k[28];	x1 ^= k[29];	x2 ^= k[30];	x3 ^= k[31];
		a = x0; b = x1; c = x2; d = x3;
		t1 = b ^ c; t2 = c & t1; t3 = d ^ t2; t4 = a ^ t3; t5 = d | t1; t6 = t4 & t5; x1 = b ^ t6; t8 = t3 | x1; t9 = a & t4; x3 = t1 ^ t9; t11 = t4 ^ t8; t12 = x3 & t11; x2 = t3 ^ t12; x0 = (~t11) ^ (x3 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);


		x0 ^= ciphertext0;
		x1 ^= ciphertext1;
		x2 ^= ciphertext2;
		x3 ^= ciphertext3;

		t0 = x0 & 0x84001080;
		t1 = x1 & 0x0c284084;
		t2 = x2 & 0x00090800;
		t3 = x3 & 0x20000084;
		//20000084000908000c28408484001080

		t0 ^= t0 >> 1;
		t0 ^= t0 >> 2;
		t0 = (t0 & 0x11111111U) * 0x11111111U;
		t0 = (t0 >> 28) & 1;

		t1 ^= t1 >> 1;
		t1 ^= t1 >> 2;
		t1 = (t1 & 0x11111111U) * 0x11111111U;
		t1 = (t1 >> 28) & 1;

		t2 ^= t2 >> 1;
		t2 ^= t2 >> 2;
		t2 = (t2 & 0x11111111U) * 0x11111111U;
		t2 = (t2 >> 28) & 1;

		t3 ^= t3 >> 1;
		t3 ^= t3 >> 2;
		t3 = (t3 & 0x11111111U) * 0x11111111U;
		t3 = (t3 >> 28) & 1;

		t0 = t0 ^ t1 ^ t2 ^ t3;

		if (t0 == 0) counter++;

		// Change plaintext
		plaintext0 = x0; plaintext1 = x1; plaintext2 = x2; plaintext3 = x3;
	}
	hit[blockIdx.x * blockDim.x + threadIdx.x] = counter;
}
__global__ void Serpent_benchmark(bit32 p0, bit32 p1, bit32 p2, bit32 p3, bit64* hit, bit32* k_d) {
	__shared__ bit32  k[132];
	bit32 x0, x1, x2, x3, plaintext0 = p0, plaintext1 = p1, plaintext2 = p2, plaintext3 = p3 + blockIdx.x * blockDim.x + threadIdx.x;
	bit32 ciphertext0;
	bit32 a, b, c, d;
	bit32 t1, t2, t3, t4, t5, t6, t7, t8, t9, t10, t11, t12;
	bit64 counter = 0;
	if (threadIdx.x < 132) k[threadIdx.x] = k_d[threadIdx.x];
	__syncthreads();
	for (int i = 0; i < loop_d; i++) {
		x0 = plaintext0; x1 = plaintext1; x2 = plaintext2; x3 = plaintext3;
//#pragma unroll
/*		for (int i = 0; i < 32; i++) {
			x0 ^= k[i*4];	x1 ^= k[i * 4 + 1];	x2 ^= k[i * 4 + 2]; x3 ^= k[i * 4 + 3];
			a = x0; b = x1; c = x2; d = x3;
			t1 = ~a; t2 = b ^ d; t3 = c & t1; x0 = t2 ^ t3; t5 = c ^ t1; t6 = c ^ x0; t7 = b & t6; x3 = t5 ^ t7; x2 = a ^ ((d | t7) & (x0 | t5)); x1 = (t2 ^ x3) ^ (x2 ^ (d | t1));
			x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		}*/

		// Round 0
		x0 ^= k[0];	x1 ^= k[1];	x2 ^= k[2];	x3 ^= k[3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t3 = c ^ t1; t4 = b ^ t3; x3 = (a & d) ^ t4; t7 = a ^ (b & t1); x2 = t4 ^ (c | t7); t12 = x3 & (t3 ^ t7); x1 = (~t3) ^ t12; x0 = t12 ^ (~t7);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 1
		x0 ^= k[4];	x1 ^= k[5];	x2 ^= k[6];	x3 ^= k[7];
		a = x0; b = x1; c = x2; d = x3;
		t2 = b ^ (~a);		t5 = c ^ (a | t2);		x2 = d ^ t5;		t7 = b ^ (d | t2);		t8 = t2 ^ x2;		x3 = t8 ^ (t5 & t7); 		t11 = t5 ^ t7; 		x1 = x3 ^ t11; 		x0 = t5 ^ (t8 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 2
		x0 ^= k[8];	x1 ^= k[9];	x2 ^= k[19];	x3 ^= k[11];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = b ^ d; t3 = c & t1; x0 = t2 ^ t3; t5 = c ^ t1; t6 = c ^ x0; t7 = b & t6; x3 = t5 ^ t7; x2 = a ^ ((d | t7) & (x0 | t5)); x1 = (t2 ^ x3) ^ (x2 ^ (d | t1));
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 3
		x0 ^= k[12];	x1 ^= k[13];	x2 ^= k[14];	x3 ^= k[15];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ b; t2 = a & c; t3 = a | d; t4 = c ^ d; t5 = t1 & t3; t6 = t2 | t5; x2 = t4 ^ t6; t8 = b ^ t3; t9 = t6 ^ t8; t10 = t4 & t9; x0 = t1 ^ t10; t12 = x2 & x0; x1 = t9 ^ t12; x3 = (b | d) ^ (t4 ^ t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 4
		x0 ^= k[16];	x1 ^= k[17];	x2 ^= k[18];	x3 ^= k[19];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 5
		x0 ^= k[20];	x1 ^= k[21];	x2 ^= k[22];	x3 ^= k[23];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 6
		x0 ^= k[24];	x1 ^= k[25];	x2 ^= k[26];	x3 ^= k[27];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 7
		x0 ^= k[28];	x1 ^= k[29];	x2 ^= k[30];	x3 ^= k[31];
		a = x0; b = x1; c = x2; d = x3;
		t1 = b ^ c; t2 = c & t1; t3 = d ^ t2; t4 = a ^ t3; t5 = d | t1; t6 = t4 & t5; x1 = b ^ t6; t8 = t3 | x1; t9 = a & t4; x3 = t1 ^ t9; t11 = t4 ^ t8; t12 = x3 & t11; x2 = t3 ^ t12; x0 = (~t11) ^ (x3 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 8
		x0 ^= k[32];	x1 ^= k[33];	x2 ^= k[34];	x3 ^= k[35];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t3 = c ^ t1; t4 = b ^ t3; x3 = (a & d) ^ t4; t7 = a ^ (b & t1); x2 = t4 ^ (c | t7); t12 = x3 & (t3 ^ t7); x1 = (~t3) ^ t12; x0 = t12 ^ (~t7);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 9
		x0 ^= k[36];	x1 ^= k[37];	x2 ^= k[38];	x3 ^= k[39];
		a = x0; b = x1; c = x2; d = x3;
		t2 = b ^ (~a);		t5 = c ^ (a | t2);		x2 = d ^ t5;		t7 = b ^ (d | t2);		t8 = t2 ^ x2;		x3 = t8 ^ (t5 & t7); 		t11 = t5 ^ t7; 		x1 = x3 ^ t11; 		x0 = t5 ^ (t8 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 10
		x0 ^= k[40];	x1 ^= k[41];	x2 ^= k[42];	x3 ^= k[43];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = b ^ d; t3 = c & t1; x0 = t2 ^ t3; t5 = c ^ t1; t6 = c ^ x0; t7 = b & t6; x3 = t5 ^ t7; x2 = a ^ ((d | t7) & (x0 | t5)); x1 = (t2 ^ x3) ^ (x2 ^ (d | t1));
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 11
		x0 ^= k[44];	x1 ^= k[45];	x2 ^= k[46];	x3 ^= k[47];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ b; t2 = a & c; t3 = a | d; t4 = c ^ d; t5 = t1 & t3; t6 = t2 | t5; x2 = t4 ^ t6; t8 = b ^ t3; t9 = t6 ^ t8; t10 = t4 & t9; x0 = t1 ^ t10; t12 = x2 & x0; x1 = t9 ^ t12; x3 = (b | d) ^ (t4 ^ t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 12
		x0 ^= k[48];	x1 ^= k[49];	x2 ^= k[50];	x3 ^= k[51];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 13
		x0 ^= k[52];	x1 ^= k[53];	x2 ^= k[54];	x3 ^= k[55];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 14
		x0 ^= k[56];	x1 ^= k[57];	x2 ^= k[58];	x3 ^= k[59];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 15
		x0 ^= k[60];	x1 ^= k[61];	x2 ^= k[62];	x3 ^= k[63];
		a = x0; b = x1; c = x2; d = x3;
		t1 = b ^ c; t2 = c & t1; t3 = d ^ t2; t4 = a ^ t3; t5 = d | t1; t6 = t4 & t5; x1 = b ^ t6; t8 = t3 | x1; t9 = a & t4; x3 = t1 ^ t9; t11 = t4 ^ t8; t12 = x3 & t11; x2 = t3 ^ t12; x0 = (~t11) ^ (x3 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);

		// Round 16
		x0 ^= k[64];	x1 ^= k[65];	x2 ^= k[66];	x3 ^= k[67];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d;		t3 = c ^ t1;		t4 = b ^ t3;		x3 = (a & d) ^ t4;		t7 = a ^ (b & t1);		x2 = t4 ^ (c | t7);		t12 = x3 & (t3 ^ t7);		x1 = (~t3) ^ t12;		x0 = t12 ^ (~t7);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 17
		x0 ^= k[68];	x1 ^= k[69];	x2 ^= k[70];	x3 ^= k[71];
		a = x0; b = x1; c = x2; d = x3;
		t2 = b ^ (~a);		t5 = c ^ (a | t2);		x2 = d ^ t5;		t7 = b ^ (d | t2);		t8 = t2 ^ x2;		x3 = t8 ^ (t5 & t7); 		t11 = t5 ^ t7; 		x1 = x3 ^ t11; 		x0 = t5 ^ (t8 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 18
		x0 ^= k[72];	x1 ^= k[73];	x2 ^= k[74];	x3 ^= k[75];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = b ^ d; t3 = c & t1; x0 = t2 ^ t3; t5 = c ^ t1; t6 = c ^ x0; t7 = b & t6; x3 = t5 ^ t7; x2 = a ^ ((d | t7) & (x0 | t5)); x1 = (t2 ^ x3) ^ (x2 ^ (d | t1));
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 19
		x0 ^= k[76];	x1 ^= k[77];	x2 ^= k[78];	x3 ^= k[79];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ b; t2 = a & c; t3 = a | d; t4 = c ^ d; t5 = t1 & t3; t6 = t2 | t5; x2 = t4 ^ t6; t8 = b ^ t3; t9 = t6 ^ t8; t10 = t4 & t9; x0 = t1 ^ t10; t12 = x2 & x0; x1 = t9 ^ t12; x3 = (b | d) ^ (t4 ^ t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 20
		x0 ^= k[80];	x1 ^= k[81];	x2 ^= k[82];	x3 ^= k[83];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 21
		x0 ^= k[84];	x1 ^= k[85];	x2 ^= k[86];	x3 ^= k[87];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 22
		x0 ^= k[88];	x1 ^= k[89];	x2 ^= k[90];	x3 ^= k[91];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 23
		x0 ^= k[92];	x1 ^= k[93];	x2 ^= k[94];	x3 ^= k[95];
		a = x0; b = x1; c = x2; d = x3;
		t1 = b ^ c; t2 = c & t1; t3 = d ^ t2; t4 = a ^ t3; t5 = d | t1; t6 = t4 & t5; x1 = b ^ t6; t8 = t3 | x1; t9 = a & t4; x3 = t1 ^ t9; t11 = t4 ^ t8; t12 = x3 & t11; x2 = t3 ^ t12; x0 = (~t11) ^ (x3 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 24
		x0 ^= k[96];	x1 ^= k[97];	x2 ^= k[98];	x3 ^= k[99];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t3 = c ^ t1; t4 = b ^ t3; x3 = (a & d) ^ t4; t7 = a ^ (b & t1); x2 = t4 ^ (c | t7); t12 = x3 & (t3 ^ t7); x1 = (~t3) ^ t12; x0 = t12 ^ (~t7);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 25
		x0 ^= k[100];	x1 ^= k[101];	x2 ^= k[102];	x3 ^= k[103];
		a = x0; b = x1; c = x2; d = x3;
		t2 = b ^ (~a);		t5 = c ^ (a | t2);		x2 = d ^ t5;		t7 = b ^ (d | t2);		t8 = t2 ^ x2;		x3 = t8 ^ (t5 & t7); 		t11 = t5 ^ t7; 		x1 = x3 ^ t11; 		x0 = t5 ^ (t8 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 26
		x0 ^= k[104];	x1 ^= k[105];	x2 ^= k[106];	x3 ^= k[107];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = b ^ d; t3 = c & t1; x0 = t2 ^ t3; t5 = c ^ t1; t6 = c ^ x0; t7 = b & t6; x3 = t5 ^ t7; x2 = a ^ ((d | t7) & (x0 | t5)); x1 = (t2 ^ x3) ^ (x2 ^ (d | t1));
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 27
		x0 ^= k[108];	x1 ^= k[109];	x2 ^= k[110];	x3 ^= k[111];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ b; t2 = a & c; t3 = a | d; t4 = c ^ d; t5 = t1 & t3; t6 = t2 | t5; x2 = t4 ^ t6; t8 = b ^ t3; t9 = t6 ^ t8; t10 = t4 & t9; x0 = t1 ^ t10; t12 = x2 & x0; x1 = t9 ^ t12; x3 = (b | d) ^ (t4 ^ t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 28
		x0 ^= k[112];	x1 ^= k[113];	x2 ^= k[114];	x3 ^= k[115];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 29
		x0 ^= k[116];	x1 ^= k[117];	x2 ^= k[118];	x3 ^= k[119];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 30
		x0 ^= k[120];	x1 ^= k[121];	x2 ^= k[122];	x3 ^= k[123];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 31
		x0 ^= k[124];	x1 ^= k[125];	x2 ^= k[126];	x3 ^= k[127];
		a = x0; b = x1; c = x2; d = x3;
		t1 = b ^ c; t2 = c & t1; t3 = d ^ t2; t4 = a ^ t3; t5 = d | t1; t6 = t4 & t5; x1 = b ^ t6; t8 = t3 | x1; t9 = a & t4; x3 = t1 ^ t9; t11 = t4 ^ t8; t12 = x3 & t11; x2 = t3 ^ t12; x0 = (~t11) ^ (x3 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 31
		x0 ^= k[128];	x1 ^= k[129];	x2 ^= k[130];	x3 ^= k[131];

		if (x0 == 0)
			if (x1 == 0)
				if (x2 == 0)
					if (x3 == 0)
						printf("Hello world\n");

		plaintext0 = x0; plaintext1 = x1; plaintext2 = x2; plaintext3 = x3;
	}
	hit[blockIdx.x * blockDim.x + threadIdx.x] = counter;
}
__global__ void Serpent5round_eprint(bit32 p0, bit32 p1, bit32 p2, bit32 p3, bit64* hit, bit32* k_d) {
	__shared__ bit32  k[132];
	bit32 x0, x1, x2, x3, plaintext0 = p0, plaintext1 = p1, plaintext2 = p2, plaintext3 = p3 + blockIdx.x * blockDim.x + threadIdx.x;
	bit32 ciphertext0, ciphertext1, ciphertext2, ciphertext3;
	bit32 a, b, c, d;
	bit32 t0, t1, t2, t3, t4, t5, t6, t7, t8, t9, t10, t11, t12;
	bit32 round;
	bit64 counter = 0;
	if (threadIdx.x < 132) k[threadIdx.x] = k_d[threadIdx.x];
	__syncthreads();
	for (int i = 0; i < loop_d; i++) {
		x0 = plaintext0; x1 = plaintext1; x2 = plaintext2; x3 = plaintext3;
		// Round 2
		round = 2;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = b ^ d; t3 = c & t1; x0 = t2 ^ t3; t5 = c ^ t1; t6 = c ^ x0; t7 = b & t6; x3 = t5 ^ t7; x2 = a ^ ((d | t7) & (x0 | t5)); x1 = (t2 ^ x3) ^ (x2 ^ (d | t1));
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 3
		round = 3;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ b; t2 = a & c; t3 = a | d; t4 = c ^ d; t5 = t1 & t3; t6 = t2 | t5; x2 = t4 ^ t6; t8 = b ^ t3; t9 = t6 ^ t8; t10 = t4 & t9; x0 = t1 ^ t10; t12 = x2 & x0; x1 = t9 ^ t12; x3 = (b | d) ^ (t4 ^ t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 4
		round = 4;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 5
		round = 5;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 6
		round = 6;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		// no linear transformation for the last round

			// Last round linear transformation
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);

		ciphertext0 = x0;
		ciphertext1 = x1;
		ciphertext2 = x2;
		ciphertext3 = x3;
		// Give difference	(eprint)	

		x0 = plaintext0 ^ (0x01000000);
		x1 = plaintext1 ^ (0x01000000);
		x2 = plaintext2 ^ (0x09000000);
		x3 = plaintext3 ^ (0x00000000);

		// Round 2
		round = 2;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = b ^ d; t3 = c & t1; x0 = t2 ^ t3; t5 = c ^ t1; t6 = c ^ x0; t7 = b & t6; x3 = t5 ^ t7; x2 = a ^ ((d | t7) & (x0 | t5)); x1 = (t2 ^ x3) ^ (x2 ^ (d | t1));
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 3
		round = 3;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ b; t2 = a & c; t3 = a | d; t4 = c ^ d; t5 = t1 & t3; t6 = t2 | t5; x2 = t4 ^ t6; t8 = b ^ t3; t9 = t6 ^ t8; t10 = t4 & t9; x0 = t1 ^ t10; t12 = x2 & x0; x1 = t9 ^ t12; x3 = (b | d) ^ (t4 ^ t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 4
		round = 4;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = a ^ d; t2 = d & t1; t3 = c ^ t2; t4 = b | t3; x3 = t1 ^ t4; t6 = ~b; t7 = t1 | t6; x0 = t3 ^ t7; t9 = a & x0; t10 = t1 ^ t6; t11 = t4 & t10; x2 = t9 ^ t11; x1 = (a ^ t3) ^ (t10 & x2);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 5
		round = 5;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ b; t3 = a ^ d; t4 = c ^ t1; t5 = t2 | t3; x0 = t4 ^ t5; t7 = d & x0; t8 = t2 ^ x0; x1 = t7 ^ t8; t10 = t1 | x0; t11 = t2 | t7; t12 = t3 ^ t10; x2 = t11 ^ t12; x3 = (b ^ t7) ^ (x1 & t12);
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);
		// Round 6
		round = 6;
		x0 = x0 ^ k[4 * round];	x1 = x1 ^ k[4 * round + 1];	x2 = x2 ^ k[4 * round + 2];	x3 = x3 ^ k[4 * round + 3];
		a = x0; b = x1; c = x2; d = x3;
		t1 = ~a; t2 = a ^ d; t3 = b ^ t2; t4 = t1 | t2; t5 = c ^ t4; x1 = b ^ t5; t7 = t2 | x1; t8 = d ^ t7; t9 = t5 & t8; x2 = t3 ^ t9; t11 = t5 ^ t8; x0 = x2 ^ t11; x3 = (~t5) ^ (t3 & t11);
		// no linear transformation for the last round

			// Last round linear transformation
		x0 = (x0 << 13) | x0 >> 19;	x2 = (x2 << 3) | (x2 >> 29); x1 = x1 ^ x0 ^ x2;	x3 = x3 ^ x2 ^ (x0 << 3); x1 = (x1 << 1) | (x1 >> 31); x3 = (x3 << 7) | (x3 >> 25);	x0 = x0 ^ x1 ^ x3; x2 = x2 ^ x3 ^ (x1 << 7); x0 = (x0 << 5) | (x0 >> 27); x2 = (x2 << 22) | (x2 >> 10);

		// Check Difference
		x0 ^= ciphertext0;
		x1 ^= ciphertext1;
		x2 ^= ciphertext2;
		x3 ^= ciphertext3;

		// Masked bits (eprint)
/*		t0 = x0 & 0x84001080;
		t1 = x1 & 0x0c284084;
		t2 = x2 & 0x00090800;
		t3 = x3 & 0x20000084;*/

		// without the final transformation
/*		t0 = x0 & 0x00000004;
		t1 = x1 & 0x00000004;
		t2 = x2 & 0x00000000;
		t3 = x3 & 0x00000004;*/

		t0 = x0 & 0x42004000;
		t1 = x1 & 0x02100228;
		t2 = x2 & 0x00000005;
		t3 = x3 & 0x02181600;



		// Our best mask
/*		t0 = x0 & 0x02100000;
		t1 = x1 & 0x00000000;
		t2 = x2 & 0x00000000;
		t3 = x3 & 0x00000000;*/

		t0 ^= t0 >> 1;
		t0 ^= t0 >> 2;
		t0 = (t0 & 0x11111111U) * 0x11111111U;
		t0 = (t0 >> 28) & 1;

		t1 ^= t1 >> 1;
		t1 ^= t1 >> 2;
		t1 = (t1 & 0x11111111U) * 0x11111111U;
		t1 = (t1 >> 28) & 1;

		t2 ^= t2 >> 1;
		t2 ^= t2 >> 2;
		t2 = (t2 & 0x11111111U) * 0x11111111U;
		t2 = (t2 >> 28) & 1;

		t3 ^= t3 >> 1;
		t3 ^= t3 >> 2;
		t3 = (t3 & 0x11111111U) * 0x11111111U;
		t3 = (t3 >> 28) & 1;

		t0 = t0 ^ t1 ^ t2 ^ t3;

		if (t0 == 0) counter++;

		// Change plaintext
		plaintext0 = x0; plaintext1 = x1; plaintext2 = x2; plaintext3 = x3;
	}
	hit[blockIdx.x * blockDim.x + threadIdx.x] = counter;
}

int main() {	
	FILE* fp3, * fp, * fp2, * fp4, * fp5;
	bit64* hit_d;
	bit64* hit_test;
	__int64 temp, hit, bias = 0, cumulative_bias[100] = { 0 }, average_bias;
	bit32* k_d, keylist[100][4];
	hit_test = (bit64*)calloc(totalthreads, sizeof(bit64));
	int t, j, i;
	hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
	get_user_inputs();
	hipSetDevice(devicenumber);
	hipMalloc((void**)&hit_d, totalthreads * sizeof(bit64));
	hipMalloc((void**)&k_d, 132 * sizeof(bit32));
	if (numberofrounds == 3) fp = fopen("3round_cumulative.txt", "ab");
	else if (numberofrounds == 4) fp = fopen("4round_cumulative.txt", "ab");
	else if (numberofrounds == 5) fp = fopen("5round_cumulative.txt", "ab");
	else if (numberofrounds == 66) fp = fopen("66round_cumulative.txt", "ab");
	else if (numberofrounds == 74) fp = fopen("74round_cumulative.txt", "ab");
	else if (numberofrounds == 75) fp = fopen("75round_cumulative.txt", "ab");
	else if (numberofrounds == 76) fp = fopen("76round_cumulative.txt", "ab");
	else if (numberofrounds == 32) fp = fopen("32round_benchmark.txt", "ab");
	else exit(1);
	fprintf(fp, "Pairs: %I64d\n", blocksize * threadsize * loop * pairs);
	plaintext[0] = 0xc7675a6e; plaintext[1] = 0xe3d21628; plaintext[2] = 0x9f090c02; plaintext[3] = 0x5eeaf89d;
	fp3 = fopen("keys.txt", "r");
	for (i = 0; i < 100; i++) { fscanf(fp3, "%x", &keylist[i][0]);	fscanf(fp3, "%x", &keylist[i][1]);	fscanf(fp3, "%x", &keylist[i][2]);	fscanf(fp3, "%x", &keylist[i][3]); }
	fclose(fp3);
	if (startingpoint) {
		fp4 = fopen("startingpoint.txt", "r");	for (i = 0; i < experiment; i++) { fscanf(fp4, "%I64d", &cumulative_bias[i]); printf("%d: %I64d\n", i, cumulative_bias[i]); }	fclose(fp4);
		for (i = 0; i < startingpoint; i++) { plaintext[0]++; plaintext[1]++; plaintext[2]++; plaintext[3]++; }
	}
	for (j = 0; j < pairs; j++) {
		StartCounter();
		for (t = 0; t < experiment; t++) {
			hit = 0;
			key[0] = keylist[t][0]; key[1] = keylist[t][1]; key[2] = keylist[t][2]; key[3] = keylist[t][3];
			key_schedule();
			hipMemcpy(k_d, k, 132 * sizeof(bit32), hipMemcpyHostToDevice);
			hipMemcpy(hit_d, hit_test, totalthreads * sizeof(bit64), hipMemcpyHostToDevice);
			
			if (numberofrounds == 3) Serpent3round_PZWD24 << <blocksize, threadsize >> > (plaintext[0], plaintext[1], plaintext[2], plaintext[3], hit_d, k_d);
			else if (numberofrounds == 4) Serpent4round << <blocksize, threadsize >> > (plaintext[0], plaintext[1], plaintext[2], plaintext[3], hit_d, k_d);
			else if (numberofrounds == 5) Serpent5round_eprint << <blocksize, threadsize >> > (plaintext[0], plaintext[1], plaintext[2], plaintext[3], hit_d, k_d);
			else if (numberofrounds == 66) Serpent6round_PZWD24_final << <blocksize, threadsize >> > (plaintext[0], plaintext[1], plaintext[2], plaintext[3], hit_d, k_d);
			else if (numberofrounds == 74) Serpent7round_PZWD24_middle4 << <blocksize, threadsize >> > (plaintext[0], plaintext[1], plaintext[2], plaintext[3], hit_d, k_d);
			else if (numberofrounds == 75) Serpent7round_PZWD24_middle5 << <blocksize, threadsize >> > (plaintext[0], plaintext[1], plaintext[2], plaintext[3], hit_d, k_d);
			else if (numberofrounds == 76) Serpent7round_PZWD24_middle6 << <blocksize, threadsize >> > (plaintext[0], plaintext[1], plaintext[2], plaintext[3], hit_d, k_d);
			else if (numberofrounds == 32) {
				float milliseconds = 0;
				hipDeviceSynchronize();
				hipEvent_t start, stop;	hipEventCreate(&start);	hipEventCreate(&stop);	hipEventRecord(start);
				Serpent_benchmark << <blocksize, threadsize >> > (plaintext[0], plaintext[1], plaintext[2], plaintext[3], hit_d, k_d);
				hipEventRecord(stop);	hipEventSynchronize(stop);
				hipEventElapsedTime(&milliseconds, start, stop);	printf("Time elapsed: %f milliseconds\n", milliseconds);
				exit(1);
			}
			hipMemcpy(hit_test, hit_d, totalthreads * sizeof(bit64), hipMemcpyDeviceToHost);
			for (i = 0; i < totalthreads; i++) hit += hit_test[i];
			cumulative_bias[t] += (hit - totalthreads*loop_d/2);
			//			printf("%2d: Key bias %I64d\n", t, cumulative_bias[t]);
			fprintf(fp, "%I64d\n", cumulative_bias[t]);
		}
		average_bias = 0;
		for (i = 0; i < experiment; i++) average_bias += cumulative_bias[i];
		average_bias /= experiment;
		printf("%5d: Time: %lf Average Bias %I64d\n", j + startingpoint, GetCounter(), average_bias);
		printf("\nTotal counter: %I64d\nDifference from the Expected Value: %I64d\nBias: 2^-%lf (For an experiment with 2^%lf data)\n", hit, average_bias, ((log(totalthreads * loop_d * (j+1))) / log(2)) - (log(abs(average_bias)) / log(2)), (log(totalthreads * loop_d * (j+1))) / log(2));
		fprintf(fp, "%5d: Time: %lf Average Bias %I64d\n", j + startingpoint, GetCounter(), average_bias);
		if (numberofrounds == 3) fp2 = fopen("3round_cumulative_bias.txt", "ab");
		else if (numberofrounds == 4) fp2 = fopen("4round_cumulative_bias.txt", "ab");
		else if (numberofrounds == 5) fp2 = fopen("5round_cumulative_bias.txt", "ab");
		else if (numberofrounds == 66) fp2 = fopen("66round_cumulative_bias.txt", "ab");
		else if (numberofrounds == 74) fp2 = fopen("74round_cumulative_bias.txt", "ab");
		else if (numberofrounds == 75) fp2 = fopen("75round_cumulative_bias.txt", "ab");
		else if (numberofrounds == 76) fp2 = fopen("76round_cumulative_bias.txt", "ab");
		fprintf(fp2, "%5d: Time: %lf Average Bias %I64d\n", j + startingpoint, GetCounter(), average_bias);
		fclose(fp2);
		fp5 = fopen("6round_cumulative_bias_shortened.txt", "ab");
		fprintf(fp5, "%I64d\n", average_bias);
		fclose(fp5);
		plaintext[0]++; plaintext[1]++; plaintext[2]++; plaintext[3]++;
	}
	hipFree(hit_d); hipFree(k_d);
	fclose(fp);
	free(hit_test);
	printf("%s\n", hipGetErrorString(hipGetLastError()));
	return 0;
}

